#include "hip/hip_runtime.h"
#include "cuda_sparse.h"
#include "cuda_fields.h"
#include "cuda_sparse_types.h"
#include "globals.h"
#include "consts.h"

#include "cuda_srk4llg.h"
#include "cuda_srk4llg_kernel.h"

#include <hiprand.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsparse.h>

#include <cmath>

#include <containers/Array.h>


void CUDALLGSolverSRK4::syncOutput()
{
    using namespace globals;
    CUDA_CALL(hipMemcpy(s.data(),s_dev,(size_t)(nspins3*sizeof(double)),hipMemcpyDeviceToHost));
}

void CUDALLGSolverSRK4::initialise(int argc, char **argv, double idt)
{
    using namespace globals;

    // initialise base class
    Solver::initialise(argc,argv,idt);

    sigma.resize(nspins);

    for(int i=0; i<nspins; ++i) {
        sigma(i) = sqrt( (2.0*boltzmann_si*alpha(i)) / (dt*mus(i)*mu_bohr_si) );
    }


    output.write("  * CUDA SRK4 LLG solver (GPU)\n");

    //-------------------------------------------------------------------
    //  Initialise hiprand
    //-------------------------------------------------------------------

    output.write("  * Initialising CURAND...\n");
    // hiprand generator
    CURAND_CALL(hiprandCreateGenerator(&gen,HIPRAND_RNG_PSEUDO_DEFAULT));

    // TODO: set random seed from config
    const unsigned long long gpuseed = rng.uniform()*18446744073709551615ULL;
    CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen, gpuseed));
    CURAND_CALL(hiprandGenerateSeeds(gen));

    //-------------------------------------------------------------------
    //  Allocate device memory
    //-------------------------------------------------------------------

    output.write("  * Converting MAP to DIA\n");
    J1ij_s.convertMAP2DIA();
    J1ij_t.convertMAP2DIA();
    J2ij_s.convertMAP2DIA();
    J2ij_t.convertMAP2DIA();

    output.write("    - J1ij scalar matrix memory (DIA): %f MB\n",J1ij_s.calculateMemory());
    output.write("    - J1ij tensor matrix memory (DIA): %f MB\n",J1ij_t.calculateMemory());
    output.write("    - J2ij scalar matrix memory (DIA): %f MB\n",J2ij_s.calculateMemory());
    output.write("    - J2ij tensor matrix memory (DIA): %f MB\n",J2ij_t.calculateMemory());

    output.write("    - J4ijkl scalar matrix memory (CSR): %f MB\n",J4ijkl_s.calculateMemoryUsage());

    output.write("  * Allocating device memory...\n");

    // Allocate double arrays
    CUDA_CALL(hipMalloc((void**)&s_dev,nspins3*sizeof(double)));   // 3*nspins
    CUDA_CALL(hipMalloc((void**)&s_old_dev,nspins3*sizeof(double)));   // 3*nspins
    CUDA_CALL(hipMalloc((void**)&k0_dev,nspins3*sizeof(double)));  // 3*nspins
    CUDA_CALL(hipMalloc((void**)&k1_dev,nspins3*sizeof(double)));  // 3*nspins
    CUDA_CALL(hipMalloc((void**)&k2_dev,nspins3*sizeof(double)));  // 3*nspins

    // Allocate float arrays
    CUDA_CALL(hipMalloc((void**)&sf_dev,nspins3*sizeof(float)));       // 3*nspins
    CUDA_CALL(hipMalloc((void**)&h_dev,nspins3*sizeof(float)));        // 3*nspins
    CUDA_CALL(hipMalloc((void**)&h_dipole_dev,nspins3*sizeof(float)));  // 3*nspins
    CUDA_CALL(hipMalloc((void**)&e_dev,nspins3*sizeof(float)));        // 3*nspins
    CUDA_CALL(hipMalloc((void**)&r_dev,nspins3*sizeof(float)));        // 3*nspins
    CUDA_CALL(hipMalloc((void**)&r_max_dev,3*sizeof(float)));          // 3
    CUDA_CALL(hipMalloc((void**)&pbc_dev,3*sizeof(bool)));             // 3
    CUDA_CALL(hipMalloc((void**)&mat_dev,nspins*4*sizeof(float)));     // 4*nspins

    // CURAND requires that the array is a multiple of 2
    CUDA_CALL(hipMalloc((void**)&w_dev,(nspins3+(nspins3%2))*sizeof(float)));  // 3*nspins (+1 if odd)

    //-------------------------------------------------------------------
    //  Transfer data to device memory
    //-------------------------------------------------------------------

    allocate_transfer_dia(J1ij_s, J1ij_s_dev);
    allocate_transfer_dia(J1ij_t, J1ij_t_dev);
    allocate_transfer_dia(J2ij_s, J2ij_s_dev);
    allocate_transfer_dia(J2ij_t, J2ij_t_dev);
    allocate_transfer_csr_4d(J4ijkl_s, J4ijkl_s_dev);

    //-------------------------------------------------------------------
    //  Copy data to device
    //-------------------------------------------------------------------

    output.write("  * Copying data to device memory...\n");


    // Initial spin configuration
    {
        jbLib::Array<float,2> sf(nspins,3);
            for(int i=0; i<nspins; ++i) {
                for(int j=0; j<3; ++j) {
                    sf(i,j) = static_cast<float>(s(i,j));
                }
            }
            
        CUDA_CALL(hipMemcpy(s_dev,s.data(),(size_t)(nspins3*sizeof(double)),hipMemcpyHostToDevice));
        CUDA_CALL(hipMemcpy(s_old_dev,s.data(),(size_t)(nspins3*sizeof(double)),hipMemcpyHostToDevice));
        CUDA_CALL(hipMemcpy(sf_dev,sf.data(),(size_t)(nspins3*sizeof(float)),hipMemcpyHostToDevice));
    }

    // Lattice dimensions
    {
        float r_maxf[3];
        lattice.getMaxDimensions(r_maxf[0],r_maxf[1],r_maxf[2]);
        CUDA_CALL(hipMemcpy(r_max_dev,r_maxf,(size_t)(3*sizeof(float)),hipMemcpyHostToDevice));
    }

    // Periodic boundary conditions
    {
        bool pbc[3];
        lattice.getBoundaries(pbc[0],pbc[1],pbc[2]);
        CUDA_CALL(hipMemcpy(pbc_dev,pbc,(size_t)(3*sizeof(bool)),hipMemcpyHostToDevice));
    }
    
    // Atom positions
    CUDA_CALL(hipMemcpy(r_dev,atom_pos.data(),(size_t)(nspins3*sizeof(float)),hipMemcpyHostToDevice));

    // Material properties
    {
        jbLib::Array<float,2> mat(nspins,4);
        for(int i=0; i<nspins; ++i){
            mat(i,0) = mus(i);
            mat(i,1) = gyro(i);
            mat(i,2) = alpha(i);
            mat(i,3) = sigma(i);
        }

        CUDA_CALL(hipMemcpy(mat_dev,mat.data(),(size_t)(nspins*4*sizeof(float)),hipMemcpyHostToDevice));
    }



    //-------------------------------------------------------------------
    //  Initialise arrays to zero
    //-------------------------------------------------------------------

    CUDA_CALL(hipMemset(w_dev, 0, nspins3*sizeof(float)));
    CUDA_CALL(hipMemset(h_dev, 0, nspins3*sizeof(float)));
    CUDA_CALL(hipMemset(h_dipole_dev, 0, nspins3*sizeof(float)));
    CUDA_CALL(hipMemset(e_dev, 0, nspins3*sizeof(float)));

    //-------------------------------------------------------------------
    //  Determine blocksizes
    //-------------------------------------------------------------------
    nblocks = (nspins+BLOCKSIZE-1)/BLOCKSIZE;

    
    J1ij_s_dev.blocks = std::min<int>(DIA_BLOCK_SIZE,(nspins+DIA_BLOCK_SIZE-1)/DIA_BLOCK_SIZE);
    J1ij_t_dev.blocks = std::min<int>(DIA_BLOCK_SIZE,(nspins3+DIA_BLOCK_SIZE-1)/DIA_BLOCK_SIZE);
    J2ij_s_dev.blocks = std::min<int>(DIA_BLOCK_SIZE,(nspins+DIA_BLOCK_SIZE-1)/DIA_BLOCK_SIZE);
    J2ij_t_dev.blocks = std::min<int>(DIA_BLOCK_SIZE,(nspins3+DIA_BLOCK_SIZE-1)/DIA_BLOCK_SIZE);
    J4ijkl_s_dev.blocks = std::min<int>(CSR_4D_BLOCK_SIZE,(nspins+CSR_4D_BLOCK_SIZE-1)/CSR_4D_BLOCK_SIZE);

    eng.resize(nspins,3);

    initialised = true;
}

void CUDALLGSolverSRK4::run()
{
  using namespace globals;

  // generate wiener trajectories
  float stmp = sqrt(temperature);
  
  if(temperature > 0.0) {
      CURAND_CALL(hiprandGenerateNormal(gen, w_dev, (nspins3+(nspins3%2)), 0.0f, stmp));
  }
  
    CUDACalculateFields(J1ij_s_dev,J1ij_t_dev,J2ij_s_dev,J2ij_t_dev,J4ijkl_s_dev,sf_dev,r_dev,r_max_dev,mat_dev,pbc_dev,h_dev,h_dipole_dev,true);
  
  // Integrate to find K0
  CUDAIntegrateLLG_SRK4<<<nblocks,BLOCKSIZE>>>
    (s_dev,s_old_dev,k0_dev,h_dev,w_dev,sf_dev,mat_dev,h_app[0],h_app[1],h_app[2],0.5,dt,nspins);
  
    CUDACalculateFields(J1ij_s_dev,J1ij_t_dev,J2ij_s_dev,J2ij_t_dev,J4ijkl_s_dev,sf_dev,r_dev,r_max_dev,mat_dev,pbc_dev,h_dev,h_dipole_dev,false);
  
  // Integrate to find K1
  CUDAIntegrateLLG_SRK4<<<nblocks,BLOCKSIZE>>>
    (s_dev,s_old_dev,k1_dev,h_dev,w_dev,sf_dev,mat_dev,h_app[0],h_app[1],h_app[2],0.5,dt,nspins);
  
    CUDACalculateFields(J1ij_s_dev,J1ij_t_dev,J2ij_s_dev,J2ij_t_dev,J4ijkl_s_dev,sf_dev,r_dev,r_max_dev,mat_dev,pbc_dev,h_dev,h_dipole_dev,false);
  
  // Integrate to find K2
  CUDAIntegrateLLG_SRK4<<<nblocks,BLOCKSIZE>>>
    (s_dev,s_old_dev,k2_dev,h_dev,w_dev,sf_dev,mat_dev,h_app[0],h_app[1],h_app[2],1.0,dt,nspins);
  
    CUDACalculateFields(J1ij_s_dev,J1ij_t_dev,J2ij_s_dev,J2ij_t_dev,J4ijkl_s_dev,sf_dev,r_dev,r_max_dev,mat_dev,pbc_dev,h_dev,h_dipole_dev,false);
  
  // Integrate to find K3
  CUDAIntegrateEndPointLLG_SRK4<<<nblocks,BLOCKSIZE>>>
    (s_dev,s_old_dev,k0_dev,k1_dev,k2_dev,h_dev,w_dev,sf_dev,mat_dev,h_app[0],h_app[1],h_app[2],dt,nspins);

  iteration++;
}

void CUDALLGSolverSRK4::calcEnergy(double &e1_s, double &e1_t, double &e2_s, double &e2_t, double &e4_s){
}

CUDALLGSolverSRK4::~CUDALLGSolverSRK4()
{
  hiprandDestroyGenerator(gen);
  
  hipsparseStatus_t status;

  status = hipsparseDestroyMatDescr(descra);
  if (status != HIPSPARSE_STATUS_SUCCESS) {
    jams_error("CUSPARSE matrix destruction failed");
  }
  CUDA_CALL(hipDeviceSynchronize());

  status = hipsparseDestroy(handle);
  if (status != HIPSPARSE_STATUS_SUCCESS) {
    jams_error("CUSPARSE Library destruction failed");
  }
  CUDA_CALL(hipDeviceSynchronize());

  //-------------------------------------------------------------------
  //  Free device memory
  //-------------------------------------------------------------------

  free_dia(J1ij_s_dev);
  free_dia(J1ij_t_dev);
  free_dia(J2ij_s_dev);
  free_dia(J2ij_t_dev);
  free_csr_4d(J4ijkl_s_dev);

  CUDA_CALL(hipFree(s_dev));
  CUDA_CALL(hipFree(s_old_dev));
  CUDA_CALL(hipFree(k0_dev));
  CUDA_CALL(hipFree(k1_dev));
  CUDA_CALL(hipFree(k2_dev));
  CUDA_CALL(hipFree(sf_dev));
  CUDA_CALL(hipFree(w_dev));
  CUDA_CALL(hipFree(r_dev));
  CUDA_CALL(hipFree(r_max_dev));
  CUDA_CALL(hipFree(pbc_dev));
  CUDA_CALL(hipFree(h_dev));
  CUDA_CALL(hipFree(h_dipole_dev));
  CUDA_CALL(hipFree(e_dev));
  CUDA_CALL(hipFree(mat_dev));
}


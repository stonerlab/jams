#include "hip/hip_runtime.h"
#include "cuda_semillg_kernel.cu"
#include "globals.h"
#include "consts.h"

#include "cuda_semillg.h"

#include <hiprand.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsparse.h>

#include <cmath>


#ifndef NDEBUG
#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
  printf("Error at %s:%d\n",__FILE__,__LINE__);\
    exit(EXIT_FAILURE);}} while(0)
#else
#define CUDA_CALL(x) x
#endif

#ifndef NDEBUG
#define CURAND_CALL(x) do { if((x) != HIPRAND_STATUS_SUCCESS) { \
  printf("Error at %s:%d\n",__FILE__,__LINE__);\
  exit(EXIT_FAILURE);}} while(0)
#else
#define CURAND_CALL(x) x
#endif

#if defined(__HIPCC__) && defined(CUDA_NO_SM_13_DOUBLE_INTRINSICS)
    #error "-arch sm_13 nvcc flag is required to compile"
#endif

// block size for GPU, 64 appears to be most efficient for current kernel
#define BLOCKSIZE 64

void CUDASemiLLGSolver::syncOutput()
{
  using namespace globals;
  CUDA_CALL(hipDeviceSynchronize());
  CUDA_CALL(hipMemcpy(s.ptr(),s_dev,(size_t)(nspins3*sizeof(double)),hipMemcpyDeviceToHost));
  CUDA_CALL(hipDeviceSynchronize());
}

void CUDASemiLLGSolver::initialise(int argc, char **argv, double idt)
{
  using namespace globals;

  // initialise base class
  Solver::initialise(argc,argv,idt);

  sigma.resize(nspins);

  for(int i=0; i<nspins; ++i) {
    sigma(i) = sqrt( (2.0*boltzmann_si*alpha(i)) / (dt*mus(i)*mu_bohr_si) );
  }


  output.write("Initialising CUDA Semi Implicit LLG solver (CPU)\n");

  output.write("Initialising CUBLAS\n");

  output.write("Allocating device memory...\n");

  //-------------------------------------------------------------------
  //  Allocate device memory
  //-------------------------------------------------------------------

  // spin arrays
  CUDA_CALL(hipMalloc((void**)&s_dev,nspins3*sizeof(double)));
  CUDA_CALL(hipDeviceSynchronize());
  CUDA_CALL(hipMalloc((void**)&sf_dev,nspins3*sizeof(float)));
  CUDA_CALL(hipDeviceSynchronize());
  CUDA_CALL(hipMalloc((void**)&s_new_dev,nspins3*sizeof(double)));
  CUDA_CALL(hipDeviceSynchronize());

  // field arrays
  CUDA_CALL(hipMalloc((void**)&h_dev,nspins3*sizeof(float)));
  CUDA_CALL(hipDeviceSynchronize());

  if(nspins3%2 == 0) {
    // wiener processes
    CUDA_CALL(hipMalloc((void**)&w_dev,nspins3*sizeof(float)));
  CUDA_CALL(hipDeviceSynchronize());
  } else {
    CUDA_CALL(hipMalloc((void**)&w_dev,(nspins3+1)*sizeof(float)));
  CUDA_CALL(hipDeviceSynchronize());
  }


  // jij matrix
  CUDA_CALL(hipMalloc((void**)&Jij_dev_row,(Jij.rows()+1)*sizeof(int)));
  CUDA_CALL(hipDeviceSynchronize());
  CUDA_CALL(hipMalloc((void**)&Jij_dev_col,Jij.nonZero()*sizeof(int)));
  CUDA_CALL(hipDeviceSynchronize());
  CUDA_CALL(hipMalloc((void**)&Jij_dev_val,Jij.nonZero()*sizeof(float)));
  CUDA_CALL(hipDeviceSynchronize());

  // material properties
  CUDA_CALL(hipMalloc((void**)&mat_dev,nspins*4*sizeof(float)));
  CUDA_CALL(hipDeviceSynchronize());

  //-------------------------------------------------------------------
  //  Copy data to device
  //-------------------------------------------------------------------

  output.write("Copying data to device memory...\n");
  // initial spins
  Array2D<float> sf(nspins,3);
  for(int i=0; i<nspins; ++i) {
    for(int j=0; j<3; ++j) {
      sf(i,j) = static_cast<float>(s(i,j));
    }
  }
  CUDA_CALL(hipMemcpy(s_dev,s.ptr(),(size_t)(nspins3*sizeof(double)),hipMemcpyHostToDevice));
  CUDA_CALL(hipDeviceSynchronize());
  CUDA_CALL(hipMemcpy(sf_dev,sf.ptr(),(size_t)(nspins3*sizeof(float)),hipMemcpyHostToDevice));
  CUDA_CALL(hipDeviceSynchronize());

  // jij matrix
  CUDA_CALL(hipMemcpy(Jij_dev_row,Jij.rowPtr(),
        (size_t)((Jij.rows()+1)*(sizeof(int))),hipMemcpyHostToDevice));
  CUDA_CALL(hipDeviceSynchronize());

  CUDA_CALL(hipMemcpy(Jij_dev_col,Jij.colPtr(),
        (size_t)((Jij.nonZero())*(sizeof(int))),hipMemcpyHostToDevice));
  CUDA_CALL(hipDeviceSynchronize());

  CUDA_CALL(hipMemcpy(Jij_dev_val,Jij.valPtr(),
        (size_t)((Jij.nonZero())*(sizeof(float))),hipMemcpyHostToDevice));
  CUDA_CALL(hipDeviceSynchronize());

  Array2D<float> mat(nspins,4);
  // material properties
  for(int i=0; i<nspins; ++i){
    mat(i,0) = mus(i);
    mat(i,1) = gyro(i);
    mat(i,2) = alpha(i);
    mat(i,3) = sigma(i);
  }
  CUDA_CALL(hipMemcpy(mat_dev,mat.ptr(),(size_t)(nspins*4*sizeof(float)),hipMemcpyHostToDevice));
  CUDA_CALL(hipDeviceSynchronize());


  //-------------------------------------------------------------------
  //  Initialise arrays to zero
  //-------------------------------------------------------------------
  for(int i=0; i<nspins; ++i) {
    for(int j=0; j<3; ++j) {
      sf(i,j) = 0.0;
    }
  }
  
  CUDA_CALL(hipMemcpy(w_dev,sf.ptr(),(size_t)(nspins3*sizeof(float)),hipMemcpyHostToDevice));
  CUDA_CALL(hipDeviceSynchronize());
  CUDA_CALL(hipMemcpy(h_dev,sf.ptr(),(size_t)(nspins3*sizeof(float)),hipMemcpyHostToDevice));
  CUDA_CALL(hipDeviceSynchronize());
  
  //-------------------------------------------------------------------
  //  Initialise hiprand
  //-------------------------------------------------------------------

  // hiprand generator
  //CURAND_CALL(hiprandCreateGenerator(&gen,HIPRAND_RNG_PSEUDO_DEFAULT));


  // TODO: set random seed from config
  const unsigned long long gpuseed = rng.uniform()*18446744073709551615ULL;
  //CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen, gpuseed));
  CUDA_CALL(hipDeviceSynchronize());
  //CUDA_CALL(cudaThreadSetLimit(hipLimitStackSize,1024));
  //CUDA_CALL(hipDeviceSynchronize());

  //-------------------------------------------------------------------
  //  Initialise cusparse
  //-------------------------------------------------------------------
  hipsparseStatus_t status;
  status = hipsparseCreate(&handle);
  if (status != HIPSPARSE_STATUS_SUCCESS) {
    jams_error("CUSPARSE Library initialization failed");
  }
  CUDA_CALL(hipDeviceSynchronize());

  // create matrix descriptor
  status = hipsparseCreateMatDescr(&descra);
  if (status != HIPSPARSE_STATUS_SUCCESS) {
    jams_error("CUSPARSE Matrix descriptor initialization failed");
  }
  CUDA_CALL(hipDeviceSynchronize());

  status = hipsparseSetMatType(descra,HIPSPARSE_MATRIX_TYPE_GENERAL);
  if (status != HIPSPARSE_STATUS_SUCCESS) {
    jams_error("CUSPARSE Matrix descriptor set type failed");
  }
  CUDA_CALL(hipDeviceSynchronize());
  status = hipsparseSetMatIndexBase(descra,HIPSPARSE_INDEX_BASE_ZERO);
  if (status != HIPSPARSE_STATUS_SUCCESS) {
    jams_error("CUSPARSE Matrix descriptor set index base failed");
  }
  CUDA_CALL(hipDeviceSynchronize());
  /*
  status = hipsparseSetMatFillMode(descra,HIPSPARSE_FILL_MODE_UPPER);
  if (status != HIPSPARSE_STATUS_SUCCESS) {
    jams_error("CUSPARSE Matrix descriptor set fill mode failed");
  }
  CUDA_CALL(hipDeviceSynchronize());
  status = hipsparseSetMatDiagType(descra,HIPSPARSE_DIAG_TYPE_NON_UNIT);
  if (status != HIPSPARSE_STATUS_SUCCESS) {
    jams_error("CUSPARSE Matrix descriptor set diag type failed");
  }
  CUDA_CALL(hipDeviceSynchronize());
  */
  nblocks = (nspins+BLOCKSIZE-1)/BLOCKSIZE;

  initialised = true;
}

void CUDASemiLLGSolver::run()
{
  using namespace globals;

  // copy s_dev to s_new_dev
  // NOTE: this is part of the SEMILLG scheme
  CUDA_CALL(hipDeviceSynchronize());
  CUDA_CALL(hipMemcpy(s_new_dev,s_dev,(size_t)(nspins3*sizeof(double)),hipMemcpyDeviceToDevice));

  // generate wiener trajectories
  float stmp = sqrt(temperature);
  
  if(temperature > 0.0) {
    if(nspins3%2 == 0) {
      CURAND_CALL(hiprandGenerateNormal(gen, w_dev, nspins3, 0.0f, stmp));
    } else {
      CURAND_CALL(hiprandGenerateNormal(gen, w_dev, (nspins3+1), 0.0f, stmp));
    }
  }
  CUDA_CALL(hipDeviceSynchronize());
  
  // calculate interaction fields (and zero field array)
  hipsparseStatus_t stat =
  hipsparseScsrmv(handle,HIPSPARSE_OPERATION_NON_TRANSPOSE,nspins3,nspins3,1.0,descra,
      Jij_dev_val,Jij_dev_row,Jij_dev_col,sf_dev,0.0,h_dev);
  if(stat != HIPSPARSE_STATUS_SUCCESS){
    jams_error("CUSPARSE FAILED\n");
}
  CUDA_CALL(hipDeviceSynchronize());

  // integrate
  cuda_semi_llg_kernelA<<<nblocks,BLOCKSIZE>>>
    (
      s_dev,
      sf_dev,
      h_dev,
      w_dev,
      mat_dev,
      h_app[0],
      h_app[1],
      h_app[2],
      nspins,
      dt
    );
  CUDA_CALL(hipDeviceSynchronize());

  // calculate interaction fields (and zero field array)
  hipsparseScsrmv(handle,HIPSPARSE_OPERATION_NON_TRANSPOSE,nspins3,nspins3,1.0,descra,
      Jij_dev_val,Jij_dev_row,Jij_dev_col,sf_dev,0.0,h_dev);
  CUDA_CALL(hipDeviceSynchronize());
  
  CUDA_CALL(hipDeviceSynchronize());
  cuda_semi_llg_kernelB<<<nblocks,BLOCKSIZE>>>
    (
      s_dev,
      sf_dev,
      s_new_dev,
      h_dev,
      w_dev,
      mat_dev,
      h_app[0],
      h_app[1],
      h_app[2],
      nspins,
      dt
    );
  CUDA_CALL(hipDeviceSynchronize());

  iteration++;
}

CUDASemiLLGSolver::~CUDASemiLLGSolver()
{
  CUDA_CALL(hipDeviceSynchronize());
  hiprandDestroyGenerator(gen);
  hipsparseStatus_t status;

  status = hipsparseDestroyMatDescr(descra);
  if (status != HIPSPARSE_STATUS_SUCCESS) {
    jams_error("CUSPARSE matrix destruction failed");
  }

  status = hipsparseDestroy(handle);
  if (status != HIPSPARSE_STATUS_SUCCESS) {
    jams_error("CUSPARSE Library destruction failed");
  }

  //-------------------------------------------------------------------
  //  Free device memory
  //-------------------------------------------------------------------

  // spin arrays
  CUDA_CALL(hipFree(s_dev));
  CUDA_CALL(hipFree(sf_dev));
  CUDA_CALL(hipFree(s_new_dev));

  // field arrays
  CUDA_CALL(hipFree(h_dev));

  // wiener processes
  CUDA_CALL(hipFree(w_dev));

  // jij matrix
  CUDA_CALL(hipFree(Jij_dev_row));
  CUDA_CALL(hipFree(Jij_dev_col));
  CUDA_CALL(hipFree(Jij_dev_val));

  // material arrays
  CUDA_CALL(hipFree(mat_dev));
}


#include "hip/hip_runtime.h"
#include "cuda_semillg_kernel.cu"
#include "globals.h"
#include "consts.h"

#include "cuda_semillg.h"

#include <hiprand.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsparse.h>

#include <cmath>


#ifndef NDEBUG
#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
  printf("Error at %s:%d\n",__FILE__,__LINE__);\
    exit(EXIT_FAILURE);}} while(0)
#else
#define CUDA_CALL(x) x
#endif

#ifndef NDEBUG
#define CURAND_CALL(x) do { if((x) != HIPRAND_STATUS_SUCCESS) { \
  printf("Error at %s:%d\n",__FILE__,__LINE__);\
  exit(EXIT_FAILURE);}} while(0)
#else
#define CURAND_CALL(x) x
#endif

#if defined(__HIPCC__) && defined(CUDA_NO_SM_13_DOUBLE_INTRINSICS)
    #error "-arch sm_13 nvcc flag is required to compile"
#endif

// block size for GPU, 64 appears to be most efficient for current kernel
#define BLOCKSIZE 32

void CUDASemiLLGSolver::initialise(int argc, char **argv, double idt)
{
  using namespace globals;

  // initialise base class
  Solver::initialise(argc,argv,idt);

  sigma.resize(nspins);

  for(int i=0; i<nspins; ++i) {
    sigma(i) = sqrt( (2.0*boltzmann_si*alpha(i)) / (dt*mus(i)) );
  }


  output.write("Initialising CUDA Semi Implicit LLG solver (CPU)\n");

  output.write("Initialising CUBLAS\n");

  output.write("Allocating device memory...\n");

  //-------------------------------------------------------------------
  //  Allocate device memory
  //-------------------------------------------------------------------

  // spin arrays
  CUDA_CALL(hipMalloc((void**)&s_dev,nspins3*sizeof(s_dev)));
  CUDA_CALL(hipMalloc((void**)&s_new_dev,nspins3*sizeof(s_new_dev)));

  // field arrays
  CUDA_CALL(hipMalloc((void**)&h_dev,nspins3*sizeof(h_dev)));

  // wiener processes
  CUDA_CALL(hipMalloc((void**)&w_dev,nspins3*sizeof(w_dev)));

  // jij matrix
  CUDA_CALL(hipMalloc((void**)&Jij_dev_row,(Jij.rows()+1)*sizeof(Jij_dev_row)));
  CUDA_CALL(hipMalloc((void**)&Jij_dev_col,Jij.nonzero()*sizeof(Jij_dev_col)));
  CUDA_CALL(hipMalloc((void**)&Jij_dev_val,Jij.nonzero()*sizeof(Jij_dev_val)));

  // material properties
  CUDA_CALL(hipMalloc((void**)&mat_dev,4*nspins*sizeof(mat_dev)));
  //CUDA_CALL(hipMalloc((void**)&mus_dev,nspins*sizeof(mus_dev)));
  //CUDA_CALL(hipMalloc((void**)&gyro_dev,nspins*sizeof(gyro_dev)));
  //CUDA_CALL(hipMalloc((void**)&alpha_dev,nspins*sizeof(alpha_dev)));
  //CUDA_CALL(hipMalloc((void**)&sigma_dev,nspins*sizeof(sigma_dev)));

  //-------------------------------------------------------------------
  //  Copy data to device
  //-------------------------------------------------------------------

  output.write("Copying data to device memory...\n");
  // initial spins
  CUDA_CALL(hipMemcpy(s_dev,s.ptr(),(size_t)(nspins3*sizeof(s_dev)),hipMemcpyHostToDevice));

  // jij matrix
  CUDA_CALL(hipMemcpy(Jij_dev_row,Jij.ptrRow(),
        (size_t)((Jij.rows()+1)*(sizeof(Jij_dev_row[0]))),hipMemcpyHostToDevice));

  CUDA_CALL(hipMemcpy(Jij_dev_col,Jij.ptrCol(),
        (size_t)((Jij.nonzero())*(sizeof(Jij_dev_col[0]))),hipMemcpyHostToDevice));

  CUDA_CALL(hipMemcpy(Jij_dev_val,Jij.ptrVal(),
        (size_t)((Jij.nonzero())*(sizeof(Jij_dev_val[0]))),hipMemcpyHostToDevice));

  Array2D<double> mat(nspins,4);
  // material properties
  for(int i=0; i<nspins; ++i){
    mat(i,0) = mus(i);
    mat(i,1) = gyro(i);
    mat(i,2) = alpha(i);
    mat(i,3) = sigma(i);
  }
  CUDA_CALL(hipMemcpy(mat_dev,mat.ptr(),(size_t)(4*nspins*sizeof(mat_dev)),hipMemcpyHostToDevice));
  //CUDA_CALL(hipMemcpy(mus_dev,mus.ptr(),(size_t)(nspins*sizeof(mus_dev)),hipMemcpyHostToDevice));
  //CUDA_CALL(hipMemcpy(gyro_dev,gyro.ptr(),(size_t)(nspins*sizeof(gyro_dev)),hipMemcpyHostToDevice));
  //CUDA_CALL(hipMemcpy(alpha_dev,alpha.ptr(),(size_t)(nspins*sizeof(alpha_dev)),hipMemcpyHostToDevice));
  //CUDA_CALL(hipMemcpy(sigma_dev,sigma.ptr(),(size_t)(nspins*sizeof(sigma_dev)),hipMemcpyHostToDevice));

  //-------------------------------------------------------------------
  //  Initialise hiprand
  //-------------------------------------------------------------------

  // hiprand generator
  CURAND_CALL(hiprandCreateGenerator(&gen,HIPRAND_RNG_PSEUDO_DEFAULT));
  CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL));

  //-------------------------------------------------------------------
  //  Initialise cusparse
  //-------------------------------------------------------------------
  hipsparseStatus_t status;
  status = hipsparseCreate(&handle);
  if (status != HIPSPARSE_STATUS_SUCCESS) {
    jams_error("CUSPARSE Library initialization failed");
  }

  // create matrix descriptor
  status = hipsparseCreateMatDescr(&descra);
  if (status != HIPSPARSE_STATUS_SUCCESS) {
    jams_error("CUSPARSE Matrix descriptor initialization failed");
  }
  hipsparseSetMatType(descra,HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatIndexBase(descra,HIPSPARSE_INDEX_BASE_ZERO);

  nblocks = (nspins+BLOCKSIZE-1)/BLOCKSIZE;

  initialised = true;
}

void CUDASemiLLGSolver::run()
{
  using namespace globals;

  // copy s_dev to s_new_dev
  // NOTE: this is part of the SEMILLG scheme
  CUDA_CALL(hipMemcpy(s_new_dev,s_dev,(size_t)(nspins3*sizeof(s_dev)),hipMemcpyDeviceToDevice));

  // generate wiener trajectories
  float stmp = sqrt(temperature);
  CURAND_CALL(hiprandGenerateNormal(gen, w_dev, nspins3, 0.0f, stmp));

  // calculate interaction fields (and zero field array)
  hipsparseDcsrmv(handle,HIPSPARSE_OPERATION_NON_TRANSPOSE,nspins3,nspins3,1.0,descra,
      Jij_dev_val,Jij_dev_row,Jij_dev_col,s_dev,0.0,h_dev);

  // integrate
  cuda_semi_llg_kernelA<<<nblocks,BLOCKSIZE>>>
    (
      s_dev,
      h_dev,
      w_dev,
      mat_dev,
//      gyro_dev,
//      alpha_dev,
      Jij_dev_row,
      Jij_dev_col,
      Jij_dev_val,
      h_app[0],
      h_app[1],
      h_app[2],
      nspins,
      dt
    );

  // calculate interaction fields (and zero field array)
  hipsparseDcsrmv(handle,HIPSPARSE_OPERATION_NON_TRANSPOSE,nspins3,nspins3,1.0,descra,
      Jij_dev_val,Jij_dev_row,Jij_dev_col,s_dev,0.0,h_dev);

  cuda_semi_llg_kernelB<<<nblocks,BLOCKSIZE>>>
    (
      s_dev,
      s_new_dev,
      h_dev,
      w_dev,
      mat_dev,
//      gyro_dev,
//      alpha_dev,
      Jij_dev_row,
      Jij_dev_col,
      Jij_dev_val,
      h_app[0],
      h_app[1],
      h_app[2],
      nspins,
      dt
    );

  if(iteration%1000 == 0){
  CUDA_CALL(hipMemcpy(s.ptr(),s_dev,(size_t)(nspins3*sizeof(s_dev)),hipMemcpyDeviceToHost));
  }

//  for(int i=0; i<nspins; ++i) {
//    std::cout<<i<<"\t"<<h(i,0)<<"\t"<<h(i,1)<<"\t"<<h(i,2)<<std::endl;
//  }

  iteration++;
}

CUDASemiLLGSolver::~CUDASemiLLGSolver()
{
  hiprandDestroyGenerator(gen);

  //-------------------------------------------------------------------
  //  Free device memory
  //-------------------------------------------------------------------

  // spin arrays
  CUDA_CALL(hipFree(s_dev));
  CUDA_CALL(hipFree(s_new_dev));

  // field arrays
  CUDA_CALL(hipFree(h_dev));

  // wiener processes
  CUDA_CALL(hipFree(w_dev));

  // jij matrix
  CUDA_CALL(hipFree(Jij_dev_row));
  CUDA_CALL(hipFree(Jij_dev_col));
  CUDA_CALL(hipFree(Jij_dev_val));

  // material arrays
  //CUDA_CALL(hipFree(mus_dev));
  //CUDA_CALL(hipFree(gyro_dev));
  //CUDA_CALL(hipFree(alpha_dev));
  CUDA_CALL(hipFree(mat_dev));
}


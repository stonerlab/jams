#include "hip/hip_runtime.h"
#include "cuda_semillg_kernel.cu"
#include "globals.h"
#include "consts.h"

#include "cuda_semillg.h"

#include <hiprand.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsparse.h>

#include <cmath>


#ifndef NDEBUG
#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
  printf("Error at %s:%d\n",__FILE__,__LINE__);\
    exit(EXIT_FAILURE);}} while(0)
#else
#define CUDA_CALL(x) x
#endif

#ifndef NDEBUG
#define CURAND_CALL(x) do { if((x) != HIPRAND_STATUS_SUCCESS) { \
  printf("Error at %s:%d\n",__FILE__,__LINE__);\
  exit(EXIT_FAILURE);}} while(0)
#else
#define CURAND_CALL(x) x
#endif

#if defined(__HIPCC__) && defined(CUDA_NO_SM_13_DOUBLE_INTRINSICS)
    #error "-arch sm_13 nvcc flag is required to compile"
#endif

// block size for GPU, 64 appears to be most efficient for current kernel
#define BLOCKSIZE 128

void CUDASemiLLGSolver::syncOutput()
{
  using namespace globals;
  CUDA_CALL(hipMemcpy(s.ptr(),s_dev,(size_t)(nspins3*sizeof(double)),hipMemcpyDeviceToHost));
}

void CUDASemiLLGSolver::initialise(int argc, char **argv, double idt)
{
  using namespace globals;

  // initialise base class
  Solver::initialise(argc,argv,idt);

  sigma.resize(nspins);

  for(int i=0; i<nspins; ++i) {
    sigma(i) = sqrt( (2.0*boltzmann_si*alpha(i)) / (dt*mus(i)*mu_bohr_si) );
  }


  output.write("Initialising CUDA Semi Implicit LLG solver (CPU)\n");

  output.write("Initialising CUBLAS\n");

  output.write("Allocating device memory...\n");

  //-------------------------------------------------------------------
  //  Allocate device memory
  //-------------------------------------------------------------------

  // spin arrays
  CUDA_CALL(hipMalloc((void**)&s_dev,nspins3*sizeof(double)));
  CUDA_CALL(hipMalloc((void**)&sf_dev,nspins3*sizeof(float)));
  CUDA_CALL(hipMalloc((void**)&s_new_dev,nspins3*sizeof(double)));

  // field arrays
  CUDA_CALL(hipMalloc((void**)&h_dev,nspins3*sizeof(float)));

  if(nspins3%2 == 0) {
    // wiener processes
    CUDA_CALL(hipMalloc((void**)&w_dev,nspins3*sizeof(float)));
  } else {
    CUDA_CALL(hipMalloc((void**)&w_dev,(nspins3+1)*sizeof(float)));
  }


  // jij matrix
  CUDA_CALL(hipMalloc((void**)&Jij_dev_row,(Jij.rows()+1)*sizeof(int)));
  CUDA_CALL(hipMalloc((void**)&Jij_dev_col,Jij.nonzero()*sizeof(int)));
  CUDA_CALL(hipMalloc((void**)&Jij_dev_val,Jij.nonzero()*sizeof(float)));

  // material properties
  CUDA_CALL(hipMalloc((void**)&mat_dev,nspins*4*sizeof(float)));

  //-------------------------------------------------------------------
  //  Copy data to device
  //-------------------------------------------------------------------

  output.write("Copying data to device memory...\n");
  // initial spins
  Array2D<float> sf(nspins,3);
  for(int i=0; i<nspins; ++i) {
    for(int j=0; j<3; ++j) {
      sf(i,j) = static_cast<float>(s(i,j));
    }
  }
  CUDA_CALL(hipMemcpy(s_dev,s.ptr(),(size_t)(nspins3*sizeof(double)),hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(sf_dev,sf.ptr(),(size_t)(nspins3*sizeof(float)),hipMemcpyHostToDevice));

  // jij matrix
  CUDA_CALL(hipMemcpy(Jij_dev_row,Jij.ptrRow(),
        (size_t)((Jij.rows()+1)*(sizeof(int))),hipMemcpyHostToDevice));

  CUDA_CALL(hipMemcpy(Jij_dev_col,Jij.ptrCol(),
        (size_t)((Jij.nonzero())*(sizeof(int))),hipMemcpyHostToDevice));

  CUDA_CALL(hipMemcpy(Jij_dev_val,Jij.ptrVal(),
        (size_t)((Jij.nonzero())*(sizeof(float))),hipMemcpyHostToDevice));

  Array2D<float> mat(nspins,4);
  // material properties
  for(int i=0; i<nspins; ++i){
    mat(i,0) = mus(i);
    mat(i,1) = gyro(i);
    mat(i,2) = alpha(i);
    mat(i,3) = sigma(i);
  }
  CUDA_CALL(hipMemcpy(mat_dev,mat.ptr(),(size_t)(nspins*4*sizeof(float)),hipMemcpyHostToDevice));


  //-------------------------------------------------------------------
  //  Initialise arrays to zero
  //-------------------------------------------------------------------
  for(int i=0; i<nspins; ++i) {
    for(int j=0; j<3; ++j) {
      sf(i,j) = 0.0;
    }
  }
  
  CUDA_CALL(hipMemcpy(w_dev,sf.ptr(),(size_t)(nspins3*sizeof(float)),hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(h_dev,sf.ptr(),(size_t)(nspins3*sizeof(float)),hipMemcpyHostToDevice));

  //-------------------------------------------------------------------
  //  Initialise hiprand
  //-------------------------------------------------------------------

  // hiprand generator
  CURAND_CALL(hiprandCreateGenerator(&gen,HIPRAND_RNG_PSEUDO_DEFAULT));
  CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL));

  //-------------------------------------------------------------------
  //  Initialise cusparse
  //-------------------------------------------------------------------
  hipsparseStatus_t status;
  status = hipsparseCreate(&handle);
  if (status != HIPSPARSE_STATUS_SUCCESS) {
    jams_error("CUSPARSE Library initialization failed");
  }

  // create matrix descriptor
  status = hipsparseCreateMatDescr(&descra);
  if (status != HIPSPARSE_STATUS_SUCCESS) {
    jams_error("CUSPARSE Matrix descriptor initialization failed");
  }
  hipsparseSetMatType(descra,HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatIndexBase(descra,HIPSPARSE_INDEX_BASE_ZERO);

  nblocks = (nspins+BLOCKSIZE-1)/BLOCKSIZE;

  initialised = true;
}

void CUDASemiLLGSolver::run()
{
  using namespace globals;

  // copy s_dev to s_new_dev
  // NOTE: this is part of the SEMILLG scheme
  CUDA_CALL(hipMemcpy(s_new_dev,s_dev,(size_t)(nspins3*sizeof(double)),hipMemcpyDeviceToDevice));

  // generate wiener trajectories
  float stmp = sqrt(temperature);
  
  if(temperature > 0.0) {
    if(nspins3%2 == 0) {
      CURAND_CALL(hiprandGenerateNormal(gen, w_dev, nspins3, 0.0f, stmp));
    } else {
      CURAND_CALL(hiprandGenerateNormal(gen, w_dev, (nspins3+1), 0.0f, stmp));
    }

  }
  
  // calculate interaction fields (and zero field array)
  hipsparseStatus_t stat =
  hipsparseScsrmv(handle,HIPSPARSE_OPERATION_NON_TRANSPOSE,nspins3,nspins3,1.0,descra,
      Jij_dev_val,Jij_dev_row,Jij_dev_col,sf_dev,0.0,h_dev);
  if(stat != HIPSPARSE_STATUS_SUCCESS){
    jams_error("CUSPARSE FAILED\n");
}
  
//  Array2D<float> tmp(nspins,3);
//  CUDA_CALL(hipMemcpy(tmp.ptr(),h_dev,(size_t)(nspins3*sizeof(float)),hipMemcpyDeviceToHost));
//
//  for(int i=0;i<nspins;++i) {
//    std::cout<< tmp(i,0) << "\t" << tmp(i,1) << "\t" << tmp(i,2) << std::endl;
//  }

  // integrate
  cuda_semi_llg_kernelA<<<nblocks,BLOCKSIZE>>>
    (
      s_dev,
      sf_dev,
      h_dev,
      w_dev,
      mat_dev,
      h_app[0],
      h_app[1],
      h_app[2],
      nspins,
      dt
    );

  // calculate interaction fields (and zero field array)
  hipsparseScsrmv(handle,HIPSPARSE_OPERATION_NON_TRANSPOSE,nspins3,nspins3,1.0,descra,
      Jij_dev_val,Jij_dev_row,Jij_dev_col,sf_dev,0.0,h_dev);
  
  cuda_semi_llg_kernelB<<<nblocks,BLOCKSIZE>>>
    (
      s_dev,
      sf_dev,
      s_new_dev,
      h_dev,
      w_dev,
      mat_dev,
      h_app[0],
      h_app[1],
      h_app[2],
      nspins,
      dt
    );

  iteration++;
}

CUDASemiLLGSolver::~CUDASemiLLGSolver()
{
  hiprandDestroyGenerator(gen);

  //-------------------------------------------------------------------
  //  Free device memory
  //-------------------------------------------------------------------

  // spin arrays
  CUDA_CALL(hipFree(s_dev));
  CUDA_CALL(hipFree(s_new_dev));

  // field arrays
  CUDA_CALL(hipFree(h_dev));

  // wiener processes
  CUDA_CALL(hipFree(w_dev));

  // jij matrix
  CUDA_CALL(hipFree(Jij_dev_row));
  CUDA_CALL(hipFree(Jij_dev_col));
  CUDA_CALL(hipFree(Jij_dev_val));

  // material arrays
  CUDA_CALL(hipFree(mat_dev));
}


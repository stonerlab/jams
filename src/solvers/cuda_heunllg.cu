#include "hip/hip_runtime.h"
#include "cuda_sparse.h"
#include "cuda_sparse_types.h"
#include "cuda_heunllg_kernel.h"
#include "globals.h"
#include "consts.h"

#include "cuda_heunllg.h"

#include <hiprand.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsparse.h>

#include <cmath>


void CUDAHeunLLGSolver::syncOutput()
{
  using namespace globals;
  CUDA_CALL(hipMemcpy(s.ptr(),s_dev,(size_t)(nspins3*sizeof(double)),hipMemcpyDeviceToHost));
}

void CUDAHeunLLGSolver::initialise(int argc, char **argv, double idt)
{
  using namespace globals;

  // initialise base class
  Solver::initialise(argc,argv,idt);

  sigma.resize(nspins);

  for(int i=0; i<nspins; ++i) {
    sigma(i) = sqrt( (2.0*boltzmann_si*alpha(i)) / (dt*mus(i)*mu_bohr_si) );
  }


  output.write("  * CUDA Heun LLG solver (GPU)\n");

  //-------------------------------------------------------------------
  //  Initialise hiprand
  //-------------------------------------------------------------------

  output.write("  * Initialising CURAND...\n");
  // hiprand generator
  CURAND_CALL(hiprandCreateGenerator(&gen,HIPRAND_RNG_PSEUDO_DEFAULT));


  // TODO: set random seed from config
  const unsigned long long gpuseed = rng.uniform()*18446744073709551615ULL;
  CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen, gpuseed));
  CURAND_CALL(hiprandGenerateSeeds(gen));
  /*CUDA_CALL(cudaThreadSetLimit(hipLimitStackSize,1024));*/
  /*CUDA_CALL(hipDeviceSynchronize());*/


  //-------------------------------------------------------------------
  //  Allocate device memory
  //-------------------------------------------------------------------

  output.write("  * Converting MAP to DIA\n");
  J1ij_s.convertMAP2DIA();
  J1ij_t.convertMAP2DIA();
  J2ij_s.convertMAP2DIA();
  J2ij_t.convertMAP2DIA();
  output.write("  * J1ij scalar matrix memory (DIA): %f MB\n",J1ij_s.calculateMemory());
  output.write("  * J1ij tensor matrix memory (DIA): %f MB\n",J1ij_t.calculateMemory());
  output.write("  * J2ij scalar matrix memory (DIA): %f MB\n",J2ij_s.calculateMemory());
  output.write("  * J2ij tensor matrix memory (DIA): %f MB\n",J2ij_t.calculateMemory());
  
  output.write("  * Converting J4 MAP to CSR\n");
  J4ijkl_s.convertMAP2CSR();
  output.write("  * J4ijkl scalar matrix memory (CSR): %f MB\n",J4ijkl_s.calculateMemory());


  output.write("  * Allocating device memory...\n");
  // spin arrays
  CUDA_CALL(hipMalloc((void**)&s_dev,nspins3*sizeof(double)));
  CUDA_CALL(hipMalloc((void**)&sf_dev,nspins3*sizeof(float)));
  CUDA_CALL(hipMalloc((void**)&s_new_dev,nspins3*sizeof(double)));

  // field arrays
  CUDA_CALL(hipMalloc((void**)&h_dev,nspins3*sizeof(float)));
  CUDA_CALL(hipMalloc((void**)&e_dev,nspins3*sizeof(float)));

  // position arrays
  CUDA_CALL(hipMalloc((void**)&r_dev,nspins3*sizeof(float)));

  if(nspins3%2 == 0) {
    // wiener processes
    CUDA_CALL(hipMalloc((void**)&w_dev,nspins3*sizeof(float)));
  } else {
    CUDA_CALL(hipMalloc((void**)&w_dev,(nspins3+1)*sizeof(float)));
  }


  // bilinear scalar
  allocate_transfer_dia(J1ij_s, J1ij_s_dev);
  
  // bilinear tensor
  allocate_transfer_dia(J1ij_t, J1ij_t_dev);
  
  // biquadratic scalar
  allocate_transfer_dia(J2ij_s, J2ij_s_dev);
  
  // bilinear tensor
  allocate_transfer_dia(J2ij_t, J2ij_t_dev);

  allocate_transfer_csr_4d(J4ijkl_s, J4ijkl_s_dev);

  // material properties
  CUDA_CALL(hipMalloc((void**)&mat_dev,nspins*4*sizeof(float)));

  //-------------------------------------------------------------------
  //  Copy data to device
  //-------------------------------------------------------------------

  output.write("  * Copying data to device memory...\n");
  // initial spins
  Array2D<float> sf(nspins,3);
  for(int i=0; i<nspins; ++i) {
    for(int j=0; j<3; ++j) {
      sf(i,j) = static_cast<float>(s(i,j));
    }
  }
  CUDA_CALL(hipMemcpy(s_dev,s.ptr(),(size_t)(nspins3*sizeof(double)),hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(sf_dev,sf.ptr(),(size_t)(nspins3*sizeof(float)),hipMemcpyHostToDevice));

  // position array
  CUDA_CALL(hipMemcpy(r_dev,r_pos.ptr(),(size_t)(nspins3*sizeof(float)),hipMemcpyHostToDevice));

  Array2D<float> mat(nspins,4);
  // material properties
  for(int i=0; i<nspins; ++i){
    mat(i,0) = mus(i);
    mat(i,1) = gyro(i);
    mat(i,2) = alpha(i);
    mat(i,3) = sigma(i);
  }
  CUDA_CALL(hipMemcpy(mat_dev,mat.ptr(),(size_t)(nspins*4*sizeof(float)),hipMemcpyHostToDevice));

  eng.resize(nspins,3);


  //-------------------------------------------------------------------
  //  Initialise arrays to zero
  //-------------------------------------------------------------------
  for(int i=0; i<nspins; ++i) {
    for(int j=0; j<3; ++j) {
      sf(i,j) = 0.0;
    }
  }
  
  CUDA_CALL(hipMemcpy(w_dev,sf.ptr(),(size_t)(nspins3*sizeof(float)),hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(h_dev,sf.ptr(),(size_t)(nspins3*sizeof(float)),hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(e_dev,sf.ptr(),(size_t)(nspins3*sizeof(float)),hipMemcpyHostToDevice));

  nblocks = (nspins+BLOCKSIZE-1)/BLOCKSIZE;

  J1ij_s_dev.blocks = std::min<int>(DIA_BLOCK_SIZE,(nspins+DIA_BLOCK_SIZE-1)/DIA_BLOCK_SIZE);
  J1ij_t_dev.blocks = std::min<int>(DIA_BLOCK_SIZE,(nspins3+DIA_BLOCK_SIZE-1)/DIA_BLOCK_SIZE);

  J2ij_s_dev.blocks = std::min<int>(DIA_BLOCK_SIZE,(nspins+DIA_BLOCK_SIZE-1)/DIA_BLOCK_SIZE);
  J2ij_t_dev.blocks = std::min<int>(DIA_BLOCK_SIZE,(nspins3+DIA_BLOCK_SIZE-1)/DIA_BLOCK_SIZE);
  
  J4ijkl_s_dev.blocks = std::min<int>(CSR_4D_BLOCK_SIZE,(nspins+CSR_4D_BLOCK_SIZE-1)/CSR_4D_BLOCK_SIZE);

  initialised = true;
}

void CUDAHeunLLGSolver::run()
{
  using namespace globals;

  // generate wiener trajectories
  float stmp = sqrt(temperature);
  
  if(temperature > 0.0) {
    if(nspins3%2 == 0) {
      CURAND_CALL(hiprandGenerateNormal(gen, w_dev, nspins3, 0.0f, stmp));
    } else {
      CURAND_CALL(hiprandGenerateNormal(gen, w_dev, (nspins3+1), 0.0f, stmp));
    }
  }
  
  // calculate interaction fields (and zero field array)

  //CUDA_CALL(hipBindTexture(0,tex_x_float,sf_dev));
  
  float beta=0;
  // bilinear scalar
  if(J1ij_s.nonZero() > 0){
    bilinear_scalar_dia_kernel<<< J1ij_s_dev.blocks, DIA_BLOCK_SIZE >>>(nspins,nspins,
      J1ij_s.diags(),J1ij_s_dev.pitch,1.0,beta,J1ij_s_dev.row,J1ij_s_dev.val,sf_dev,h_dev);
    beta = 1.0;
  }

  // bilinear tensor
  if(J1ij_t.nonZero() > 0){
    spmv_dia_kernel<<< J1ij_t_dev.blocks, DIA_BLOCK_SIZE >>>(nspins3,nspins3,
      J1ij_t.diags(),J1ij_t_dev.pitch,beta,1.0,J1ij_t_dev.row,J1ij_t_dev.val,sf_dev,h_dev);
    beta = 1.0;
  }
  
  // biquadratic scalar
  if(J2ij_s.nonZero() > 0){
    biquadratic_scalar_dia_kernel<<< J2ij_s_dev.blocks, DIA_BLOCK_SIZE >>>(nspins,nspins,
      J2ij_s.diags(),J2ij_s_dev.pitch,2.0,beta,J2ij_s_dev.row,J2ij_s_dev.val,sf_dev,h_dev);
    beta = 1.0;
  }
  
  // biquadratic tensor
  if(J2ij_t.nonZero() > 0){
    spmv_dia_kernel<<< J2ij_t_dev.blocks, DIA_BLOCK_SIZE >>>(nspins3,nspins3,
      J2ij_t.diags(),J2ij_t_dev.pitch,2.0,beta,J2ij_t_dev.row,J2ij_t_dev.val,sf_dev,h_dev);
    beta = 1.0;
  }
  
  if(J4ijkl_s.nonZero() > 0){
    fourspin_scalar_csr_kernel<<< J4ijkl_s_dev.blocks,CSR_4D_BLOCK_SIZE>>>(nspins,nspins,1.0,beta,
        J4ijkl_s_dev.pointers,J4ijkl_s_dev.coords,J4ijkl_s_dev.val,sf_dev,h_dev);
    beta = 1.0;
  }
  
  //CUDA_CALL(hipUnbindTexture(tex_x_float));
  
  // integrate
  cuda_heun_llg_kernelA<<<nblocks,BLOCKSIZE>>>
    (
      s_dev,
      sf_dev,
      s_new_dev,
      h_dev,
      w_dev,
      mat_dev,
      h_app[0],
      h_app[1],
      h_app[2],
      nspins,
      dt
    );

  // calculate interaction fields (and zero field array)

  beta=0.0;
  // bilinear scalar
  if(J1ij_s.nonZero() > 0){
    bilinear_scalar_dia_kernel<<< J1ij_s_dev.blocks, DIA_BLOCK_SIZE >>>(nspins,nspins,
      J1ij_s.diags(),J1ij_s_dev.pitch,1.0,beta,J1ij_s_dev.row,J1ij_s_dev.val,sf_dev,h_dev);
    beta = 1.0;
  }

  // bilinear tensor
  if(J1ij_t.nonZero() > 0){
    spmv_dia_kernel<<< J1ij_t_dev.blocks, DIA_BLOCK_SIZE >>>(nspins3,nspins3,
      J1ij_t.diags(),J1ij_t_dev.pitch,1.0,beta,J1ij_t_dev.row,J1ij_t_dev.val,sf_dev,h_dev);
    beta = 1.0;
  }
  
  // biquadratic scalar
  if(J2ij_s.nonZero() > 0){
    biquadratic_scalar_dia_kernel<<< J2ij_s_dev.blocks, DIA_BLOCK_SIZE >>>(nspins,nspins,
      J2ij_s.diags(),J2ij_s_dev.pitch,2.0,beta,J2ij_s_dev.row,J2ij_s_dev.val,sf_dev,h_dev);
    beta = 1.0;
  }
  
  // biquadratic tensor
  if(J2ij_t.nonZero() > 0){
    spmv_dia_kernel<<< J2ij_t_dev.blocks, DIA_BLOCK_SIZE >>>(nspins3,nspins3,
      J2ij_t.diags(),J2ij_t_dev.pitch,2.0,beta,J2ij_t_dev.row,J2ij_t_dev.val,sf_dev,h_dev);
    beta = 1.0;
  }
  
  if(J4ijkl_s.nonZero() > 0){
    fourspin_scalar_csr_kernel<<< J4ijkl_s_dev.blocks,CSR_4D_BLOCK_SIZE>>>(nspins,nspins,1.0,beta,
        J4ijkl_s_dev.pointers,J4ijkl_s_dev.coords,J4ijkl_s_dev.val,sf_dev,h_dev);
    beta = 1.0;
  }

  /*Array2D<float> hf(nspins,3);*/
  /*Array2D<float> sf(nspins,3);*/
  /*CUDA_CALL(hipMemcpy(hf.ptr(),h_dev,(size_t)(nspins3*sizeof(float)),hipMemcpyDeviceToHost));*/
  /*CUDA_CALL(hipMemcpy(sf.ptr(),sf_dev,(size_t)(nspins3*sizeof(float)),hipMemcpyDeviceToHost));*/

  /*for(int i=0; i<nspins; ++i){*/
      /*std::cout<<i<<sf(i,0)<<"\t"<<sf(i,1)<<"\t"<<sf(i,2)<<"\t"<<hf(i,0)<<"\t"<<hf(i,1)<<"\t"<<hf(i,2)<<std::endl;*/
  /*}*/
  
  //CUDA_CALL(hipUnbindTexture(tex_x_float));
  
  cuda_heun_llg_kernelB<<<nblocks,BLOCKSIZE>>>
    (
      s_dev,
      sf_dev,
      s_new_dev,
      h_dev,
      w_dev,
      mat_dev,
      h_app[0],
      h_app[1],
      h_app[2],
      nspins,
      dt
    );
  iteration++;
}

void CUDAHeunLLGSolver::calcEnergy(double &e1_s, double &e1_t, double &e2_s, double &e2_t, double &e4_s){
  using namespace globals;
  const float beta=0.0;

  e1_s = 0.0; e1_t = 0.0; e2_s = 0.0; e2_t = 0.0;
  
  //size_t offset = size_t(-1);
  //CUDA_CALL(hipBindTexture(&offset,tex_x_float,sf_dev));

  // bilinear scalar
  if(J1ij_s.nonZero() > 0){
    bilinear_scalar_dia_kernel<<< J1ij_s_dev.blocks, DIA_BLOCK_SIZE >>>(nspins,nspins,
      J1ij_s.diags(),J1ij_s_dev.pitch,1.0,beta,J1ij_s_dev.row,J1ij_s_dev.val,sf_dev,e_dev);
    CUDA_CALL(hipMemcpy(eng.ptr(),e_dev,(size_t)(nspins3*sizeof(float)),hipMemcpyDeviceToHost));
    for(int i=0; i<nspins; ++i){
      e1_s = e1_s + (s(i,0)*eng(i,0)+s(i,1)*eng(i,1)+s(i,2)*eng(i,2));
    }
    e1_s = e1_s/nspins;
  }


  // bilinear tensor
  if(J1ij_t.nonZero() > 0){
    spmv_dia_kernel<<< J1ij_t_dev.blocks, DIA_BLOCK_SIZE >>>(nspins3,nspins3,
      J1ij_t.diags(),J1ij_t_dev.pitch,1.0,beta,J1ij_t_dev.row,J1ij_t_dev.val,sf_dev,e_dev);
    CUDA_CALL(hipMemcpy(eng.ptr(),e_dev,(size_t)(nspins3*sizeof(float)),hipMemcpyDeviceToHost));
    for(int i=0; i<nspins; ++i){
      e1_t = e1_t + (s(i,0)*eng(i,0)+s(i,1)*eng(i,1)+s(i,2)*eng(i,2));
    }
    e1_t = e1_t/nspins;
  }

  
  // biquadratic scalar
  if(J2ij_s.nonZero() > 0){
    biquadratic_scalar_dia_kernel<<< J2ij_s_dev.blocks, DIA_BLOCK_SIZE >>>(nspins,nspins,
      J2ij_s.diags(),J2ij_s_dev.pitch,1.0,beta,J2ij_s_dev.row,J2ij_s_dev.val,sf_dev,e_dev);
    CUDA_CALL(hipMemcpy(eng.ptr(),e_dev,(size_t)(nspins3*sizeof(float)),hipMemcpyDeviceToHost));
    for(int i=0; i<nspins; ++i){
      e2_s = e2_s + (s(i,0)*eng(i,0)+s(i,1)*eng(i,1)+s(i,2)*eng(i,2));
    }
    
    e2_s = e2_s/nspins;
  }

  // biquadratic tensor
  if(J2ij_t.nonZero() > 0){
    spmv_dia_kernel<<< J2ij_t_dev.blocks, DIA_BLOCK_SIZE >>>(nspins3,nspins3,
      J2ij_t.diags(),J2ij_t_dev.pitch,1.0,beta,J2ij_t_dev.row,J2ij_t_dev.val,sf_dev,e_dev);
    CUDA_CALL(hipMemcpy(eng.ptr(),e_dev,(size_t)(nspins3*sizeof(float)),hipMemcpyDeviceToHost));

    for(int i=0; i<nspins; ++i){
      e2_t = e2_t + (s(i,0)*eng(i,0)+s(i,1)*eng(i,1)+s(i,2)*eng(i,2));
    }
    
    e2_t = e2_t/nspins;
  }
  
  if(J4ijkl_s.nonZero() > 0){
    fourspin_scalar_csr_kernel<<< J4ijkl_s_dev.blocks,CSR_4D_BLOCK_SIZE>>>(nspins,nspins,1.0,beta,
        J4ijkl_s_dev.pointers,J4ijkl_s_dev.coords,J4ijkl_s_dev.val,sf_dev,e_dev);
    CUDA_CALL(hipMemcpy(eng.ptr(),e_dev,(size_t)(nspins3*sizeof(float)),hipMemcpyDeviceToHost));
    for(int i=0; i<nspins; ++i){
      e4_s = e4_s + (s(i,0)*eng(i,0)+s(i,1)*eng(i,1)+s(i,2)*eng(i,2));
    }
    
    e4_s = e4_s/nspins;
  }
  
  
  //CUDA_CALL(hipUnbindTexture(tex_x_float));
}

CUDAHeunLLGSolver::~CUDAHeunLLGSolver()
{
  hiprandDestroyGenerator(gen);
  
  hipsparseStatus_t status;

  status = hipsparseDestroyMatDescr(descra);
  if (status != HIPSPARSE_STATUS_SUCCESS) {
    jams_error("CUSPARSE matrix destruction failed");
  }
  CUDA_CALL(hipDeviceSynchronize());

  status = hipsparseDestroy(handle);
  if (status != HIPSPARSE_STATUS_SUCCESS) {
    jams_error("CUSPARSE Library destruction failed");
  }
  CUDA_CALL(hipDeviceSynchronize());

  //-------------------------------------------------------------------
  //  Free device memory
  //-------------------------------------------------------------------

  free_dia(J1ij_s_dev);
  free_dia(J1ij_t_dev);
  free_dia(J2ij_s_dev);
  free_dia(J2ij_t_dev);
  free_csr_4d(J4ijkl_s_dev);

  // spin arrays
  CUDA_CALL(hipFree(s_dev));
  CUDA_CALL(hipFree(sf_dev));
  CUDA_CALL(hipFree(s_new_dev));

  // field arrays
  CUDA_CALL(hipFree(r_dev));
  CUDA_CALL(hipFree(h_dev));
  CUDA_CALL(hipFree(e_dev));

  // wiener processes
  CUDA_CALL(hipFree(w_dev));


  // material arrays
  CUDA_CALL(hipFree(mat_dev));
}


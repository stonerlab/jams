#include "hip/hip_runtime.h"

#include "cuda_spmv.h"
#include "cuda_heunllg_kernel.cu"
#include "globals.h"
#include "consts.h"

#include "cuda_heunllg.h"

#include <hiprand.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsparse.h>

#include <cmath>

// block size for GPU, 64 appears to be most efficient for current kernel
#define BLOCKSIZE 32

#ifndef NDEBUG
#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
  printf("Error at %s:%d\n",__FILE__,__LINE__);\
    exit(EXIT_FAILURE);}} while(0)
#else
#define CUDA_CALL(x) x
#endif

#ifndef NDEBUG
#define CURAND_CALL(x) do { if((x) != HIPRAND_STATUS_SUCCESS) { \
  printf("Error at %s:%d\n",__FILE__,__LINE__);\
  exit(EXIT_FAILURE);}} while(0)
#else
#define CURAND_CALL(x) x
#endif

#if defined(__HIPCC__) && defined(CUDA_NO_SM_13_DOUBLE_INTRINSICS)
    #error "-arch sm_13 nvcc flag is required to compile"
#endif

void CUDAHeunLLGSolver::syncOutput()
{
  using namespace globals;
  CUDA_CALL(hipMemcpy(s.ptr(),s_dev,(size_t)(nspins3*sizeof(double)),hipMemcpyDeviceToHost));
}

void CUDAHeunLLGSolver::initialise(int argc, char **argv, double idt)
{
  using namespace globals;

  // initialise base class
  Solver::initialise(argc,argv,idt);

  int deviceCount = 0;
  if (hipGetDeviceCount(&deviceCount) != hipSuccess) {
    jams_error("hipGetDeviceCount FAILED CUDA Driver and Runtime version may be mismatched.\n");
  }

  if(deviceCount == 0){
    jams_error("There is no device supporting CUDA\n");
  }

  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, 0);
  
  output.write("  * CUDA Device compute capability %d.%d\n",deviceProp.major,deviceProp.minor);

  sigma.resize(nspins);

  for(int i=0; i<nspins; ++i) {
    sigma(i) = sqrt( (2.0*boltzmann_si*alpha(i)) / (dt*mus(i)*mu_bohr_si) );
  }


  output.write("  * CUDA Heun LLG solver (GPU)\n");

  //-------------------------------------------------------------------
  //  Initialise hiprand
  //-------------------------------------------------------------------

  output.write("  * Initialising CURAND...\n");
  // hiprand generator
  CURAND_CALL(hiprandCreateGenerator(&gen,HIPRAND_RNG_PSEUDO_DEFAULT));


  // TODO: set random seed from config
  const unsigned long long gpuseed = rng.uniform()*18446744073709551615ULL;
  CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen, gpuseed));
  CUDA_CALL(hipDeviceSynchronize());
  CUDA_CALL(cudaThreadSetLimit(hipLimitStackSize,1024));


  //-------------------------------------------------------------------
  //  Allocate device memory
  //-------------------------------------------------------------------

  output.write("  * Allocating device memory...\n");
  // spin arrays
  CUDA_CALL(hipMalloc((void**)&s_dev,nspins3*sizeof(double)));
  CUDA_CALL(hipMalloc((void**)&sf_dev,nspins3*sizeof(float)));
  CUDA_CALL(hipMalloc((void**)&s_new_dev,nspins3*sizeof(double)));

  // field arrays
  CUDA_CALL(hipMalloc((void**)&h_dev,nspins3*sizeof(float)));

  if(nspins3%2 == 0) {
    // wiener processes
    CUDA_CALL(hipMalloc((void**)&w_dev,nspins3*sizeof(float)));
  } else {
    CUDA_CALL(hipMalloc((void**)&w_dev,(nspins3+1)*sizeof(float)));
  }


#ifdef FORCE_CUDA_DIA
  CUDA_CALL(hipMalloc((void**)&Jij_dev_row,(Jij.diags())*sizeof(int)));
//  CUDA_CALL(hipMalloc((void**)&Jij_dev_val,(Jij.rows()*Jij.diags())*sizeof(float)));
  CUDA_CALL(hipMallocPitch((void**)&Jij_dev_val,&diaPitch,(Jij.rows())*sizeof(float),Jij.diags()));
#else
  // jij matrix
  CUDA_CALL(hipMalloc((void**)&Jij_dev_row,(Jij.rows()+1)*sizeof(int)));
  CUDA_CALL(hipMalloc((void**)&Jij_dev_col,Jij.nonZero()*sizeof(int)));
  CUDA_CALL(hipMalloc((void**)&Jij_dev_val,Jij.nonZero()*sizeof(float)));
#endif

  // material properties
  CUDA_CALL(hipMalloc((void**)&mat_dev,nspins*4*sizeof(float)));

  //-------------------------------------------------------------------
  //  Copy data to device
  //-------------------------------------------------------------------

  output.write("  * Copying data to device memory...\n");
  // initial spins
  Array2D<float> sf(nspins,3);
  for(int i=0; i<nspins; ++i) {
    for(int j=0; j<3; ++j) {
      sf(i,j) = static_cast<float>(s(i,j));
    }
  }
  CUDA_CALL(hipMemcpy(s_dev,s.ptr(),(size_t)(nspins3*sizeof(double)),hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(sf_dev,sf.ptr(),(size_t)(nspins3*sizeof(float)),hipMemcpyHostToDevice));

#ifdef FORCE_CUDA_DIA
  CUDA_CALL(hipMemcpy(Jij_dev_row,Jij.dia_offPtr(),
        (size_t)((Jij.diags())*(sizeof(int))),hipMemcpyHostToDevice));
//  CUDA_CALL(hipMemcpy(Jij_dev_val,Jij.valPtr(),
//        (size_t)((Jij.diags()*Jij.rows())*(sizeof(float))),hipMemcpyHostToDevice));
//  diaPitch = Jij.rows();
   CUDA_CALL(hipMemcpy2D(Jij_dev_val,diaPitch,Jij.valPtr(),Jij.rows()*sizeof(float),Jij.rows()*sizeof(float),Jij.diags(),hipMemcpyHostToDevice));
   diaPitch = diaPitch/sizeof(float);

#else
  // jij matrix
  CUDA_CALL(hipMemcpy(Jij_dev_row,Jij.rowPtr(),
        (size_t)((Jij.rows()+1)*(sizeof(int))),hipMemcpyHostToDevice));

  CUDA_CALL(hipMemcpy(Jij_dev_col,Jij.colPtr(),
        (size_t)((Jij.nonZero())*(sizeof(int))),hipMemcpyHostToDevice));

  CUDA_CALL(hipMemcpy(Jij_dev_val,Jij.valPtr(),
        (size_t)((Jij.nonZero())*(sizeof(float))),hipMemcpyHostToDevice));
#endif

  Array2D<float> mat(nspins,4);
  // material properties
  for(int i=0; i<nspins; ++i){
    mat(i,0) = mus(i);
    mat(i,1) = gyro(i);
    mat(i,2) = alpha(i);
    mat(i,3) = sigma(i);
  }
  CUDA_CALL(hipMemcpy(mat_dev,mat.ptr(),(size_t)(nspins*4*sizeof(float)),hipMemcpyHostToDevice));


  //-------------------------------------------------------------------
  //  Initialise arrays to zero
  //-------------------------------------------------------------------
  for(int i=0; i<nspins; ++i) {
    for(int j=0; j<3; ++j) {
      sf(i,j) = 0.0;
    }
  }
  
  CUDA_CALL(hipMemcpy(w_dev,sf.ptr(),(size_t)(nspins3*sizeof(float)),hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(h_dev,sf.ptr(),(size_t)(nspins3*sizeof(float)),hipMemcpyHostToDevice));

  //-------------------------------------------------------------------
  //  Initialise cusparse
  //-------------------------------------------------------------------

#ifndef FORCE_CUDA_DIA
  output.write("  * Initialising CUSPARSE...\n");
  hipsparseStatus_t status;
  status = hipsparseCreate(&handle);
  if (status != HIPSPARSE_STATUS_SUCCESS) {
    jams_error("CUSPARSE Library initialization failed");
  }

  // create matrix descriptor
  status = hipsparseCreateMatDescr(&descra);
  if (status != HIPSPARSE_STATUS_SUCCESS) {
    jams_error("CUSPARSE Matrix descriptor initialization failed");
  }
  hipsparseSetMatType(descra,HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatIndexBase(descra,HIPSPARSE_INDEX_BASE_ZERO);
#endif

  nblocks = (nspins+BLOCKSIZE-1)/BLOCKSIZE;

  spmvblocks = std::min<int>(DIA_BLOCK_SIZE,(nspins3+DIA_BLOCK_SIZE-1)/DIA_BLOCK_SIZE);

  initialised = true;
}

void CUDAHeunLLGSolver::run()
{
  using namespace globals;

  // generate wiener trajectories
  float stmp = sqrt(temperature);
  
  if(temperature > 0.0) {
    if(nspins3%2 == 0) {
      CURAND_CALL(hiprandGenerateNormal(gen, w_dev, nspins3, 0.0f, stmp));
    } else {
      CURAND_CALL(hiprandGenerateNormal(gen, w_dev, (nspins3+1), 0.0f, stmp));
    }
  }
  
  // calculate interaction fields (and zero field array)
#ifdef FORCE_CUDA_DIA
  size_t offset = size_t(-1);
  CUDA_CALL(hipBindTexture(&offset,tex_x_float,sf_dev));
//  if(offset !=0){
//    jams_error("Failed to bind texture");
//  }
  spmv_dia_kernel<<< spmvblocks, DIA_BLOCK_SIZE >>>(nspins3,nspins3,
    Jij.diags(),diaPitch,Jij_dev_row,Jij_dev_val,sf_dev,h_dev);
  CUDA_CALL(hipUnbindTexture(tex_x_float));
  
#else
  hipsparseStatus_t stat =
  hipsparseScsrmv(handle,HIPSPARSE_OPERATION_NON_TRANSPOSE,nspins3,nspins3,1.0,descra,
      Jij_dev_val,Jij_dev_row,Jij_dev_col,sf_dev,0.0,h_dev);
  if(stat != HIPSPARSE_STATUS_SUCCESS){
    jams_error("CUSPARSE FAILED\n");
  }
#endif
  // integrate
  cuda_heun_llg_kernelA<<<nblocks,BLOCKSIZE>>>
    (
      s_dev,
      sf_dev,
      s_new_dev,
      h_dev,
      w_dev,
      mat_dev,
      h_app[0],
      h_app[1],
      h_app[2],
      nspins,
      dt
    );

  // calculate interaction fields (and zero field array)
#ifdef FORCE_CUDA_DIA
  CUDA_CALL(hipBindTexture(&offset,tex_x_float,sf_dev));
//  if(offset !=0){
//    jams_error("Failed to bind texture");
//  }
  spmv_dia_kernel<<< spmvblocks, DIA_BLOCK_SIZE >>>(nspins3,nspins3,
    Jij.diags(),diaPitch,Jij_dev_row,Jij_dev_val,sf_dev,h_dev);
  CUDA_CALL(hipUnbindTexture(tex_x_float));
#else
  hipsparseScsrmv(handle,HIPSPARSE_OPERATION_NON_TRANSPOSE,nspins3,nspins3,1.0,descra,
      Jij_dev_val,Jij_dev_row,Jij_dev_col,sf_dev,0.0,h_dev);
#endif
  
  cuda_heun_llg_kernelB<<<nblocks,BLOCKSIZE>>>
    (
      s_dev,
      sf_dev,
      s_new_dev,
      h_dev,
      w_dev,
      mat_dev,
      h_app[0],
      h_app[1],
      h_app[2],
      nspins,
      dt
    );
  iteration++;
}

CUDAHeunLLGSolver::~CUDAHeunLLGSolver()
{
  hiprandDestroyGenerator(gen);
  
  hipsparseStatus_t status;

  status = hipsparseDestroyMatDescr(descra);
  if (status != HIPSPARSE_STATUS_SUCCESS) {
    jams_error("CUSPARSE matrix destruction failed");
  }
  CUDA_CALL(hipDeviceSynchronize());

  status = hipsparseDestroy(handle);
  if (status != HIPSPARSE_STATUS_SUCCESS) {
    jams_error("CUSPARSE Library destruction failed");
  }
  CUDA_CALL(hipDeviceSynchronize());

  //-------------------------------------------------------------------
  //  Free device memory
  //-------------------------------------------------------------------

  // spin arrays
  CUDA_CALL(hipFree(s_dev));
  CUDA_CALL(hipFree(sf_dev));
  CUDA_CALL(hipFree(s_new_dev));

  // field arrays
  CUDA_CALL(hipFree(h_dev));

  // wiener processes
  CUDA_CALL(hipFree(w_dev));

  // jij matrix
  CUDA_CALL(hipFree(Jij_dev_row));
  CUDA_CALL(hipFree(Jij_dev_col));
  CUDA_CALL(hipFree(Jij_dev_val));

  // material arrays
  CUDA_CALL(hipFree(mat_dev));
}


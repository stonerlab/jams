#include "hip/hip_runtime.h"
#include "cuda_spmv.h"
#include "cuda_biquadratic.h"
#include "cuda_fourspin.h"
#include "cuda_heunllg_kernel.cu"
#include "globals.h"
#include "consts.h"

#include "cuda_heunllg.h"

#include <hiprand.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsparse.h>

#include <cmath>

// block size for GPU, 64 appears to be most efficient for current kernel
#define BLOCKSIZE 64

#ifndef NDEBUG
#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
  printf("Error at %s:%d\n",__FILE__,__LINE__);\
    exit(EXIT_FAILURE);}} while(0)
#else
#define CUDA_CALL(x) x
#endif

#ifndef NDEBUG
#define CURAND_CALL(x) do { if((x) != HIPRAND_STATUS_SUCCESS) { \
  printf("Error at %s:%d\n",__FILE__,__LINE__);\
  exit(EXIT_FAILURE);}} while(0)
#else
#define CURAND_CALL(x) x
#endif

#if defined(__HIPCC__) && defined(CUDA_NO_SM_13_DOUBLE_INTRINSICS)
    #error "-arch sm_13 nvcc flag is required to compile"
#endif

void allocate_transfer_dia(SparseMatrix<float> &Jij, devDIA &Jij_dev)
{
  CUDA_CALL(hipMalloc((void**)&Jij_dev.row,(Jij.diags())*sizeof(int)));
  CUDA_CALL(hipMallocPitch((void**)&Jij_dev.val,&Jij_dev.pitch,(Jij.rows())*sizeof(float),Jij.diags()));
  
  CUDA_CALL(hipMemcpy(Jij_dev.row,Jij.dia_offPtr(),(size_t)((Jij.diags())*(sizeof(int))),hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy2D(Jij_dev.val,Jij_dev.pitch,Jij.valPtr(),Jij.rows()*sizeof(float),Jij.rows()*sizeof(float),Jij.diags(),hipMemcpyHostToDevice));
  Jij_dev.pitch = Jij_dev.pitch/sizeof(float);
}

void free_dia(devDIA &Jij_dev)
{
  CUDA_CALL(hipFree(Jij_dev.row));
  CUDA_CALL(hipFree(Jij_dev.col));
  CUDA_CALL(hipFree(Jij_dev.val));
}

void allocate_transfer_csr_4d(SparseMatrix4D<float> &Jij, devCSR &
    Jij_dev)
{
  CUDA_CALL(hipMalloc((void**)&Jij_dev.pointers,(Jij.size(0)+1)*sizeof(int)));
  CUDA_CALL(hipMalloc((void**)&Jij_dev.coords,(3*Jij.nonZero())*sizeof(int)));
  CUDA_CALL(hipMalloc((void**)&Jij_dev.val,(Jij.nonZero())*sizeof(float)));

  CUDA_CALL(hipMemcpy(Jij_dev.pointers,Jij.pointersPtr(),(size_t)((Jij.size(0)+1)*(sizeof(int))),hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(Jij_dev.coords,Jij.cooPtr(),(size_t)((3*Jij.nonZero())*(sizeof(int))),hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(Jij_dev.val,Jij.valPtr(),(size_t)((Jij.nonZero())*(sizeof(float))),hipMemcpyHostToDevice));
}

void free_csr_4d(devCSR &Jij_dev)
{
  CUDA_CALL(hipFree(Jij_dev.pointers));
  CUDA_CALL(hipFree(Jij_dev.coords));
  CUDA_CALL(hipFree(Jij_dev.val));
}

void CUDAHeunLLGSolver::syncOutput()
{
  using namespace globals;
  CUDA_CALL(hipMemcpy(s.ptr(),s_dev,(size_t)(nspins3*sizeof(double)),hipMemcpyDeviceToHost));
}

void CUDAHeunLLGSolver::initialise(int argc, char **argv, double idt)
{
  using namespace globals;

  // initialise base class
  Solver::initialise(argc,argv,idt);

// POSSIBLY BUGGY AT THE MOMENT -> INVESTIGATE
//  int deviceCount = 0;
//  if (hipGetDeviceCount(&deviceCount) != hipSuccess) {
//    jams_error("hipGetDeviceCount FAILED CUDA Driver and Runtime version may be mismatched.\n");
//  }
//
//  if(deviceCount == 0){
//    jams_error("There is no device supporting CUDA\n");
//  }

  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, 0);
  
  output.write("  * CUDA Device compute capability %d.%d\n",deviceProp.major,deviceProp.minor);

  sigma.resize(nspins);

  for(int i=0; i<nspins; ++i) {
    sigma(i) = sqrt( (2.0*boltzmann_si*alpha(i)) / (dt*mus(i)*mu_bohr_si) );
  }


  output.write("  * CUDA Heun LLG solver (GPU)\n");

  //-------------------------------------------------------------------
  //  Initialise hiprand
  //-------------------------------------------------------------------

  output.write("  * Initialising CURAND...\n");
  // hiprand generator
  CURAND_CALL(hiprandCreateGenerator(&gen,HIPRAND_RNG_PSEUDO_DEFAULT));


  // TODO: set random seed from config
  const unsigned long long gpuseed = rng.uniform()*18446744073709551615ULL;
  CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen, gpuseed));
  CURAND_CALL(hiprandGenerateSeeds(gen));
  CUDA_CALL(cudaThreadSetLimit(hipLimitStackSize,1024));
  CUDA_CALL(hipDeviceSynchronize());


  //-------------------------------------------------------------------
  //  Allocate device memory
  //-------------------------------------------------------------------

#ifdef FORCE_CUDA_DIA
  output.write("  * Converting MAP to DIA\n");
  J1ij_s.convertMAP2DIA();
  J1ij_t.convertMAP2DIA();
  J2ij_s.convertMAP2DIA();
  J2ij_t.convertMAP2DIA();
  output.write("  * J1ij scalar matrix memory (DIA): %f MB\n",J1ij_s.calculateMemory());
  output.write("  * J1ij tensor matrix memory (DIA): %f MB\n",J1ij_t.calculateMemory());
  output.write("  * J2ij scalar matrix memory (DIA): %f MB\n",J2ij_s.calculateMemory());
  output.write("  * J2ij tensor matrix memory (DIA): %f MB\n",J2ij_t.calculateMemory());
#else
#error "CUDA CSR is not supported in this build"
#endif
  
  output.write("  * Converting J4 MAP to CSR\n");
  J4ijkl_s.convertMAP2CSR();
  output.write("  * J2ij scalar matrix memory (DIA): %f MB\n",J4ijkl_s.calculateMemory());


  output.write("  * Allocating device memory...\n");
  // spin arrays
  CUDA_CALL(hipMalloc((void**)&s_dev,nspins3*sizeof(double)));
  CUDA_CALL(hipMalloc((void**)&sf_dev,nspins3*sizeof(float)));
  CUDA_CALL(hipMalloc((void**)&s_new_dev,nspins3*sizeof(double)));

  // field arrays
  CUDA_CALL(hipMalloc((void**)&h_dev,nspins3*sizeof(float)));
  CUDA_CALL(hipMalloc((void**)&e_dev,nspins3*sizeof(float)));

  if(nspins3%2 == 0) {
    // wiener processes
    CUDA_CALL(hipMalloc((void**)&w_dev,nspins3*sizeof(float)));
  } else {
    CUDA_CALL(hipMalloc((void**)&w_dev,(nspins3+1)*sizeof(float)));
  }


#ifdef FORCE_CUDA_DIA
  // bilinear scalar
  allocate_transfer_dia(J1ij_s, J1ij_s_dev);
  
  // bilinear tensor
  allocate_transfer_dia(J1ij_t, J1ij_t_dev);
  
  // biquadratic scalar
  allocate_transfer_dia(J2ij_s, J2ij_s_dev);
  
  // bilinear tensor
  allocate_transfer_dia(J2ij_t, J2ij_t_dev);
#else
#error "CUDA CSR is not supported in this build"
#endif

  allocate_transfer_csr_4d(J4ijkl_s, J4ijkl_s_dev);

  // material properties
  CUDA_CALL(hipMalloc((void**)&mat_dev,nspins*4*sizeof(float)));

  //-------------------------------------------------------------------
  //  Copy data to device
  //-------------------------------------------------------------------

  output.write("  * Copying data to device memory...\n");
  // initial spins
  Array2D<float> sf(nspins,3);
  for(int i=0; i<nspins; ++i) {
    for(int j=0; j<3; ++j) {
      sf(i,j) = static_cast<float>(s(i,j));
    }
  }
  CUDA_CALL(hipMemcpy(s_dev,s.ptr(),(size_t)(nspins3*sizeof(double)),hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(sf_dev,sf.ptr(),(size_t)(nspins3*sizeof(float)),hipMemcpyHostToDevice));

  Array2D<float> mat(nspins,4);
  // material properties
  for(int i=0; i<nspins; ++i){
    mat(i,0) = mus(i);
    mat(i,1) = gyro(i);
    mat(i,2) = alpha(i);
    mat(i,3) = sigma(i);
  }
  CUDA_CALL(hipMemcpy(mat_dev,mat.ptr(),(size_t)(nspins*4*sizeof(float)),hipMemcpyHostToDevice));

  eng.resize(nspins,3);


  //-------------------------------------------------------------------
  //  Initialise arrays to zero
  //-------------------------------------------------------------------
  for(int i=0; i<nspins; ++i) {
    for(int j=0; j<3; ++j) {
      sf(i,j) = 0.0;
    }
  }
  
  CUDA_CALL(hipMemcpy(w_dev,sf.ptr(),(size_t)(nspins3*sizeof(float)),hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(h_dev,sf.ptr(),(size_t)(nspins3*sizeof(float)),hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(e_dev,sf.ptr(),(size_t)(nspins3*sizeof(float)),hipMemcpyHostToDevice));

  //-------------------------------------------------------------------
  //  Initialise cusparse
  //-------------------------------------------------------------------

#ifndef FORCE_CUDA_DIA
  output.write("  * Initialising CUSPARSE...\n");
  hipsparseStatus_t status;
  status = hipsparseCreate(&handle);
  if (status != HIPSPARSE_STATUS_SUCCESS) {
    jams_error("CUSPARSE Library initialization failed");
  }

  // create matrix descriptor
  status = hipsparseCreateMatDescr(&descra);
  if (status != HIPSPARSE_STATUS_SUCCESS) {
    jams_error("CUSPARSE Matrix descriptor initialization failed");
  }
  hipsparseSetMatType(descra,HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatIndexBase(descra,HIPSPARSE_INDEX_BASE_ZERO);
#endif

  nblocks = (nspins+BLOCKSIZE-1)/BLOCKSIZE;

  J1ij_s_dev.blocks = std::min<int>(DIA_BLOCK_SIZE,(nspins+DIA_BLOCK_SIZE-1)/DIA_BLOCK_SIZE);
  J1ij_t_dev.blocks = std::min<int>(DIA_BLOCK_SIZE,(nspins3+DIA_BLOCK_SIZE-1)/DIA_BLOCK_SIZE);

  J2ij_s_dev.blocks = std::min<int>(DIA_BLOCK_SIZE,(nspins+DIA_BLOCK_SIZE-1)/DIA_BLOCK_SIZE);
  J2ij_t_dev.blocks = std::min<int>(DIA_BLOCK_SIZE,(nspins3+DIA_BLOCK_SIZE-1)/DIA_BLOCK_SIZE);
  
  J4ijkl_s_dev.blocks = std::min<int>(CSR_4D_BLOCK_SIZE,(nspins+CSR_4D_BLOCK_SIZE-1)/CSR_4D_BLOCK_SIZE);

  initialised = true;
}

void CUDAHeunLLGSolver::run()
{
  using namespace globals;

  // generate wiener trajectories
  float stmp = sqrt(temperature);
  
  if(temperature > 0.0) {
    if(nspins3%2 == 0) {
      CURAND_CALL(hiprandGenerateNormal(gen, w_dev, nspins3, 0.0f, stmp));
    } else {
      CURAND_CALL(hiprandGenerateNormal(gen, w_dev, (nspins3+1), 0.0f, stmp));
    }
  }
  
  // calculate interaction fields (and zero field array)
#ifdef FORCE_CUDA_DIA

  CUDA_CALL(hipBindTexture(0,tex_x_float,sf_dev));
  
  float beta=0;
  // bilinear scalar
  if(J1ij_s.nonZero() > 0){
    bilinear_scalar_dia_kernel<<< J1ij_s_dev.blocks, DIA_BLOCK_SIZE >>>(nspins,nspins,
      J1ij_s.diags(),J1ij_s_dev.pitch,1.0,beta,J1ij_s_dev.row,J1ij_s_dev.val,sf_dev,h_dev);
    beta = 1.0;
  }

  // bilinear tensor
  if(J1ij_t.nonZero() > 0){
    spmv_dia_kernel<<< J1ij_t_dev.blocks, DIA_BLOCK_SIZE >>>(nspins3,nspins3,
      J1ij_t.diags(),J1ij_t_dev.pitch,beta,1.0,J1ij_t_dev.row,J1ij_t_dev.val,sf_dev,h_dev);
    beta = 1.0;
  }
  
  // biquadratic scalar
  if(J2ij_s.nonZero() > 0){
    biquadratic_scalar_dia_kernel<<< J2ij_s_dev.blocks, DIA_BLOCK_SIZE >>>(nspins,nspins,
      J2ij_s.diags(),J2ij_s_dev.pitch,2.0,beta,J2ij_s_dev.row,J2ij_s_dev.val,sf_dev,h_dev);
    beta = 1.0;
  }
  
  // biquadratic tensor
  if(J2ij_t.nonZero() > 0){
    spmv_dia_kernel<<< J2ij_t_dev.blocks, DIA_BLOCK_SIZE >>>(nspins3,nspins3,
      J2ij_t.diags(),J2ij_t_dev.pitch,2.0,beta,J2ij_t_dev.row,J2ij_t_dev.val,sf_dev,h_dev);
    beta = 1.0;
  }
  
  if(J4ijkl_s.nonZero() > 0){
    fourspin_scalar_csr_kernel<<< J4ijkl_s_dev.blocks,CSR_4D_BLOCK_SIZE>>>(nspins,nspins,1.0,beta,
        J4ijkl_s_dev.pointers,J4ijkl_s_dev.coords,J4ijkl_s_dev.val,h_dev);
    beta = 1.0;
  }
  
  CUDA_CALL(hipUnbindTexture(tex_x_float));
#else
#error "CUDA CSR is not supported in this build"
#endif
  // integrate
  cuda_heun_llg_kernelA<<<nblocks,BLOCKSIZE>>>
    (
      s_dev,
      sf_dev,
      s_new_dev,
      h_dev,
      w_dev,
      mat_dev,
      h_app[0],
      h_app[1],
      h_app[2],
      nspins,
      dt
    );

  // calculate interaction fields (and zero field array)
#ifdef FORCE_CUDA_DIA
  CUDA_CALL(hipBindTexture(0,tex_x_float,sf_dev));

  beta=0.0;
  // bilinear scalar
  if(J1ij_s.nonZero() > 0){
    bilinear_scalar_dia_kernel<<< J1ij_s_dev.blocks, DIA_BLOCK_SIZE >>>(nspins,nspins,
      J1ij_s.diags(),J1ij_s_dev.pitch,1.0,beta,J1ij_s_dev.row,J1ij_s_dev.val,sf_dev,h_dev);
    beta = 1.0;
  }

  // bilinear tensor
  if(J1ij_t.nonZero() > 0){
    spmv_dia_kernel<<< J1ij_t_dev.blocks, DIA_BLOCK_SIZE >>>(nspins3,nspins3,
      J1ij_t.diags(),J1ij_t_dev.pitch,1.0,beta,J1ij_t_dev.row,J1ij_t_dev.val,sf_dev,h_dev);
    beta = 1.0;
  }
  
  // biquadratic scalar
  if(J2ij_s.nonZero() > 0){
    biquadratic_scalar_dia_kernel<<< J2ij_s_dev.blocks, DIA_BLOCK_SIZE >>>(nspins,nspins,
      J2ij_s.diags(),J2ij_s_dev.pitch,2.0,beta,J2ij_s_dev.row,J2ij_s_dev.val,sf_dev,h_dev);
    beta = 1.0;
  }
  
  // biquadratic tensor
  if(J2ij_t.nonZero() > 0){
    spmv_dia_kernel<<< J2ij_t_dev.blocks, DIA_BLOCK_SIZE >>>(nspins3,nspins3,
      J2ij_t.diags(),J2ij_t_dev.pitch,2.0,beta,J2ij_t_dev.row,J2ij_t_dev.val,sf_dev,h_dev);
    beta = 1.0;
  }
  
  if(J4ijkl_s.nonZero() > 0){
    fourspin_scalar_csr_kernel<<< J4ijkl_s_dev.blocks,CSR_4D_BLOCK_SIZE>>>(nspins,nspins1.0,beta,
        J4ijkl_s_dev.pointers,J4ijkl_s_dev.coords,J4ijkl_s_dev.val,h_dev);
    beta = 1.0;
  }
  
  CUDA_CALL(hipUnbindTexture(tex_x_float));
#else
#error "CUDA CSR is not supported in this build"
#endif
  
  cuda_heun_llg_kernelB<<<nblocks,BLOCKSIZE>>>
    (
      s_dev,
      sf_dev,
      s_new_dev,
      h_dev,
      w_dev,
      mat_dev,
      h_app[0],
      h_app[1],
      h_app[2],
      nspins,
      dt
    );
  iteration++;
}

void CUDAHeunLLGSolver::calcEnergy(double &e1_s, double &e1_t, double &e2_s, double &e2_t){
  using namespace globals;
  const float beta=0.0;

  e1_s = 0.0; e1_t = 0.0; e2_s = 0.0; e2_t = 0.0;
  
  size_t offset = size_t(-1);
  CUDA_CALL(hipBindTexture(&offset,tex_x_float,sf_dev));

  // bilinear scalar
  if(J1ij_s.nonZero() > 0){
    bilinear_scalar_dia_kernel<<< J1ij_s_dev.blocks, DIA_BLOCK_SIZE >>>(nspins,nspins,
      J1ij_s.diags(),J1ij_s_dev.pitch,1.0,beta,J1ij_s_dev.row,J1ij_s_dev.val,sf_dev,e_dev);
    CUDA_CALL(hipMemcpy(eng.ptr(),e_dev,(size_t)(nspins3*sizeof(float)),hipMemcpyDeviceToHost));
    for(int i=0; i<nspins; ++i){
      e1_s = e1_s + (s(i,0)*eng(i,0)+s(i,1)*eng(i,1)+s(i,2)*eng(i,2));
    }
    e1_s = e1_s/nspins;
  }


  // bilinear tensor
  if(J1ij_t.nonZero() > 0){
    spmv_dia_kernel<<< J1ij_t_dev.blocks, DIA_BLOCK_SIZE >>>(nspins3,nspins3,
      J1ij_t.diags(),J1ij_t_dev.pitch,1.0,beta,J1ij_t_dev.row,J1ij_t_dev.val,sf_dev,e_dev);
    CUDA_CALL(hipMemcpy(eng.ptr(),e_dev,(size_t)(nspins3*sizeof(float)),hipMemcpyDeviceToHost));
    for(int i=0; i<nspins; ++i){
      e1_t = e1_t + (s(i,0)*eng(i,0)+s(i,1)*eng(i,1)+s(i,2)*eng(i,2));
    }
    e1_t = e1_t/nspins;
  }

  
  // biquadratic scalar
  if(J2ij_s.nonZero() > 0){
    biquadratic_scalar_dia_kernel<<< J2ij_s_dev.blocks, DIA_BLOCK_SIZE >>>(nspins,nspins,
      J2ij_s.diags(),J2ij_s_dev.pitch,1.0,beta,J2ij_s_dev.row,J2ij_s_dev.val,sf_dev,e_dev);
    CUDA_CALL(hipMemcpy(eng.ptr(),e_dev,(size_t)(nspins3*sizeof(float)),hipMemcpyDeviceToHost));
    for(int i=0; i<nspins; ++i){
      e2_s = e2_s + (s(i,0)*eng(i,0)+s(i,1)*eng(i,1)+s(i,2)*eng(i,2));
    }
    
    e2_s = e2_s/nspins;
  }

  // biquadratic tensor
  if(J2ij_t.nonZero() > 0){
    spmv_dia_kernel<<< J2ij_t_dev.blocks, DIA_BLOCK_SIZE >>>(nspins3,nspins3,
      J2ij_t.diags(),J2ij_t_dev.pitch,1.0,beta,J2ij_t_dev.row,J2ij_t_dev.val,sf_dev,e_dev);
    CUDA_CALL(hipMemcpy(eng.ptr(),e_dev,(size_t)(nspins3*sizeof(float)),hipMemcpyDeviceToHost));

    for(int i=0; i<nspins; ++i){
      e2_t = e2_t + (s(i,0)*eng(i,0)+s(i,1)*eng(i,1)+s(i,2)*eng(i,2));
    }
    
    e2_t = e2_t/nspins;
  }
  
  
  CUDA_CALL(hipUnbindTexture(tex_x_float));
}

CUDAHeunLLGSolver::~CUDAHeunLLGSolver()
{
  hiprandDestroyGenerator(gen);
  
  hipsparseStatus_t status;

  status = hipsparseDestroyMatDescr(descra);
  if (status != HIPSPARSE_STATUS_SUCCESS) {
    jams_error("CUSPARSE matrix destruction failed");
  }
  CUDA_CALL(hipDeviceSynchronize());

  status = hipsparseDestroy(handle);
  if (status != HIPSPARSE_STATUS_SUCCESS) {
    jams_error("CUSPARSE Library destruction failed");
  }
  CUDA_CALL(hipDeviceSynchronize());

  //-------------------------------------------------------------------
  //  Free device memory
  //-------------------------------------------------------------------

  free_dia(J1ij_s_dev);
  free_dia(J1ij_t_dev);
  free_dia(J2ij_s_dev);
  free_dia(J2ij_t_dev);
  free_csr_4d(J4ijkl_s_dev);

  // spin arrays
  CUDA_CALL(hipFree(s_dev));
  CUDA_CALL(hipFree(sf_dev));
  CUDA_CALL(hipFree(s_new_dev));

  // field arrays
  CUDA_CALL(hipFree(h_dev));
  CUDA_CALL(hipFree(e_dev));

  // wiener processes
  CUDA_CALL(hipFree(w_dev));


  // material arrays
  CUDA_CALL(hipFree(mat_dev));
}


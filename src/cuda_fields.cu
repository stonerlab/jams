#include "globals.h"
#include "cuda_sparse.h"
#include "cuda_sparse_types.h"

#include <hipblas.h>

#include "cuda_fields.h"

void CUDACalculateFields(
        const devDIA & J1ij_s_dev,
        const devDIA & J1ij_t_dev,
        const devDIA & J2ij_s_dev,
        const devDIA & J2ij_t_dev,
        const devCSR & J4ijkl_s_dev,
        const float *  sf_dev, 
        const float *  r_dev,
        const float *  r_max_dev,
        const float *  mat_dev,
        const bool *   pbc_dev,
        float *        h_dev,
        float *        h_dipole_dev,
        const bool     dipole_toggle
){
    using namespace globals;

    // used to zero the first field which is calculated (i.e. not to use the
    // field from the last time step)
    float beta = 0.0;

    // Bilinear Scalar Fields
    if(J1ij_s.nonZero() > 0){

        bilinear_scalar_dia_kernel<<< J1ij_s_dev.blocks, DIA_BLOCK_SIZE >>>
            (nspins,nspins,J1ij_s.diags(),J1ij_s_dev.pitch,1.0,beta,
             J1ij_s_dev.row,J1ij_s_dev.val,sf_dev,h_dev);

        beta = 1.0;
    }

    // Bilinear Tensor Fields
    if(J1ij_t.nonZero() > 0){

        spmv_dia_kernel<<< J1ij_t_dev.blocks, DIA_BLOCK_SIZE >>>
            (nspins3,nspins3,J1ij_t.diags(),J1ij_t_dev.pitch,beta,1.0,
             J1ij_t_dev.row,J1ij_t_dev.val,sf_dev,h_dev);

        beta = 1.0;

    }

    // Biquadratic Scalar Fields
    if(J2ij_s.nonZero() > 0){

        biquadratic_scalar_dia_kernel<<< J2ij_s_dev.blocks, DIA_BLOCK_SIZE >>>
            (nspins,nspins,J2ij_s.diags(),J2ij_s_dev.pitch,2.0,beta,
             J2ij_s_dev.row,J2ij_s_dev.val,sf_dev,h_dev);

        beta = 1.0;

    }

    // Biquadratic Tensor Fields
    if(J2ij_t.nonZero() > 0){

        spmv_dia_kernel<<< J2ij_t_dev.blocks, DIA_BLOCK_SIZE >>>
            (nspins3,nspins3,J2ij_t.diags(),J2ij_t_dev.pitch,2.0,beta,
             J2ij_t_dev.row,J2ij_t_dev.val,sf_dev,h_dev);

        beta = 1.0;

    }

    // Fourspin Scalar Fields
    if(J4ijkl_s.nonZero() > 0){

        fourspin_scalar_csr_kernel<<< J4ijkl_s_dev.blocks,CSR_4D_BLOCK_SIZE>>>
            (nspins,nspins,1.0,beta,
             J4ijkl_s_dev.pointers,J4ijkl_s_dev.coords,J4ijkl_s_dev.val,sf_dev,h_dev);

        beta = 1.0;

    }

    // Dipole-Dipole Fields
    const float dipole_omega = 0.00092740096; // (muB*mu0/4pi)/nm^3

    // We only really need to update the dipole field for the first integration
    // step in any scheme. This toggle is used to determine this. If the field
    // is not updated then the cached result is still added below
    if( dipole_toggle == true ){
        if(globalSteps%100 == 0){
            const int nblocks = (nspins+BLOCKSIZE-1)/BLOCKSIZE;
            dipole_brute_kernel<<<nblocks, BLOCKSIZE >>>
                (dipole_omega,0.0,sf_dev,mat_dev,h_dipole_dev,
                 r_dev,r_max_dev,pbc_dev,nspins);
        }
    }

    // add cached dipole-dipole field
    hipblasSaxpy(nspins3,1.0,h_dipole_dev,1,h_dev,1);

}

#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>

#include "jams/core/solver.h"

#include "jams/hamiltonian/uniaxial_anisotropy.h"
#include "jams/hamiltonian/cuda_uniaxial_anisotropy.h"
#include "jams/hamiltonian/cuda_uniaxial_anisotropy_kernel.cuh"

CudaUniaxialHamiltonian::CudaUniaxialHamiltonian(const libconfig::Setting &settings, const unsigned int num_spins)
        : UniaxialHamiltonian(settings, num_spins)
{
  dev_energy_ = jblib::CudaArray<double, 1>(energy_);
  dev_field_ = jblib::CudaArray<double, 1>(field_);

  dev_power_ = jblib::CudaArray<unsigned, 1>(power_);
  dev_magnitude_ = jblib::CudaArray<double, 1>(magnitude_);

  jblib::Array<double3, 1> tmp_axis(axis_.elements());

  for (auto i = 0; i < axis_.elements(); ++i) {
    tmp_axis[i] = {axis_[i][0], axis_[i][1], axis_[i][2]};
  }

  dev_axis_ = jblib::CudaArray<double3, 1>(tmp_axis);

  hipStreamCreate(&dev_stream_);

  dev_blocksize_ = 128;
}

void CudaUniaxialHamiltonian::calculate_fields() {
  cuda_uniaxial_field_kernel<<<(globals::num_spins+dev_blocksize_-1)/dev_blocksize_, dev_blocksize_, 0, dev_stream_>>>
            (globals::num_spins, num_coefficients_, dev_power_.data(), dev_magnitude_.data(), dev_axis_.data(), solver->dev_ptr_spin(), dev_field_.data());
}

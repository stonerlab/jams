#include <hip/hip_runtime_api.h>

#include "jams/core/solver.h"
#include "jams/cuda/cuda_common.h"

#include "jams/hamiltonian/uniaxial_anisotropy.h"
#include "jams/hamiltonian/cuda_uniaxial_anisotropy.h"
#include "jams/hamiltonian/cuda_uniaxial_anisotropy_kernel.cuh"

CudaUniaxialHamiltonian::CudaUniaxialHamiltonian(const libconfig::Setting &settings, const unsigned int num_spins)
        : UniaxialHamiltonian(settings, num_spins)
{
}

void CudaUniaxialHamiltonian::calculate_fields() {
  cuda_uniaxial_field_kernel<<<(globals::num_spins+dev_blocksize_-1)/dev_blocksize_, dev_blocksize_, 0, dev_stream_.get()>>>
            (globals::num_spins, num_coefficients_, power_.device_data(), magnitude_.device_data(), axis_.device_data(), globals::s.device_data(), field_.device_data());
  DEBUG_CHECK_CUDA_ASYNC_STATUS;
}

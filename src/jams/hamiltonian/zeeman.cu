#include "hip/hip_runtime.h"
#include "jams/core/globals.h"
#include "jams/core/utils.h"
#include "jams/core/maths.h"
#include "jams/core/consts.h"
#include "jams/core/cuda_defs.h"

#include "jams/hamiltonian/zeeman.h"
#include "jams/hamiltonian/zeeman_kernel.h"

ZeemanHamiltonian::ZeemanHamiltonian(const libconfig::Setting &settings)
: Hamiltonian(settings)
{
    ::output.write("initialising Zeeman Hamiltonian\n");
    // output in default format for now
    outformat_ = TEXT;

    // resize member arrays
    energy_.resize(globals::num_spins);
    energy_.zero();
    field_.resize(globals::num_spins, 3);
    field_.zero();

    dc_local_field_.resize(globals::num_spins, 3);
    dc_local_field_.zero();


    ac_local_field_.resize(globals::num_spins, 3);
    ac_local_frequency_.resize(globals::num_spins);

    ac_local_field_.zero();
    ac_local_frequency_.zero();


    if(settings.exists("dc_local_field")) {
        if (settings["dc_local_field"].getLength() != lattice.num_materials()) {
            jams_error("ZeemanHamiltonian: dc_local_field must be specified for every material");
        }


        for (int i = 0; i < globals::num_spins; ++i) {
            for (int j = 0; j < 3; ++j) {
                dc_local_field_(i, j) = settings["dc_local_field"][lattice.atom_material(i)][j];
                dc_local_field_(i, j) *= globals::mus(i);
            }
        }
    }

    if(settings.exists("ac_local")) {
        if (settings["ac_local"].getLength() != lattice.num_materials()) {
            jams_error("ZeemanHamiltonian: ac_local must be specified for every material");
        }
    }

    has_ac_local_field_ = false;
    if(settings.exists("ac_local_field") || settings.exists("ac_local_frequency")) {
        if(!(settings.exists("ac_local_field") && settings.exists("ac_local_frequency"))) {
            jams_error("ZeemanHamiltonian: ac_local must have a field and a frequency");
        }
        if (settings["ac_local_frequency"].getLength() != lattice.num_materials()) {
            jams_error("ZeemanHamiltonian: ac_local_frequency must be specified for every material");
        }
        if (settings["ac_local_field"].getLength() != lattice.num_materials()) {
            jams_error("ZeemanHamiltonian: ac_local_field must be specified for every material");
        }

        has_ac_local_field_ = true;

        for (int i = 0; i < globals::num_spins; ++i) {
            for (int j = 0; j < 3; ++j) {
                ac_local_field_(i, j) = settings["ac_local_field"][lattice.atom_material(i)][j];
                ac_local_field_(i, j) *= globals::mus(i);
            }
        }

        for (int i = 0; i < globals::num_spins; ++i) {
            ac_local_frequency_(i) = settings["ac_local_frequency"][lattice.atom_material(i)];
            ac_local_frequency_(i) = kTwoPi*ac_local_frequency_(i);
        }
    }

    // transfer arrays to cuda device if needed
#ifdef CUDA
    if (solver->is_cuda_solver()) {
        hipStreamCreate(&dev_stream_);

        dev_energy_ = jblib::CudaArray<double, 1>(energy_);
        dev_field_  = jblib::CudaArray<double, 1>(field_);

        dev_dc_local_field_ = jblib::CudaArray<double, 1>(dc_local_field_);

        dev_ac_local_field_ = jblib::CudaArray<double, 1>(ac_local_field_);
        dev_ac_local_frequency_ = jblib::CudaArray<double, 1>(ac_local_frequency_);
    }
#endif

}

// --------------------------------------------------------------------------

double ZeemanHamiltonian::calculate_total_energy() {
    double e_total = 0.0;
    for (int i = 0; i < globals::num_spins; ++i) {
        e_total += calculate_one_spin_energy(i);
    }
     return e_total;
}

// --------------------------------------------------------------------------

double ZeemanHamiltonian::calculate_one_spin_energy(const int i) {
    using namespace globals;
    double one_spin_field[3];

    calculate_one_spin_field(i, one_spin_field);

    return -(s(i, 0)*one_spin_field[0] + s(i, 1)*one_spin_field[1] + s(i, 2)*one_spin_field[2]);
}

// --------------------------------------------------------------------------

double ZeemanHamiltonian::calculate_one_spin_energy_difference(const int i, const jblib::Vec3<double> &spin_initial, const jblib::Vec3<double> &spin_final) {
    using std::pow;

    double e_initial = 0.0;
    double e_final = 0.0;

    double h_local[3];

    calculate_one_spin_field(i, h_local);

    for (int n = 0; n < 3; ++n) {
        e_initial += -spin_initial[n]*h_local[n];
    }

    for (int n = 0; n < 3; ++n) {
        e_final += -spin_final[n]*h_local[n];
    }

    return (e_final - e_initial);
}

// --------------------------------------------------------------------------

void ZeemanHamiltonian::calculate_energies() {
    for (int i = 0; i < globals::num_spins; ++i) {
        energy_[i] = calculate_one_spin_energy(i);
    }
}

// --------------------------------------------------------------------------

void ZeemanHamiltonian::calculate_one_spin_field(const int i, double local_field[3]) {
    using namespace globals;
    using std::pow;

    for (int j = 0; j < 3; ++j) {
        local_field[j] = dc_local_field_(i, j);
    }

    if (has_ac_local_field_) {
        for (int j = 0; j < 3; ++j) {
            local_field[j] += ac_local_field_(i, j) * cos(ac_local_frequency_(i) * solver->time());
        }
    }
}



// --------------------------------------------------------------------------

void ZeemanHamiltonian::calculate_fields() {
    if (solver->is_cuda_solver()) {
#ifdef CUDA
        dim3 block_size;
        block_size.x = 32;
        block_size.y = 4;

        dim3 grid_size;
        grid_size.x = (globals::num_spins + block_size.x - 1) / block_size.x;
        grid_size.y = (3 + block_size.y - 1) / block_size.y;

        cuda_api_error_check(
          hipMemcpyAsync(dev_field_.data(),           // void *               dst
                     dev_dc_local_field_.data(),               // const void *         src
                     globals::num_spins3*sizeof(double),   // size_t               count
                     hipMemcpyDeviceToDevice,    // enum hipMemcpyKind  kind
                     dev_stream_)                   // device stream
        );

        if (has_ac_local_field_) {
            cuda_zeeman_ac_field_kernel<<<grid_size, block_size, 0, dev_stream_>>>
                (globals::num_spins, solver->time(),
                    dev_ac_local_field_.data(), dev_ac_local_frequency_.data(),
                    solver->dev_ptr_spin(), dev_field_.data());
            cuda_kernel_error_check();
        }
#endif  // CUDA
    } else {
        for (int i = 0; i < globals::num_spins; ++i) {
            for (int j = 0; j < 3; ++j) {
                field_(i, j) = dc_local_field_(i, j) + ac_local_field_(i, j) * cos(ac_local_frequency_(i) * solver->time());
            }
        }
    }
}
// --------------------------------------------------------------------------

void ZeemanHamiltonian::output_energies(OutputFormat format) {
    switch(format) {
        case TEXT:
            output_energies_text();
        case HDF5:
            jams_error("Zeeman energy output: HDF5 not yet implemented");
        default:
            jams_error("Zeeman energy output: unknown format");
    }
}

// --------------------------------------------------------------------------

void ZeemanHamiltonian::output_fields(OutputFormat format) {
    switch(format) {
        case TEXT:
            output_fields_text();
        case HDF5:
            jams_error("Zeeman energy output: HDF5 not yet implemented");
        default:
            jams_error("Zeeman energy output: unknown format");
    }
}

// --------------------------------------------------------------------------

void ZeemanHamiltonian::output_energies_text() {

}

// --------------------------------------------------------------------------

void ZeemanHamiltonian::output_fields_text() {

}

double ZeemanHamiltonian::calculate_bond_energy_difference(const int i, const int j, const Vec3 &sj_initial, const Vec3 &sj_final) {
  if (i != j) {
    return 0.0;
    } else {
  return calculate_one_spin_energy_difference(i, sj_initial, sj_final);
    }
}

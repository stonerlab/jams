#include <iosfwd>

#include <hip/hip_runtime_api.h>

#include "jams/core/solver.h"
#include "jams/helpers/cuda_exception.h"
#include "jams/hamiltonian/cuda_exchange.h"

namespace {
#if HAS_CUSPARSE_MIXED_PREC
    // alg is a required argument even from CUDA 9, but the types are not implemented until CUDA 10
#if __CUDACC_VER_MAJOR__ >= 10
    cusparseAlgMode_t alg = CUSPARSE_ALG_NAIVE;
#else
    cusparseAlgMode_t alg;
#endif
#endif
}


CudaExchangeHamiltonian::CudaExchangeHamiltonian(const libconfig::Setting &settings, const unsigned int size)
: ExchangeHamiltonian(settings, size)
{
    dev_energy_ = jblib::CudaArray<double, 1>(energy_);
    dev_field_  = jblib::CudaArray<double, 1>(field_);

    std::cout << "    init cusparse\n";
    hipsparseStatus_t status = hipsparseCreate(&cusparse_handle_);
    if (status != HIPSPARSE_STATUS_SUCCESS) {
      die("cusparse Library initialization failed");
    }
    hipsparseSetStream(cusparse_handle_, dev_stream_.get());

    sparsematrix_copy_host_csr_to_cuda_csr(interaction_matrix_, dev_csr_interaction_matrix_);

#if HAS_CUSPARSE_MIXED_PREC

  float one = 1.0;
  float zero = 0.0;
  const int num_rows = globals::num_spins3;
  const int num_cols = globals::num_spins3;
  cusparseCsrmvEx_bufferSize(
          cusparse_handle_,
          alg,
          HIPSPARSE_OPERATION_NON_TRANSPOSE,
          num_rows,
          num_cols,
          interaction_matrix_.nonZero(),
          &one, HIP_R_32F,
          dev_csr_interaction_matrix_.descr,
          dev_csr_interaction_matrix_.val, HIP_R_32F,
          dev_csr_interaction_matrix_.row,
          dev_csr_interaction_matrix_.col,
          solver->dev_ptr_spin(), HIP_R_64F,
          &zero, HIP_R_32F,
          dev_field_.data(), HIP_R_64F,
          HIP_R_32F, // execution type
          &dev_csr_buffer_size_);

  cuda_api_error_check(
          hipMalloc((void**)&dev_csr_buffer_, dev_csr_buffer_size_));
#endif
}

double CudaExchangeHamiltonian::calculate_total_energy() {
  double total_energy = 0.0;
  calculate_fields();
  dev_field_.copy_to_host_array(field_);
  for (auto i = 0; i < globals::num_spins; ++i) {
    total_energy += -(  globals::s(i,0)*field_(i,0)
                        + globals::s(i,1)*field_(i,1)
                        + globals::s(i,2)*field_(i,2) );
  }
  return 0.5*total_energy;
}

void CudaExchangeHamiltonian::calculate_fields() {
  assert(interaction_matrix_.getMatrixType() == SPARSE_MATRIX_TYPE_GENERAL);

  const int num_rows = globals::num_spins3;
  const int num_cols = globals::num_spins3;

#if HAS_CUSPARSE_MIXED_PREC
  float one = 1.0;
  float zero = 0.0;

  hipsparseStatus_t stat = cusparseCsrmvEx(
          cusparse_handle_,
          alg,
          HIPSPARSE_OPERATION_NON_TRANSPOSE,
          num_rows,
          num_cols,
          interaction_matrix_.nonZero(),
          &one, HIP_R_32F,
          dev_csr_interaction_matrix_.descr,
          dev_csr_interaction_matrix_.val, HIP_R_32F,
          dev_csr_interaction_matrix_.row,
          dev_csr_interaction_matrix_.col,
          solver->dev_ptr_spin(), HIP_R_64F,
          &zero, HIP_R_32F,
          dev_field_.data(), HIP_R_64F,
          HIP_R_32F, // execution type
          dev_csr_buffer_);
#else
  double one = 1.0;
  double zero = 0.0;

  hipsparseStatus_t stat =
          hipsparseDcsrmv(cusparse_handle_,
                         HIPSPARSE_OPERATION_NON_TRANSPOSE,
                         num_rows,
                         num_cols,
                         interaction_matrix_.nonZero(),
                         &one,
                         dev_csr_interaction_matrix_.descr,
                         dev_csr_interaction_matrix_.val,
                         dev_csr_interaction_matrix_.row,
                         dev_csr_interaction_matrix_.col,
                         solver->dev_ptr_spin(),
                         &zero,
                         dev_field_.data());
#endif

  if (debug_is_enabled()) {
    if (stat != HIPSPARSE_STATUS_SUCCESS) {
      throw cuda_api_exception("cusparse failure", __FILE__, __LINE__, __PRETTY_FUNCTION__);
    }
  }
}
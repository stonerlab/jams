#include <iosfwd>

#include <hip/hip_runtime_api.h>

#include "jams/core/solver.h"
#include "jams/cuda/cuda_sparse_interaction_matrix.h"
#include "jams/hamiltonian/cuda_exchange.h"

CudaExchangeHamiltonian::CudaExchangeHamiltonian(const libconfig::Setting &settings, const unsigned int size)
: ExchangeHamiltonian(settings, size)
{
    dev_energy_ = jblib::CudaArray<double, 1>(energy_);
    dev_field_  = jblib::CudaArray<double, 1>(field_);

    dev_interaction_matrix_.create_matrix(interaction_matrix_);
    dev_interaction_matrix_.set_cuda_stream(dev_stream_.get());
}

double CudaExchangeHamiltonian::calculate_total_energy() {
  double total_energy = 0.0;
  calculate_fields();
  dev_field_.copy_to_host_array(field_);
  for (auto i = 0; i < globals::num_spins; ++i) {
    total_energy += -(  globals::s(i,0)*field_(i,0)
                        + globals::s(i,1)*field_(i,1)
                        + globals::s(i,2)*field_(i,2) );
  }
  return 0.5*total_energy;
}

void CudaExchangeHamiltonian::calculate_fields() {
  dev_interaction_matrix_.calculate_fields(solver->dev_ptr_spin(), dev_ptr_field());
}
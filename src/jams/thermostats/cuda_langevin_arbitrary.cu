#include "hip/hip_runtime.h"
// Copyright 2014 Joseph Barker. All rights reserved.

#include <cmath>
#include <string>
#include <iomanip>
#include <random>
#include <mutex>

#include "jams/helpers/utils.h"
#include "jams/cuda/cuda_array_kernels.h"

#include "jams/core/solver.h"
#include "jams/core/globals.h"
#include "jams/core/lattice.h"
#include "jams/core/solver.h"
#include "jams/cuda/cuda_array_kernels.h"
#include "jams/helpers/consts.h"
#include "jams/helpers/error.h"
#include "jams/helpers/random.h"
#include "jams/helpers/utils.h"
#include "jams/cuda/cuda_common.h"
#include "jams/monitors/magnetisation.h"
#include "jams/thermostats/cuda_langevin_arbitrary.h"
#include "jams/thermostats/cuda_langevin_arbitrary_kernel.h"

using namespace std;

namespace {

// convert linear array index (in fftw ordering) into k index (+/-)
// size is the total size of the array
inline int fftw_k_index(const int i, const int size) {
  assert(i < (2 * size - 1));
  if (i < size) {
    return i;
  } else {
    return i - (2 * size - 1);
  }
}

double coth(const double x) {
  return 1 / tanh(x);
}

//// arbitrary correlation function
//double correlator(const double omega) {
//  if (omega == 0.0) return 1.0;
//  return abs(omega)*coth(abs(omega));
//}

// arbitrary correlation function
double correlator(const double omega, const double temperature) {
  if (omega == 0.0) return 1.0;
  double x = (kHBar * abs(omega)) / (kBoltzmann * temperature);
  return 0.5 * x / (exp(x) - 1);
}

double timestep_mismatch_inv_correlator(const double omega, const double bath_time_step) {
  // TODO: check if this should be 1 or 0
  if (omega == 0.0) return 1.0;
  return (0.5 * omega * bath_time_step) / sin(0.5 * omega * bath_time_step);
}

// filter function
double filter(const double omega, const double temperature, const double bath_time_step) {
  auto x = correlator(omega, temperature); // * timestep_mismatch_inv_correlator(omega, bath_time_step);
  assert(!(x < 0.0));
  return sqrt(x);
}

template<typename T1, typename... Args>
vector<T1> discretize_function(std::function<T1(double, Args...)> f, const double delta, const int num_freq, Args... args) {
  vector<T1> result(2 * num_freq - 1);

  for(auto i = 0; i < result.size(); ++i) {
    const auto k = fftw_k_index(i, num_freq);
    result[i] = f(k * delta, args...);
  }

  return result;
}

vector<double> real_discrete_ft(const vector<double> &x) {
  const int num_freq = (x.size() + 1) / 2;
  vector<double> result(x.size());

  for (auto i = 0; i < result.size(); ++i) {
    const auto n = fftw_k_index(i, num_freq);
    double sum = 0.0;
    for (auto j = 0; j < x.size(); ++j) {
      const auto k = fftw_k_index(j, num_freq);
      sum += x[j] * cos(k * n * M_PI / double(num_freq));
    }

    result[i] = sum / double(2 * num_freq);
  }
  return result;
}

}

CudaLangevinArbitraryThermostat::CudaLangevinArbitraryThermostat(const double &temperature, const double &sigma, const int num_spins)
: Thermostat(temperature, sigma, num_spins),
  debug_(false),
  filter_temperature_(0.0)
  {
   cout << "\n  initialising CUDA Langevin arbitrary noise thermostat\n";

    num_freq_ = 10000;
    delta_t_ = solver->time_step();
    max_omega_ = kPi / delta_t_;
//    config->lookupValue("thermostat.w_max", max_omega_);

    delta_omega_ = max_omega_ / double(num_freq_);

//   num_freq_ = 100;
//   max_omega_ = 100.0 * kTHz * kTwoPi;
//   config->lookupValue("thermostat.w_max", max_omega_);
//
//   delta_omega_ = max_omega_ / double(num_freq_);
//   delta_t_ = kPi / max_omega_;

   cout << "    max_omega (THz) " << std::fixed << max_omega_ / (kTwoPi * kTHz) << "\n";
   cout << "    delta_t " << std::scientific << delta_t_ << "\n";
   cout << "    num_freq " << num_freq_ << "\n";

   cout << "    initialising CUDA streams\n";

   if (hipStreamCreate(&dev_stream_) != hipSuccess) {
     jams_die("Failed to create CUDA stream in CudaLangevinArbitraryThermostat");
   }

   if (hipStreamCreate(&dev_curand_stream_) != hipSuccess) {
     jams_die("Failed to create CURAND stream in CudaLangevinArbitraryThermostat");
   }

   cout << "    initialising CURAND\n";

   CHECK_CURAND_STATUS(hiprandSetStream(jams::instance().curand_generator(), dev_curand_stream_));

   for (int i = 0; i < num_spins; ++i) {
     for (int j = 0; j < 3; ++j) {
        sigma_(i,j) = sqrt((2.0 * globals::alpha(i) * kBoltzmann) / (globals::mus(i) * kBohrMagneton * kGyromagneticRatio));
     }
   }

   white_noise_.resize((num_spins * 3) * (2*num_freq_ - 1));

   CHECK_CURAND_STATUS(hiprandGenerateNormalDouble(jams::instance().curand_generator(), white_noise_.device_data(), white_noise_.size(), 0.0, 1.0));


    debug_file_.open("noise.tsv");
  }

void CudaLangevinArbitraryThermostat::update() {

  if (filter_temperature_ != this->temperature()) {
    filter_temperature_ = this->temperature();
    auto discrete_filter = discretize_function(std::function<double(double, double, double)>(filter), delta_omega_, num_freq_, temperature_, delta_t_);
    auto convoluted_filter = real_discrete_ft(discrete_filter);
    filter_.resize(convoluted_filter.size());
    std::copy(convoluted_filter.begin(), convoluted_filter.end(), filter_.begin());
  }
  assert(filter_.size() != 0);

  int block_size = 256;
  int grid_size = (globals::num_spins3 + block_size - 1) / block_size;

  const double temperature = this->temperature();

  CHECK_CURAND_STATUS(hiprandSetStream(jams::instance().curand_generator(),
                                      dev_curand_stream_));

  const auto n = pbc(solver->iteration(), (2 * num_freq_ - 1));
  arbitrary_stochastic_process_cuda_kernel<<<grid_size, block_size, 0, dev_stream_ >>>(
      noise_.device_data(),
      filter_.device_data(),
      white_noise_.device_data(),
      delta_t_,
      n,
      num_freq_,
      globals::num_spins3);
  DEBUG_CHECK_CUDA_ASYNC_STATUS;


  // scale by sigma
  // TODO: does temperature need to go here or in the kernel above?
  cuda_array_elementwise_scale(globals::num_spins, 3, sigma_.device_data(), temperature, noise_.device_data(), 1, noise_.device_data(), 1, dev_stream_);


  debug_file_ << solver->iteration() * delta_t_ << " " << noise_(0, 0)
              << "\n";

  // generate new random numbers
  CHECK_CURAND_STATUS(
      hiprandGenerateNormalDouble(jams::instance().curand_generator(),
                                 white_noise_.device_data() +
                                 globals::num_spins3 *
                                 pbc(solver->iteration() + num_freq_,
                                     2 * num_freq_ - 1),
                                 globals::num_spins3, 0.0, 1.0));
}

CudaLangevinArbitraryThermostat::~CudaLangevinArbitraryThermostat() {
  debug_file_.close();

  if (dev_stream_ != nullptr) {
    hipStreamDestroy(dev_stream_);
  }

  if (dev_curand_stream_ != nullptr) {
    hipStreamDestroy(dev_curand_stream_);
  }
}

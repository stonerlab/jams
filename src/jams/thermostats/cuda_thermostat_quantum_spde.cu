#include "hip/hip_runtime.h"
// Copyright 2014 Joseph Barker. All rights reserved.

#include <cmath>
#include <string>
#include <iomanip>
#include <random>
#include <mutex>

#include <jams/common.h>
#include "jams/helpers/utils.h"
#include "jams/cuda/cuda_array_kernels.h"

#include "jams/thermostats/cuda_thermostat_quantum_spde.h"
#include "jams/thermostats/cuda_thermostat_quantum_spde_kernel.cuh"

#include "jams/core/solver.h"
#include "jams/core/globals.h"
#include "jams/core/lattice.h"
#include "jams/core/solver.h"
#include "jams/cuda/cuda_array_kernels.h"
#include "jams/helpers/consts.h"
#include "jams/helpers/error.h"
#include "jams/helpers/random.h"
#include "jams/helpers/utils.h"
#include "jams/cuda/cuda_common.h"
#include "jams/monitors/magnetisation.h"
#include <jams/helpers/exception.h>

CudaThermostatQuantumSpde::CudaThermostatQuantumSpde(const double &temperature, const double &sigma, const double timestep, const int num_spins)
: Thermostat(temperature, sigma, timestep, num_spins),
  debug_(false)
  {
   std::cout << "\n  initialising quantum-spde-gpu thermostat\n";

   globals::config->lookupValue("thermostat.zero_point", do_zero_point_);

   double t_warmup = 1e-10 / 1e-12; // 0.1 ns
   globals::config->lookupValue("thermostat.warmup_time", t_warmup);

   omega_max_ = 25.0 * kTwoPi;
   globals::config->lookupValue("thermostat.w_max", omega_max_);

   double dt_thermostat = timestep;
   delta_tau_ = (dt_thermostat * kBoltzmannIU) / kHBarIU;

   std::cout << "    omega_max (THz) " << omega_max_ / (kTwoPi) << "\n";
   std::cout << "    hbar*w/kB " << (kHBarIU * omega_max_) / (kBoltzmannIU) << "\n";
   std::cout << "    t_step " << dt_thermostat << "\n";
   std::cout << "    delta tau " << delta_tau_ << "\n";

   std::cout << "    initialising CUDA streams\n";

   if (hipStreamCreate(&dev_stream_) != hipSuccess) {
     throw jams::GeneralException("Failed to create CUDA stream in CudaLangevinBoseThermostat");
   }

   if (hipStreamCreate(&dev_curand_stream_) != hipSuccess) {
     throw jams::GeneralException("Failed to create CURAND stream in CudaLangevinBoseThermostat");
   }

   std::cout << "    initialising CURAND\n";

   CHECK_CURAND_STATUS(hiprandSetStream(jams::instance().curand_generator(), dev_curand_stream_));
   CHECK_CURAND_STATUS(hiprandGenerateNormalDouble(jams::instance().curand_generator(), eta0_.device_data(), eta0_.size(), 0.0, 1.0));
   CHECK_CURAND_STATUS(hiprandGenerateNormalDouble(jams::instance().curand_generator(), eta1a_.device_data(), eta1a_.size(), 0.0, 1.0));
   CHECK_CURAND_STATUS(hiprandGenerateNormalDouble(jams::instance().curand_generator(), eta1b_.device_data(), eta1b_.size(), 0.0, 1.0));

    for (int i = 0; i < num_spins; ++i) {
      for (int j = 0; j < 3; ++j) {
        sigma_(i,j) = (kBoltzmannIU) * sqrt((2.0 * globals::alpha(i))
                                            / (kHBarIU * globals::gyro(i) * globals::mus(i)));
      }
    }

   num_warm_up_steps_ = static_cast<unsigned>(t_warmup / dt_thermostat);


  zero(zeta5_.resize(num_spins * 3));
  zero(zeta5p_.resize(num_spins * 3));
  zero(zeta6_.resize(num_spins * 3));
  zero(zeta6p_.resize(num_spins * 3));
  zero(eta1a_.resize(2 * num_spins * 3));
  zero(eta1b_.resize(2 * num_spins * 3));

  if (do_zero_point_) {
    zero(zeta0_.resize(4 * num_spins * 3));
    zero(eta0_.resize(4 * num_spins * 3));
  }
}

void CudaThermostatQuantumSpde::update() {
  if (!is_warmed_up_) {
    is_warmed_up_ = true;
    warmup(num_warm_up_steps_);
  }

  if (this->temperature() == 0) {
    CHECK_CUDA_STATUS(hipMemset(noise_.device_data(), 0, noise_.elements()*sizeof(double)));
    return;
  }

  int block_size = 96;
  int grid_size = (globals::num_spins3 + block_size - 1) / block_size;

  const double reduced_omega_max = (kHBarIU * omega_max_) / (kBoltzmannIU * this->temperature());
  const double reduced_delta_tau = delta_tau_ * this->temperature();
  const double temperature = this->temperature();

  swap(eta1a_, eta1b_);
  CHECK_CURAND_STATUS(hiprandSetStream(jams::instance().curand_generator(), dev_curand_stream_));
  CHECK_CURAND_STATUS(hiprandGenerateNormalDouble(jams::instance().curand_generator(), eta1a_.device_data(), eta1a_.size(), 0.0, 1.0));

  cuda_thermostat_quantum_spde_no_zero_kernel<<<grid_size, block_size, 0, dev_stream_ >>> (
    noise_.device_data(), zeta5_.device_data(), zeta5p_.device_data(), zeta6_.device_data(), zeta6p_.device_data(),
    eta1b_.device_data(), sigma_.device_data(), reduced_delta_tau, temperature, reduced_omega_max, globals::num_spins3);
  DEBUG_CHECK_CUDA_ASYNC_STATUS;

  if (do_zero_point_) {
    CHECK_CURAND_STATUS(hiprandSetStream(jams::instance().curand_generator(), dev_curand_stream_));
    CHECK_CURAND_STATUS(hiprandGenerateNormalDouble(jams::instance().curand_generator(), eta0_.device_data(), eta0_.size(), 0.0, 1.0));

    cuda_thermostat_quantum_spde_zero_point_kernel <<< grid_size, block_size, 0, dev_stream_ >>> (
        noise_.device_data(), zeta0_.device_data(), eta0_.device_data(), sigma_.device_data(), reduced_delta_tau,
        temperature, reduced_omega_max, globals::num_spins3);
    DEBUG_CHECK_CUDA_ASYNC_STATUS;
  }
}

CudaThermostatQuantumSpde::~CudaThermostatQuantumSpde() {
  if (dev_stream_ != nullptr) {
    hipStreamDestroy(dev_stream_);
  }

  if (dev_curand_stream_ != nullptr) {
    hipStreamDestroy(dev_curand_stream_);
  }
}

void CudaThermostatQuantumSpde::warmup(const unsigned steps) {
  std::cout << "warming up thermostat " << steps << " steps @ " << this->temperature() << "K" << std::endl;

  for (auto i = 0; i < steps; ++i) {
    update();
  }
}

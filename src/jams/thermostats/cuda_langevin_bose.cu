#include "hip/hip_runtime.h"
// Copyright 2014 Joseph Barker. All rights reserved.

#include <cmath>
#include <string>
#include <iomanip>
#include <random>
#include <mutex>

#include "jams/helpers/utils.h"
#include "jams/cuda/cuda_array_kernels.h"

#include "cuda_langevin_bose.h"
#include "cuda_langevin_bose_kernel.h"

#include "jams/core/solver.h"
#include "jams/core/globals.h"
#include "jams/core/lattice.h"
#include "jams/helpers/consts.h"
#include "jams/helpers/random.h"
#include "jams/helpers/error.h"

#include "jams/monitors/magnetisation.h"

using namespace std;

CudaLangevinBoseThermostat::CudaLangevinBoseThermostat(const double &temperature, const double &sigma, const int num_spins)
: Thermostat(temperature, sigma, num_spins),
  debug_(false),
  dev_noise_(3 * num_spins, 0.0),
  dev_zeta0_(4 * num_spins * 3, 0.0),
  dev_zeta5_(num_spins * 3, 0.0),
  dev_zeta5p_(num_spins * 3, 0.0),
  dev_zeta6_(num_spins * 3, 0.0),
  dev_zeta6p_(num_spins * 3, 0.0),
  dev_eta0_(4 * num_spins * 3, 0.0),
  dev_eta1a_(2 * num_spins * 3, 0.0),
  dev_eta1b_(2 * num_spins * 3, 0.0),
  dev_sigma_(num_spins, 0.0)
 {
   cout << "\n  initialising CUDA Langevin semi-quantum noise thermostat\n";

   debug_ = false;

   if(debug_) {
     debug_noise_outfile_.open(seedname + "_qnoise.tsv");
     debug_noise_outfile_ << "time\tnoise0\tnoise1\tnoise2\tnoise3\tnoise4\tnoise5\tnoise6\tnoise7\tnoise8\tnoise9\n";
   }

   config->lookupValue("thermostat.zero_point", do_zero_point_);

   double t_warmup = 1e-10; // 0.1 ns
   config->lookupValue("thermostat.warmup_time", t_warmup);

   omega_max_ = 25.0 * kTwoPi * kTHz;
   config->lookupValue("thermostat.w_max", omega_max_);

   double dt_thermostat = ::config->lookup("solver.t_step");
   delta_tau_ = (dt_thermostat * kBoltzmann) / kHBar;

   uint64_t dev_rng_seed = jams::random_generator()();

   cout << "    seed " << dev_rng_seed << "\n";
   cout << "    omega_max (THz) " << omega_max_ / (kTwoPi * kTHz) << "\n";
   cout << "    hbar*w/kB " << (kHBar * omega_max_) / (kBoltzmann) << "\n";
   cout << "    t_step " << dt_thermostat << "\n";
   cout << "    delta tau " << delta_tau_ << "\n";

   cout << "    initialising CUDA streams\n";

   if (hipStreamCreate(&dev_stream_) != hipSuccess) {
     die("Failed to create CUDA stream in CudaLangevinBoseThermostat");
   }

   if (hipStreamCreate(&dev_curand_stream_) != hipSuccess) {
     die("Failed to create CURAND stream in CudaLangevinBoseThermostat");
   }

   cout << "    initialising CURAND\n";

   // initialize and seed the CURAND generator on the device
   if (hiprandCreateGenerator(&dev_rng_, HIPRAND_RNG_PSEUDO_DEFAULT) != HIPRAND_STATUS_SUCCESS) {
     die("Failed to create CURAND generator in CudaLangevinBoseThermostat");
   }

   // initialize zeta and eta with random variables
   hiprandSetStream(dev_rng_, dev_curand_stream_);

   cout << "    seeding CURAND " << dev_rng_seed << "\n";

   if (hiprandSetPseudoRandomGeneratorSeed(dev_rng_, dev_rng_seed) != HIPRAND_STATUS_SUCCESS) {
     die("Failed to set CURAND seed in CudaLangevinBoseThermostat");
   }

   if (hiprandGenerateSeeds(dev_rng_) != HIPRAND_STATUS_SUCCESS) {
     die("Failed to generate CURAND seeds in CudaLangevinBoseThermostat");
   }

   cout << "    allocating GPU memory\n";

   if (hiprandGenerateNormalDouble(dev_rng_, dev_eta0_.data(), dev_eta0_.size(), 0.0, 1.0)
       != HIPRAND_STATUS_SUCCESS) {
     die("hiprandGenerateNormalDouble failure in CudaLangevinBoseThermostat::constructor");
   }

   if (hiprandGenerateNormalDouble(dev_rng_, dev_eta1a_.data(), dev_eta1a_.size(), 0.0, 1.0)
       != HIPRAND_STATUS_SUCCESS) {
     die("hiprandGenerateNormalDouble failure in CudaLangevinBoseThermostat::constructor");
   }

   if (hiprandGenerateNormalDouble(dev_rng_, dev_eta1b_.data(), dev_eta1b_.size(), 0.0, 1.0)
       != HIPRAND_STATUS_SUCCESS) {
     die("hiprandGenerateNormalDouble failure in CudaLangevinBoseThermostat::constructor");
   }

   jblib::Array<double, 2> scale(num_spins, 3);
   for (int i = 0; i < num_spins; ++i) {
     for (int j = 0; j < 3; ++j) {
       scale(i, j) = (kBoltzmann) *
                     sqrt((2.0 * globals::alpha(i) * globals::mus(i)) / (kHBar * kGyromagneticRatio * kBohrMagneton));
     }
   }

   dev_sigma_ = jblib::CudaArray<double, 1>(scale);

   num_warm_up_steps_ = static_cast<unsigned>(t_warmup / dt_thermostat);
 }

void CudaLangevinBoseThermostat::update() {
  if (!is_warmed_up_) {
    is_warmed_up_ = true;
    warmup(num_warm_up_steps_);
  }

  int block_size = 96;
  int grid_size = (globals::num_spins3 + block_size - 1) / block_size;

  swap(dev_eta1a_, dev_eta1b_);
  hiprandGenerateNormalDouble(dev_rng_, dev_eta1a_.data(), dev_eta1a_.size(), 0.0, 1.0);

  bose_coth_stochastic_process_cuda_kernel<<<grid_size, block_size, 0, dev_stream_ >>> (
    dev_noise_.data(),
    dev_zeta5_.data(),
    dev_zeta5p_.data(),
    dev_zeta6_.data(),
    dev_zeta6p_.data(),
    dev_eta1b_.data(),
    dev_sigma_.data(),
    delta_tau_ * this->temperature(),
    this->temperature(),
    (kHBar * omega_max_) / (kBoltzmann * this->temperature()),  // w_m
    globals::num_spins3);

  if (do_zero_point_) {
    hiprandGenerateNormalDouble(dev_rng_, dev_eta0_.data(), dev_eta0_.size(), 0.0, 1.0);

    bose_zero_point_stochastic_process_cuda_kernel << < grid_size, block_size, 0, dev_stream_ >> > (
            dev_noise_.data(),
                    dev_zeta0_.data(),
                    dev_eta0_.data(),
                    dev_sigma_.data(),
                    delta_tau_ * this->temperature(),
                    this->temperature(),
                    (kHBar * omega_max_) / (kBoltzmann * this->temperature()),  // w_m
                    globals::num_spins3);
  }

  if (debug_ && is_warmed_up_) {
    dev_noise_.copy_to_host_array(noise_);
    debug_noise_outfile_ << solver->time() << "\t";
    for (auto i = 0; i < 10; ++i) {
      debug_noise_outfile_ << noise_[i] << "\t";
    }
    debug_noise_outfile_ << std::endl;
  }
}

CudaLangevinBoseThermostat::~CudaLangevinBoseThermostat() {
  if (dev_rng_ != nullptr) {
    hiprandDestroyGenerator(dev_rng_);
  }

  if (dev_stream_ != nullptr) {
    hipStreamDestroy(dev_stream_);
  }

  if (dev_curand_stream_ != nullptr) {
    hipStreamDestroy(dev_curand_stream_);
  }
}

void CudaLangevinBoseThermostat::warmup(const unsigned steps) {
  cout << "warming up thermostat " << steps << " steps @ " << this->temperature() << "K" << std::endl;

  for (auto i = 0; i < steps; ++i) {
    update();
  }
}

#include "hip/hip_runtime.h"
// Copyright 2014 Joseph Barker. All rights reserved.

#include <cmath>
#include <string>
#include <iomanip>
#include <random>
#include <mutex>

#include "jams/helpers/utils.h"
#include "jams/cuda/cuda_array_kernels.h"

#include "cuda_langevin_bose.h"
#include "cuda_langevin_bose_kernel.h"

#include "jams/core/globals.h"
#include "jams/core/lattice.h"
#include "jams/helpers/consts.h"
#include "jams/helpers/random.h"
#include "jams/helpers/error.h"

#include "jams/monitors/magnetisation.h"

using namespace std;

CudaLangevinBoseThermostat::CudaLangevinBoseThermostat(const double &temperature, const double &sigma, const int num_spins)
: Thermostat(temperature, sigma, num_spins),
  debug_(false),
  dev_noise_(3 * num_spins, 0.0),
  dev_zeta5_(num_spins * 3, 0.0),
  dev_zeta5p_(num_spins * 3, 0.0),
  dev_zeta6_(num_spins * 3, 0.0),
  dev_zeta6p_(num_spins * 3, 0.0),
  dev_eta0_(4 * num_spins * 3, 0.0),
  dev_eta1a_(2 * num_spins * 3, 0.0),
  dev_eta1b_(2 * num_spins * 3, 0.0),
  dev_sigma_(num_spins, 0.0)
 {
   cout << "\n  initialising CUDA Langevin semi-quantum noise thermostat\n";

   debug_ = false;

   if (debug_) {
     cout << "    DEBUG ON\n";
     std::string name = seedname + "_noise.dat";
     outfile_.open(name.c_str());
   }

   double t_warmup = 1e-10; // 0.1 ns
   config->lookupValue("thermostat.warmup_time", t_warmup);

   omega_max_ = 100 * kTHz;
   config->lookupValue("thermostat.w_max", omega_max_);

   double dt_thermostat = ::config->lookup("solver.t_step");
   delta_tau_ = (dt_thermostat * kBoltzmann) / kHBar;

   std::random_device rdev;
   uint64_t dev_rng_seed = concatenate_32_bit(rdev(), rdev());

   unsigned long long cfg_seed = 0;
   config->lookupValue("thermostat.seed", cfg_seed);

   if (cfg_seed != 0) {
     dev_rng_seed = cfg_seed;
   }

   // check the seed populates msw and lsw of the 64bit number
   if (dev_rng_seed < std::numeric_limits<uint32_t>::max()) {
     jams_warning("Random seed does not fill 64 bits. Try making the seed larger");
   }

   cout << "    seed " << dev_rng_seed << "\n";
   cout << "    omega_max (THz) " << omega_max_ / kTHz << "\n";
   cout << "    hbar*w/kB " << (kHBar * omega_max_) / (kBoltzmann) << "\n";
   cout << "    t_step " << dt_thermostat << "\n";
   cout << "    delta tau " << delta_tau_ << "\n";

   cout << "    initialising CUDA streams\n";

   if (hipStreamCreate(&dev_stream_) != hipSuccess) {
     die("Failed to create CUDA stream in CudaLangevinBoseThermostat");
   }

   if (hipStreamCreate(&dev_curand_stream_) != hipSuccess) {
     die("Failed to create CURAND stream in CudaLangevinBoseThermostat");
   }

   cout << "    initialising CURAND\n";

   // initialize and seed the CURAND generator on the device
   if (hiprandCreateGenerator(&dev_rng_, HIPRAND_RNG_PSEUDO_DEFAULT) != HIPRAND_STATUS_SUCCESS) {
     die("Failed to create CURAND generator in CudaLangevinBoseThermostat");
   }

   // initialize zeta and eta with random variables
   hiprandSetStream(dev_rng_, dev_curand_stream_);

   cout << "    seeding CURAND " << dev_rng_seed << "\n";

   if (hiprandSetPseudoRandomGeneratorSeed(dev_rng_, dev_rng_seed) != HIPRAND_STATUS_SUCCESS) {
     die("Failed to set CURAND seed in CudaLangevinBoseThermostat");
   }

   if (hiprandGenerateSeeds(dev_rng_) != HIPRAND_STATUS_SUCCESS) {
     die("Failed to generate CURAND seeds in CudaLangevinBoseThermostat");
   }

   cout << "    allocating GPU memory\n";

   if (hiprandGenerateNormalDouble(dev_rng_, dev_eta0_.data(), dev_eta0_.size(), 0.0, 1.0)
       != HIPRAND_STATUS_SUCCESS) {
     die("hiprandGenerateNormalDouble failure in CudaLangevinBoseThermostat::constructor");
   }

   if (hiprandGenerateNormalDouble(dev_rng_, dev_eta1a_.data(), dev_eta1a_.size(), 0.0, 1.0)
       != HIPRAND_STATUS_SUCCESS) {
     die("hiprandGenerateNormalDouble failure in CudaLangevinBoseThermostat::constructor");
   }

   if (hiprandGenerateNormalDouble(dev_rng_, dev_eta1b_.data(), dev_eta1b_.size(), 0.0, 1.0)
       != HIPRAND_STATUS_SUCCESS) {
     die("hiprandGenerateNormalDouble failure in CudaLangevinBoseThermostat::constructor");
   }

   jblib::Array<double, 2> scale(num_spins, 3);
   for (int i = 0; i < num_spins; ++i) {
     for (int j = 0; j < 3; ++j) {
       scale(i, j) = (kBoltzmann) *
                     sqrt((2.0 * globals::alpha(i) * globals::mus(i)) / (kHBar * kGyromagneticRatio * kBohrMagneton));
     }
   }

   dev_sigma_ = jblib::CudaArray<double, 1>(scale);

   num_warm_up_steps_ = static_cast<unsigned>(t_warmup / dt_thermostat);
 }

void CudaLangevinBoseThermostat::update() {
  if (!is_warmed_up_) {
    is_warmed_up_ = true;
    warmup(num_warm_up_steps_);
  }

  int block_size = 96;
  int grid_size = (globals::num_spins3 + block_size - 1) / block_size;

  swap(dev_eta1a_, dev_eta1b_);

  hiprandGenerateNormalDouble(dev_rng_, dev_eta1a_.data(), dev_eta1a_.size(), 0.0, 1.0);

  bose_coth_stochastic_process_cuda_kernel<<<grid_size, block_size, 0, dev_stream_ >>> (
    dev_noise_.data(),
    dev_zeta5_.data(),
    dev_zeta5p_.data(),
    dev_zeta6_.data(),
    dev_zeta6p_.data(),
    dev_eta1b_.data(),
    dev_sigma_.data(),
    delta_tau_ * this->temperature(),
    this->temperature(),
    (kHBar * omega_max_) / (kBoltzmann * this->temperature()),  // w_m
    globals::num_spins3);
}

CudaLangevinBoseThermostat::~CudaLangevinBoseThermostat() {
  if (dev_rng_ != nullptr) {
    hiprandDestroyGenerator(dev_rng_);
  }

  if (dev_stream_ != nullptr) {
    hipStreamDestroy(dev_stream_);
  }

  if (dev_curand_stream_ != nullptr) {
    hipStreamDestroy(dev_curand_stream_);
  }
  
  if (debug_) {
    outfile_.close();
  }
}

void CudaLangevinBoseThermostat::warmup(const unsigned steps) {
  cout << "warming up thermostat " << steps << " steps @ " << this->temperature() << "K" << std::endl;

  for (auto i = 0; i < steps; ++i) {
    update();
  }
}

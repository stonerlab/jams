#include "hip/hip_runtime.h"
// Copyright 2014 Joseph Barker. All rights reserved.

#define __STDC_FORMAT_MACROS
#include <inttypes.h>

#include <cmath>
#include <string>
#include <iomanip>
#include "jams/core/cuda_array_kernels.h"

#include "jams/thermostats/cuda_langevin_bose.h"
#include "jams/thermostats/cuda_langevin_bose_kernel.h"

#include "jams/core/globals.h"
#include "jams/core/lattice.h"
#include "jams/core/consts.h"
#include "jams/core/output.h"
#include "jams/core/rand.h"
#include "jams/core/error.h"

#include "jams/monitors/magnetisation.h"

CudaLangevinBoseThermostat::CudaLangevinBoseThermostat(const double &temperature, const double &sigma, const int num_spins)
: Thermostat(temperature, sigma, num_spins),
  debug_(false),
  dev_noise_(3 * num_spins, 0.0),
  dev_zeta5_(num_spins * 3, 0.0),
  dev_zeta5p_(num_spins * 3, 0.0),
  dev_zeta6_(num_spins * 3, 0.0),
  dev_zeta6p_(num_spins * 3, 0.0),
  dev_eta0_(4 * num_spins * 3, 0.0),
  dev_eta1a_(2 * num_spins * 3, 0.0),
  dev_eta1b_(2 * num_spins * 3, 0.0),
  dev_sigma_(num_spins, 0.0)
 {
  ::output->write("\n  initialising CUDA Langevin semi-quantum noise thermostat\n");

  debug_ = false;

  if (debug_) {
    ::output->write("    DEBUG ON\n");
    std::string name = seedname + "_noise.dat";
    outfile_.open(name.c_str());
  }

  w_max_ = 100*kTHz;

  config->lookupValue("sim.w_max", w_max_);

  const double dt = ::config->lookup("sim.t_step");
  tau_ = (dt * kBoltzmann) / kHBar;

  ::output->write("    omega_max = %6.6f (THz)\n", w_max_ / kTHz);
  ::output->write("    hbar*w/kB = %4.4e\n", (kHBar * w_max_) / (kBoltzmann));
  ::output->write("    delta tau = %4.4e * T\n", tau_);

  ::output->write("    initialising CUDA streams\n");

  if (hipStreamCreate(&dev_stream_) != hipSuccess){
    jams_error("Failed to create CUDA stream in CudaLangevinBoseThermostat");
  }

  if (hipStreamCreate(&dev_curand_stream_) != hipSuccess){
    jams_error("Failed to create CURAND stream in CudaLangevinBoseThermostat");
  }

  ::output->write("    initialising CURAND\n");

  // initialize and seed the CURAND generator on the device
  if (hiprandCreateGenerator(&dev_rng_, HIPRAND_RNG_PSEUDO_DEFAULT) != HIPRAND_STATUS_SUCCESS) {
    jams_error("Failed to create CURAND generator in CudaLangevinBoseThermostat");
  }

  // initialize zeta and eta with random variables
  hiprandSetStream(dev_rng_, dev_curand_stream_);

  const uint64_t dev_rng_seed = rng->uniform()*18446744073709551615ULL;
  ::output->write("    seeding CURAND (%" PRIu64 ")\n", dev_rng_seed);

  if (hiprandSetPseudoRandomGeneratorSeed(dev_rng_, dev_rng_seed) != HIPRAND_STATUS_SUCCESS) {
    jams_error("Failed to set CURAND seed in CudaLangevinBoseThermostat");
  }

  if (hiprandGenerateSeeds(dev_rng_) != HIPRAND_STATUS_SUCCESS) {
    jams_error("Failed to generate CURAND seeds in CudaLangevinBoseThermostat");
  }

  ::output->write("    allocating GPU memory\n");

  if (hiprandGenerateNormalDouble(dev_rng_, dev_eta0_.data(), dev_eta0_.size(), 0.0, 1.0)
       != HIPRAND_STATUS_SUCCESS) {
    jams_error("hiprandGenerateNormalDouble failure in CudaLangevinBoseThermostat::constructor");
  }

  if (hiprandGenerateNormalDouble(dev_rng_, dev_eta1a_.data(), dev_eta1a_.size(), 0.0, 1.0)
       != HIPRAND_STATUS_SUCCESS) {
    jams_error("hiprandGenerateNormalDouble failure in CudaLangevinBoseThermostat::constructor");
  }

  if (hiprandGenerateNormalDouble(dev_rng_, dev_eta1b_.data(), dev_eta1b_.size(), 0.0, 1.0)
       != HIPRAND_STATUS_SUCCESS) {
    jams_error("hiprandGenerateNormalDouble failure in CudaLangevinBoseThermostat::constructor");
  }

  jblib::Array<double, 2> scale(num_spins, 3);
  for(int i = 0; i < num_spins; ++i) {
    for(int j = 0; j < 3; ++j) {
      scale(i, j) = (kBoltzmann) * sqrt( (2.0 * globals::alpha(i) * globals::mus(i)) / ( kHBar * kGyromagneticRatio * kBohrMagneton) );
    }
  }

  dev_sigma_ = jblib::CudaArray<double, 1>(scale);

  // const int num_warmup_steps = 10;
  const int num_warmup_steps = 1000000;

  ::output->write("    warming up thermostat (%8.2f ns @ %8.2f K)\n", ((dt *num_warmup_steps) / 1.0e-9), this->temperature());

  for (int i = 0; i < num_warmup_steps; ++i) {
    update();
  }
}

void CudaLangevinBoseThermostat::update() {
  int block_size = 96;
  int grid_size = (globals::num_spins3 + block_size - 1) / block_size;

  swap(dev_eta1a_, dev_eta1b_);

  hiprandGenerateNormalDouble(dev_rng_, dev_eta1a_.data(), dev_eta1a_.size(), 0.0, 1.0);

  bose_coth_stochastic_process_cuda_kernel<<<grid_size, block_size, 0, dev_stream_ >>> (
    dev_noise_.data(),
    dev_zeta5_.data(),
    dev_zeta5p_.data(),
    dev_zeta6_.data(),
    dev_zeta6p_.data(),
    dev_eta1b_.data(),
    dev_sigma_.data(),
    tau_ * this->temperature(),
    this->temperature(),
    (kHBar * w_max_) / (kBoltzmann * this->temperature()),  // w_m
    globals::num_spins3);
}

CudaLangevinBoseThermostat::~CudaLangevinBoseThermostat() {
  hiprandDestroyGenerator(dev_rng_);
  hipStreamDestroy(dev_stream_);
  hipStreamDestroy(dev_curand_stream_);
  if (debug_) {
    outfile_.close();
  }
}

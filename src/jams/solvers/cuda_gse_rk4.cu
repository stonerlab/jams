// cuda_gse_rk4.cu                                                     -*-C++-*-

#include <jams/solvers/cuda_gse_rk4.h>

#include <hip/hip_runtime.h>

#include <algorithm>
#include <cmath>

#include <jams/common.h>
#include <jams/interface/config.h>
#include <jams/helpers/defaults.h>

#include "jams/helpers/consts.h"
#include "jams/core/globals.h"
#include "jams/core/thermostat.h"
#include "jams/core/physics.h"
#include "jams/helpers/error.h"
#include "jams/cuda/cuda_common.h"

#include "cuda_gse_rk4_kernel.cuh"

void CUDAGSERK4Solver::initialize(const libconfig::Setting& settings)
{
  // convert input in seconds to picoseconds for internal units
  step_size_ = jams::config_required<double>(settings, "t_step") / 1e-12;
  auto t_max = jams::config_required<double>(settings, "t_max") / 1e-12;
  auto t_min = jams::config_optional<double>(settings, "t_min", 0.0) / 1e-12;


  max_steps_ = static_cast<int>(t_max / step_size_);
  min_steps_ = static_cast<int>(t_min / step_size_);

  std::cout << "\ntimestep (ps) " << step_size_ << "\n";
  std::cout << "\nt_max (ps) " << t_max << " steps " << max_steps_ << "\n";
  std::cout << "\nt_min (ps) " << t_min << " steps " << min_steps_ << "\n";

  std::cout << "timestep " << step_size_ << "\n";
  std::cout << "t_max " << t_max << " steps (" <<  max_steps_ << ")\n";
  std::cout << "t_min " << t_min << " steps (" << min_steps_ << ")\n";

  std::string thermostat_name = jams::config_optional<std::string>(globals::config->lookup("solver"), "thermostat", jams::defaults::solver_gpu_thermostat);
  register_thermostat(Thermostat::create(thermostat_name, this->time_step()));

  std::cout << "  thermostat " << thermostat_name.c_str() << "\n";

  std::cout << "done\n";

  s_old_.resize(globals::num_spins, 3);
  for (auto i = 0; i < globals::num_spins; ++i) {
    for (auto j = 0; j < 3; ++j) {
      s_old_(i, j) = globals::s(i, j);
    }
  }

  k1_.resize(globals::num_spins, 3);
  k2_.resize(globals::num_spins, 3);
  k3_.resize(globals::num_spins, 3);
  k4_.resize(globals::num_spins, 3);

}

void CUDAGSERK4Solver::run()
{
  double t0 = time_;

  const dim3 block_size = {64, 1, 1};
  auto grid_size = cuda_grid_size(block_size, {static_cast<unsigned int>(globals::num_spins), 1, 1});

  hipMemcpyAsync(s_old_.device_data(),           // void *               dst
                  globals::s.device_data(),               // const void *         src
                  globals::num_spins3*sizeof(double),   // size_t               count
                  hipMemcpyDeviceToDevice,    // enum hipMemcpyKind  kind
                  dev_stream_.get());                   // device stream

  DEBUG_CHECK_CUDA_ASYNC_STATUS

  update_thermostat();

  compute_fields();

  // k1
  cuda_gse_rk4_kernel<<<grid_size, block_size>>>
      (globals::s.device_data(), k1_.device_data(),
       globals::h.device_data(), thermostat_->device_data(),
       globals::gyro.device_data(), globals::mus.device_data(),
       globals::alpha.device_data(), globals::num_spins);
  DEBUG_CHECK_CUDA_ASYNC_STATUS

  double mid_time_step = 0.5 * step_size_;
  time_ = t0 + mid_time_step;

  CHECK_CUBLAS_STATUS(hipblasDcopy(jams::instance().cublas_handle(), globals::num_spins3, s_old_.device_data(), 1, globals::s.device_data(), 1));
  CHECK_CUBLAS_STATUS(hipblasDaxpy(jams::instance().cublas_handle(), globals::num_spins3, &mid_time_step, k1_.device_data(), 1, globals::s.device_data(), 1));

  compute_fields();

  // k2
  cuda_gse_rk4_kernel<<<grid_size, block_size>>>
      (globals::s.device_data(), k2_.device_data(),
       globals::h.device_data(), thermostat_->device_data(),
       globals::gyro.device_data(), globals::mus.device_data(),
       globals::alpha.device_data(), globals::num_spins);
  DEBUG_CHECK_CUDA_ASYNC_STATUS

      mid_time_step = 0.5 * step_size_;
  time_ = t0 + mid_time_step;

  CHECK_CUBLAS_STATUS(hipblasDcopy(jams::instance().cublas_handle(), globals::num_spins3, s_old_.device_data(), 1, globals::s.device_data(), 1));
  CHECK_CUBLAS_STATUS(hipblasDaxpy(jams::instance().cublas_handle(), globals::num_spins3, &mid_time_step, k2_.device_data(), 1, globals::s.device_data(), 1));

  compute_fields();

  // k3
  cuda_gse_rk4_kernel<<<grid_size, block_size>>>
      (globals::s.device_data(), k3_.device_data(),
       globals::h.device_data(), thermostat_->device_data(),
       globals::gyro.device_data(), globals::mus.device_data(),
       globals::alpha.device_data(), globals::num_spins);
  DEBUG_CHECK_CUDA_ASYNC_STATUS

      mid_time_step = step_size_;
  time_ = t0 + mid_time_step;

  CHECK_CUBLAS_STATUS(hipblasDcopy(jams::instance().cublas_handle(), globals::num_spins3, s_old_.device_data(), 1, globals::s.device_data(), 1));
  CHECK_CUBLAS_STATUS(hipblasDaxpy(jams::instance().cublas_handle(), globals::num_spins3, &mid_time_step, k3_.device_data(), 1, globals::s.device_data(), 1));

  compute_fields();

  // k4
  cuda_gse_rk4_kernel<<<grid_size, block_size>>>
      (globals::s.device_data(), k4_.device_data(),
       globals::h.device_data(), thermostat_->device_data(),
       globals::gyro.device_data(), globals::mus.device_data(),
       globals::alpha.device_data(), globals::num_spins);
  DEBUG_CHECK_CUDA_ASYNC_STATUS

      cuda_gse_rk4_combination_kernel<<<grid_size, block_size>>>
      (globals::s.device_data(), s_old_.device_data(),
       k1_.device_data(), k2_.device_data(), k3_.device_data(), k4_.device_data(),
       step_size_, globals::num_spins);

  iteration_++;
  time_ = iteration_ * step_size_;
}


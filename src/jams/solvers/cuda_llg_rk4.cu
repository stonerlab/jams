// cuda_ll_lorentzian_rk4.cu                                                          -*-C++-*-
// Copyright 2014 Joseph Barker. All rights reserved.

#include "jams/solvers/cuda_llg_rk4.h"

#include <hip/hip_runtime.h>

#include "jams/core/globals.h"
#include "jams/core/thermostat.h"

#include "jams/cuda/cuda_common.h"

#include "cuda_llg_rk4_kernel.cuh"
#include <jams/cuda/cuda_spin_ops.h>


void CUDALLGRK4Solver::function_kernel(jams::MultiArray<double, 2>& spins, jams::MultiArray<double, 2>& k) {
  compute_fields();

  const dim3 block_size = {64, 1, 1};
  auto grid_size = cuda_grid_size(block_size, {static_cast<unsigned int>(globals::num_spins), 1, 1});

  // using default stream blocks all streams until complete to force synchronisation
  cuda_llg_rk4_kernel<<<grid_size, block_size>>>
      (spins.device_data(), k.device_data(),
       globals::h.device_data(), thermostat_->device_data(),
       globals::gyro.device_data(), globals::mus.device_data(),
       globals::alpha.device_data(), globals::num_spins);
  DEBUG_CHECK_CUDA_ASYNC_STATUS
}


void CUDALLGRK4Solver::post_step(jams::MultiArray<double, 2> &spins) {
  jams::normalise_spins_cuda(spins);
}

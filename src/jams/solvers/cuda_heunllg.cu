// Copyright 2014 Joseph Barker. All rights reserved.

#include "jams/solvers/cuda_heunllg.h"

#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hipsparse.h>

#include <algorithm>
#include <cmath>
#include <jams/core/config.h>
#include <jams/core/defaults.h>

#include "jams/core/consts.h"
#include "jams/core/exception.h"
#include "jams/core/cuda_sparsematrix.h"
#include "jams/core/globals.h"
#include "jams/core/thermostat.h"
#include "jams/core/output.h"

#include "jams/solvers/cuda_heunllg_kernel.h"

#include "jblib/containers/array.h"

void CUDAHeunLLGSolver::initialize(const libconfig::Setting& settings)
{
  using namespace globals;

  CudaSolver::initialize(settings);

  time_step_ = jams::config_required<double>(settings, "t_step");
  double dt = time_step_ * kGyromagneticRatio;

  auto t_max = jams::config_required<double>(settings, "t_max");
  auto t_min = jams::config_optional<double>(settings, "t_min", 0.0);

  max_steps_ = static_cast<int>(t_max / time_step_);
  min_steps_ = static_cast<int>(t_min / time_step_);

  output->write("\ntimestep\n  %1.8e\n", dt);
  output->write("\nt_max\n  %1.8e (%lu steps)\n", t_max, max_steps_);
  output->write("\nt_min\n  %1.8e (%lu steps)\n", t_min, min_steps_);

  ::output->write("\ninitializing CUDA Heun LLG solver\n");

  ::output->write("  creating stream\n");
  if(hipStreamCreate(&dev_stream_) != hipSuccess) {
    throw cuda_api_exception("", __FILE__, __LINE__, __PRETTY_FUNCTION__);
  }

  ::output->write("  copy time_step to symbol\n");
  if(hipMemcpyToSymbol(HIP_SYMBOL(dev_dt), &time_step_, sizeof(double)) != hipSuccess) {
    throw cuda_api_exception("", __FILE__, __LINE__, __PRETTY_FUNCTION__);
  }

  ::output->write("  copy num_spins to symbol\n");
  if(hipMemcpyToSymbol(HIP_SYMBOL(dev_num_spins), &globals::num_spins, sizeof(unsigned int)) != hipSuccess) {
    throw cuda_api_exception("", __FILE__, __LINE__, __PRETTY_FUNCTION__);
  }

  std::string thermostat_name = jams::config_optional<string>(config->lookup("sim"), "thermostat", jams::default_solver_gpu_thermostat);
  thermostat_ = Thermostat::create(thermostat_name);

  ::output->write("  thermostat: %s\n", thermostat_name.c_str());

  nblocks = (num_spins+BLOCKSIZE-1)/BLOCKSIZE;

  ::output->write("done\n");
}

void CUDAHeunLLGSolver::run()
{
  using namespace globals;

  dim3 block_size;
  block_size.x = 85;
  block_size.y = 3;

  dim3 grid_size;
  grid_size.x = (globals::num_spins + block_size.x - 1) / block_size.x;
  grid_size.y = (3 + block_size.y - 1) / block_size.y;

    hipMemcpyAsync(dev_s_old_.data(),           // void *               dst
               dev_s_.data(),               // const void *         src
               num_spins3*sizeof(double),   // size_t               count
               hipMemcpyDeviceToDevice,    // enum hipMemcpyKind  kind
               dev_stream_);                   // device stream

#ifdef DEBUG
  if (hipPeekAtLastError() != hipSuccess) {
    throw cuda_api_exception("", __FILE__, __LINE__, __PRETTY_FUNCTION__);
  }
#endif



    thermostat_->set_temperature(physics_module_->temperature());
    thermostat_->update();

    compute_fields();

    cuda_heun_llg_kernelA<<<grid_size, block_size>>>
        (dev_s_.data(), dev_ds_dt_.data(), dev_s_old_.data(),
          dev_h_.data(), thermostat_->noise(),
          dev_gyro_.data(), dev_alpha_.data());

#ifdef DEBUG
    if (hipPeekAtLastError() != hipSuccess) {
      throw cuda_api_exception("", __FILE__, __LINE__, __PRETTY_FUNCTION__);
    }
#endif

    compute_fields();

    cuda_heun_llg_kernelB<<<grid_size, block_size>>>
      (dev_s_.data(), dev_ds_dt_.data(), dev_s_old_.data(),
        dev_h_.data(), thermostat_->noise(),
        dev_gyro_.data(), dev_alpha_.data());

#ifdef DEBUG
    if (hipPeekAtLastError() != hipSuccess) {
      throw cuda_api_exception("", __FILE__, __LINE__, __PRETTY_FUNCTION__);
    }
#endif


    iteration_++;
}


CUDAHeunLLGSolver::~CUDAHeunLLGSolver()
{
  if (dev_stream_ != nullptr) {
    hipStreamDestroy(dev_stream_);
  }
}


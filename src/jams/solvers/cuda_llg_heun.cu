// Copyright 2014 Joseph Barker. All rights reserved.

#include "cuda_llg_heun.h"

#include <hip/hip_runtime.h>

#include <algorithm>
#include <cmath>
#include <jams/interface/config.h>
#include <jams/helpers/defaults.h>

#include "jams/helpers/consts.h"
#include "jams/core/globals.h"
#include "jams/core/thermostat.h"
#include "jams/core/physics.h"
#include "jams/helpers/error.h"
#include "jams/cuda/cuda_common.h"

#include "cuda_llg_heun_kernel.cuh"

#include "jblib/containers/array.h"

using namespace std;

void CUDAHeunLLGSolver::initialize(const libconfig::Setting& settings)
{
  using namespace globals;

  CudaSolver::initialize(settings);

  time_step_ = jams::config_required<double>(settings, "t_step");
  double dt = time_step_ * kGyromagneticRatio;

  auto t_max = jams::config_required<double>(settings, "t_max");
  auto t_min = jams::config_optional<double>(settings, "t_min", 0.0);

  max_steps_ = static_cast<int>(t_max / time_step_);
  min_steps_ = static_cast<int>(t_min / time_step_);

  cout << "timestep " << time_step_ << "\n";
  cout << "t_max " << t_max << " steps (" <<  max_steps_ << ")\n";
  cout << "t_min " << t_min << " steps (" << min_steps_ << ")\n";

  cout << "  copy time_step to symbol\n";
  CHECK_CUDA_STATUS(hipMemcpyToSymbol(HIP_SYMBOL(dev_dt), &dt, sizeof(double)));

  cout << "  copy num_spins to symbol\n";
  CHECK_CUDA_STATUS(hipMemcpyToSymbol(HIP_SYMBOL(dev_num_spins), &globals::num_spins, sizeof(unsigned int)));

  std::string thermostat_name = jams::config_optional<string>(config->lookup("solver"), "thermostat", jams::default_solver_gpu_thermostat);
  thermostat_ = Thermostat::create(thermostat_name);

  cout << "  thermostat " << thermostat_name.c_str() << "\n";

  cout << "done\n";

  // check if we need to use zero safe versions of the kernels (for |S| = 0)
  zero_safe_kernels_required_ = false;
  for (auto i = 0; i < globals::num_spins; ++i) {
    if (globals::s(i, 0) == 0.0 && globals::s(i, 1) == 0.0 && globals::s(i, 2) == 0.0) {
      zero_safe_kernels_required_ = true;
      break;
    }
  }

  if (zero_safe_kernels_required_) {
    jams_warning("Some spins have zero length so zero safe kernels will be used.");
  }
}

void CUDAHeunLLGSolver::run()
{
  using namespace globals;

  const dim3 block_size = {84, 3, 1};
  auto grid_size = cuda_grid_size(block_size, {globals::num_spins, 3, 1});

  hipMemcpyAsync(dev_s_old_.data(),           // void *               dst
             dev_s_.data(),               // const void *         src
             num_spins3*sizeof(double),   // size_t               count
             hipMemcpyDeviceToDevice,    // enum hipMemcpyKind  kind
             dev_stream_.get());                   // device stream

  DEBUG_CHECK_CUDA_ASYNC_STATUS

  thermostat_->set_temperature(physics_module_->temperature());
  thermostat_->update();

  compute_fields();

  if (zero_safe_kernels_required_) {
    cuda_zero_safe_heun_llg_kernelA<<<grid_size, block_size>>>
      (dev_s_.data(), dev_ds_dt_.data(), dev_s_old_.data(),
       dev_h_.data(), thermostat_->noise(),
       dev_gyro_.data(), dev_alpha_.data());
    DEBUG_CHECK_CUDA_ASYNC_STATUS
  } else {
    cuda_heun_llg_kernelA<<<grid_size, block_size>>>
      (dev_s_.data(), dev_ds_dt_.data(), dev_s_old_.data(),
       dev_h_.data(), thermostat_->noise(),
       dev_gyro_.data(), dev_alpha_.data());
    DEBUG_CHECK_CUDA_ASYNC_STATUS
  }

  compute_fields();

  if (zero_safe_kernels_required_) {
    cuda_zero_safe_heun_llg_kernelB<<<grid_size, block_size>>>
      (dev_s_.data(), dev_ds_dt_.data(), dev_s_old_.data(),
       dev_h_.data(), thermostat_->noise(),
       dev_gyro_.data(), dev_alpha_.data());
    DEBUG_CHECK_CUDA_ASYNC_STATUS
  } else {
    cuda_heun_llg_kernelB<<<grid_size, block_size>>>
      (dev_s_.data(), dev_ds_dt_.data(), dev_s_old_.data(),
       dev_h_.data(), thermostat_->noise(),
       dev_gyro_.data(), dev_alpha_.data());
    DEBUG_CHECK_CUDA_ASYNC_STATUS
  }

  iteration_++;
}


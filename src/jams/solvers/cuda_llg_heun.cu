// Copyright 2014 Joseph Barker. All rights reserved.

#include "cuda_llg_heun.h"

#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hipsparse.h>

#include <algorithm>
#include <cmath>
#include <jams/interface/config.h>
#include <jams/helpers/defaults.h>

#include "jams/helpers/consts.h"
#include "jams/helpers/exception.h"
#include "jams/cuda/cuda_sparsematrix.h"
#include "jams/core/globals.h"
#include "jams/core/thermostat.h"
#include "jams/core/output.h"

#include "cuda_llg_heun_kernel.h"

#include "jblib/containers/array.h"

using namespace std;

void CUDAHeunLLGSolver::initialize(const libconfig::Setting& settings)
{
  using namespace globals;

  CudaSolver::initialize(settings);

  time_step_ = jams::config_required<double>(settings, "t_step");
  double dt = time_step_ * kGyromagneticRatio;

  auto t_max = jams::config_required<double>(settings, "t_max");
  auto t_min = jams::config_optional<double>(settings, "t_min", 0.0);

  max_steps_ = static_cast<int>(t_max / time_step_);
  min_steps_ = static_cast<int>(t_min / time_step_);

  cout << "timestep " << dt << "\n";
  cout << "t_max " << t_max << " steps (" <<  max_steps_ << ")\n";
  cout << "t_min " << t_min << " steps (" << min_steps_ << ")\n";

  cout << "  creating stream\n";
  if(hipStreamCreate(&dev_stream_) != hipSuccess) {
    throw cuda_api_exception("", __FILE__, __LINE__, __PRETTY_FUNCTION__);
  }

  cout << "  copy time_step to symbol\n";
  if(hipMemcpyToSymbol(HIP_SYMBOL(dev_dt), &dt, sizeof(double)) != hipSuccess) {
    throw cuda_api_exception("", __FILE__, __LINE__, __PRETTY_FUNCTION__);
  }

  cout << "  copy num_spins to symbol\n";
  if(hipMemcpyToSymbol(HIP_SYMBOL(dev_num_spins), &globals::num_spins, sizeof(unsigned int)) != hipSuccess) {
    throw cuda_api_exception("", __FILE__, __LINE__, __PRETTY_FUNCTION__);
  }

  std::string thermostat_name = jams::config_optional<string>(config->lookup("sim"), "thermostat", jams::default_solver_gpu_thermostat);
  thermostat_ = Thermostat::create(thermostat_name);

  cout << "  thermostat " << thermostat_name.c_str() << "\n";

  nblocks = (num_spins+BLOCKSIZE-1)/BLOCKSIZE;

  cout << "done\n";
}

void CUDAHeunLLGSolver::run()
{
  using namespace globals;

  dim3 block_size;
  block_size.x = 85;
  block_size.y = 3;

  dim3 grid_size;
  grid_size.x = (globals::num_spins + block_size.x - 1) / block_size.x;
  grid_size.y = (3 + block_size.y - 1) / block_size.y;

    hipMemcpyAsync(dev_s_old_.data(),           // void *               dst
               dev_s_.data(),               // const void *         src
               num_spins3*sizeof(double),   // size_t               count
               hipMemcpyDeviceToDevice,    // enum hipMemcpyKind  kind
               dev_stream_);                   // device stream

  if (debug_is_enabled()) {
    if (hipPeekAtLastError() != hipSuccess) {
     throw cuda_api_exception("", __FILE__, __LINE__, __PRETTY_FUNCTION__);
    }
  }



    thermostat_->set_temperature(physics_module_->temperature());
    thermostat_->update();

    compute_fields();

    cuda_heun_llg_kernelA<<<grid_size, block_size>>>
        (dev_s_.data(), dev_ds_dt_.data(), dev_s_old_.data(),
          dev_h_.data(), thermostat_->noise(),
          dev_gyro_.data(), dev_alpha_.data());

  if (debug_is_enabled()) {
    if (hipPeekAtLastError() != hipSuccess) {
      throw cuda_api_exception("", __FILE__, __LINE__, __PRETTY_FUNCTION__);
    }
  }

    compute_fields();

    cuda_heun_llg_kernelB<<<grid_size, block_size>>>
      (dev_s_.data(), dev_ds_dt_.data(), dev_s_old_.data(),
        dev_h_.data(), thermostat_->noise(),
        dev_gyro_.data(), dev_alpha_.data());

  if (debug_is_enabled()) {
    if (hipPeekAtLastError() != hipSuccess) {
      throw cuda_api_exception("", __FILE__, __LINE__, __PRETTY_FUNCTION__);
    }
  }


    iteration_++;
}


CUDAHeunLLGSolver::~CUDAHeunLLGSolver()
{
  if (dev_stream_ != nullptr) {
    hipStreamDestroy(dev_stream_);
  }
}


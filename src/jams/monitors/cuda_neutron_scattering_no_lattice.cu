#include "hip/hip_runtime.h"
// cuda_neutron_scattering_no_lattice.cc                               -*-C++-*-
#include "jams/monitors/cuda_neutron_scattering_no_lattice.h"
#include "jams/monitors/cuda_neutron_scattering_no_lattice_kernels.cuh"

#include "jams/core/globals.h"
#include "jams/core/lattice.h"
#include "jams/core/solver.h"
#include "jams/helpers/output.h"

#include <hip/hip_runtime.h>

CudaNeutronScatteringNoLatticeMonitor::CudaNeutronScatteringNoLatticeMonitor(const libconfig::Setting &settings)
    : Monitor(settings){

  configure_kspace_vectors(settings);

  do_rspace_windowing_ = jams::config_optional(settings, "rspace_windowing", do_rspace_windowing_);
  std::cout << "rspace windowing: " << do_rspace_windowing_ << std::endl;

//  // default to 1.0 in case no form factor is given in the settings
//  fill(neutron_form_factors_.resize(lattice->num_materials(), num_k_), 1.0);
//  if (settings.exists("form_factor")) {
//    configure_form_factors(settings["form_factor"]);
//  }
//
//  if (settings.exists("polarizations")) {
//    configure_polarizations(settings["polarizations"]);
//  }

  if (settings.exists("periodogram")) {
    configure_periodogram(settings["periodogram"]);
  }

  periodogram_props_.sample_time = output_step_freq_ * solver->time_step();


  // NOTE: the memory layout here is DIFFERENT for the CPU version
  zero(spin_timeseries_.resize(periodogram_props_.length, globals::num_spins, 3));
  zero(spin_frequencies_.resize(periodogram_props_.length / 2 + 1, globals::num_spins, 3));

  zero(total_unpolarized_neutron_cross_section_.resize(
      periodogram_props_.length, kspace_path_.size()));
  zero(total_polarized_neutron_cross_sections_.resize(
      neutron_polarizations_.size(),periodogram_props_.length, kspace_path_.size()));
}

void CudaNeutronScatteringNoLatticeMonitor::configure_periodogram(libconfig::Setting &settings) {
  periodogram_props_.length = settings["length"];
  periodogram_props_.overlap = settings["overlap"];
}


void CudaNeutronScatteringNoLatticeMonitor::configure_kspace_vectors(const libconfig::Setting &settings) {
  kmax_ = jams::config_required<double>(settings, "kmax");
  kvector_ = jams::config_required<Vec3>(settings, "kvector");
  num_k_ = jams::config_required<int>(settings, "num_k");

  kspace_path_.resize(num_k_ + 1);
  for (auto i = 0; i < kspace_path_.size(); ++i) {
    kspace_path_(i) = kvector_ * i * (kmax_ / num_k_);
  }

}

void CudaNeutronScatteringNoLatticeMonitor::store_spin_data() {
  auto t = periodogram_index_;

  auto ptr_offset = t * globals::num_spins3;

  hipMemcpy(spin_timeseries_.device_data() + ptr_offset,
             globals::s.device_data(),
             globals::num_spins3*sizeof(double), hipMemcpyDeviceToDevice);
}

void CudaNeutronScatteringNoLatticeMonitor::output_fixed_spectrum() {
  // Do temporal fourier transform of spin data

  const int num_time_samples = periodogram_props_.length;



  int rank = 1;
  int transform_size[1] = {num_time_samples};
  int num_transforms = globals::num_spins3;
  int nembed[1] = {num_time_samples};
  int stride = globals::num_spins3;
  int dist = 1;

  hipfftHandle fft_plan;

  CHECK_CUFFT_STATUS(
      hipfftCreate(&fft_plan));

  CHECK_CUFFT_STATUS(
      hipfftPlanMany(&fft_plan, rank, transform_size, nembed,
                        stride, dist, nembed, stride,
                    dist, HIPFFT_D2Z, num_transforms));

//  jams::MultiArray<double, 2> spin_averages(globals::num_spins, 3);
//  zero(spin_averages);
//  for (auto i = 0; i < globals::num_spins; ++i) {
//    for (auto j = 0; j < 3; ++j) {
//      for (auto t = 0; t < num_time_samples; ++t) {
//        spin_averages(i, j) += spin_timeseries_(i, j, t);
//      }
//    }
//  }
//  element_scale(spin_averages, 1.0/double(num_time_samples));
//
//
//  for (auto i = 0; i < globals::num_spins; ++i) {
//    for (auto j = 0; j < 3; ++j) {
//      for (auto t = 0; t < num_time_samples; ++t) {
//        spin_frequencies_(i, j, t) = fft_window_default(t, num_time_samples) * (spin_timeseries_(i, j, t) - spin_averages(i,j));
//      }
//    }
//  }

  CHECK_CUFFT_STATUS(
    hipfftExecD2Z(fft_plan, reinterpret_cast<hipfftDoubleReal*>(spin_timeseries_.device_data()),  reinterpret_cast<hipfftDoubleComplex*>(spin_frequencies_.device_data())));

  CHECK_CUFFT_STATUS(
      hipfftDestroy(fft_plan));

  std::ofstream debug(jams::output::full_path_filename("debug.tsv"));
  for (auto t = 0; t < num_time_samples / 2 + 1; ++t) {
    debug << t << " " << spin_frequencies_(t, 0, 0).real() << " " << spin_frequencies_(t, 0, 0).imag() << " " << spin_frequencies_(t, 0, 1).real() << " " << spin_frequencies_(t, 0, 1).imag() << std::endl;
  }
  debug.close();


  // Calculate conj(S_i^a(w)) S_j^b(w) for every i and store in a structure like Sw(i, w) (i.e. a frequency spectrum for every spin)
  jams::MultiArray<std::complex<double>,2> s_conv(globals::num_spins, num_time_samples / 2 + 1);

  zero(s_conv);
  // this assumes out kspace_path is a single straight line
  const auto delta_q = kspace_path_(1) - kspace_path_(0);
  auto unit_q = unit_vector(delta_q);


  const int num_freq = num_time_samples / 2 + 1;
  const int num_k = kspace_path_.size();

  jams::MultiArray<std::complex<double>, 2> sqw(kspace_path_.size(),
                                                num_time_samples / 2 + 1);

  dim3 block_size = {32, 32, 1};
  dim3 grid_size = {(num_k + block_size.x - 1) / block_size.x,
                    (num_freq + block_size.y - 1) / block_size.y,
                    1};

  for (auto i = 0; i < globals::num_spins; ++i) {
    std::cout << i << std::endl;

    Vec3 r_i = lattice->atom_position(i);

    spectrum_i_equal_j<<<grid_size, block_size>>>(
        i, i, globals::num_spins, num_k, num_freq, unit_q[0], unit_q[1], unit_q[2],
        reinterpret_cast<const hipfftDoubleComplex*>(spin_frequencies_.device_data()),
        reinterpret_cast<hipfftDoubleComplex*>(sqw.device_data())
    );
    DEBUG_CHECK_CUDA_ASYNC_STATUS;

    for (auto j = i+1; j < globals::num_spins; ++j) {
      const Vec3 r_ij = lattice->displacement(r_i, lattice->atom_position(j));

      spectrum_i_not_equal_j<<<grid_size, block_size>>>(
          i, j, globals::num_spins, num_k, num_freq, unit_q[0], unit_q[1], unit_q[2],
          r_ij[0], r_ij[1], r_ij[2],
          reinterpret_cast<double*>(kspace_path_.device_data()),
          reinterpret_cast<const hipfftDoubleComplex*>(spin_frequencies_.device_data()),
          reinterpret_cast<hipfftDoubleComplex*>(sqw.device_data())
          );
      DEBUG_CHECK_CUDA_ASYNC_STATUS;
    }

//    if (i%100 == 0) {
//      std::ofstream ofs(jams::output::full_path_filename("neutron_scattering_fixed.tsv"));
//
//      ofs << "index\t" << "qx\t" << "qy\t" << "qz\t" << "q_A-1\t";
//      ofs << "freq_THz\t" << "energy_meV\t" << "sigma_unpol_re\t" << "sigma_unpol_im\t";
//      ofs << "\n";
//
//      // sample time is here because the fourier transform in time is not an integral
//      // but a discrete sum
//      auto prefactor = (periodogram_props_.sample_time / double(total_periods_)) * (1.0 / (kTwoPi * kHBarIU))
//                       * pow2((0.5 * kNeutronGFactor * pow2(kElementaryCharge)) / (kElectronMass * pow2(kSpeedOfLight)));
//      auto barns_unitcell = prefactor / (1e-28);
//      auto freq_delta = 1.0 / (periodogram_props_.length * periodogram_props_.sample_time);
//
//      for (auto w = 0; w <  num_time_samples / 2 + 1; ++w) {
//        for (auto k = 0; k < kspace_path_.size(); ++k) {
//          ofs << jams::fmt::integer << k << "\t";
//          ofs << jams::fmt::decimal << kspace_path_(k) << "\t";
//          ofs << jams::fmt::decimal << kTwoPi * norm(kspace_path_(k)) / (lattice->parameter() * 1e10) << "\t";
//          ofs << jams::fmt::decimal << (w * freq_delta) << "\t"; // THz
//          ofs << jams::fmt::decimal << (w * freq_delta) * 4.135668 << "\t"; // meV
//          // cross section output units are Barns Steradian^-1 Joules^-1 unitcell^-1
//          ofs << jams::fmt::sci << barns_unitcell * sqw(k, w).real() << "\t";
//          ofs << jams::fmt::sci << barns_unitcell * sqw(k, w).imag() << "\t";
//          ofs << "\n";
//        }
//        ofs << std::endl;
//      }
//
//      ofs.close();
//    }
  }

  std::ofstream ofs(jams::output::full_path_filename("neutron_scattering_fixed.tsv"));

  ofs << "index\t" << "qx\t" << "qy\t" << "qz\t" << "q_A-1\t";
  ofs << "freq_THz\t" << "energy_meV\t" << "sigma_unpol_re\t" << "sigma_unpol_im\t";
  ofs << "\n";

  // sample time is here because the fourier transform in time is not an integral
  // but a discrete sum
  auto prefactor = (periodogram_props_.sample_time / double(total_periods_)) * (1.0 / (kTwoPi * kHBarIU))
                   * pow2((0.5 * kNeutronGFactor * pow2(kElementaryCharge)) / (kElectronMass * pow2(kSpeedOfLight)));
  auto barns_unitcell = prefactor / (1e-28);
  auto freq_delta = 1.0 / (periodogram_props_.length * periodogram_props_.sample_time);

  for (auto w = 0; w <  num_time_samples / 2 + 1; ++w) {
    for (auto k = 0; k < kspace_path_.size(); ++k) {
      ofs << jams::fmt::integer << k << "\t";
      ofs << jams::fmt::decimal << kspace_path_(k) << "\t";
      ofs << jams::fmt::decimal << kTwoPi * norm(kspace_path_(k)) / (lattice->parameter() * 1e10) << "\t";
      ofs << jams::fmt::decimal << (w * freq_delta) << "\t"; // THz
      ofs << jams::fmt::decimal << (w * freq_delta) * 4.135668 << "\t"; // meV
      // cross section output units are Barns Steradian^-1 Joules^-1 unitcell^-1
      ofs << jams::fmt::sci << barns_unitcell * sqw(k, w).real() << "\t";
      ofs << jams::fmt::sci << barns_unitcell * sqw(k, w).imag() << "\t";
      ofs << "\n";
    }
    ofs << std::endl;
  }

  ofs.close();


}


void CudaNeutronScatteringNoLatticeMonitor::update(Solver *solver) {
  store_spin_data();
  periodogram_index_++;

  if (is_multiple_of(periodogram_index_, periodogram_props_.length)) {


//    shift_periodogram_overlap();
    total_periods_++;


    output_fixed_spectrum();
  }
}

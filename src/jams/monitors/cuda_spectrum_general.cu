#include "hip/hip_runtime.h"
//
// Created by Joseph Barker on 2018-11-22.
//

#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>

#include "jblib/containers/array.h"
#include "jblib/containers/cuda_array.h"

#include "jams/core/globals.h"
#include "jams/core/lattice.h"
#include "jams/cuda/cuda_common.h"
#include "jams/helpers/duration.h"
#include "jams/helpers/random.h"

#include "jams/monitors/spectrum_general.h"
#include "jams/monitors/cuda_spectrum_general.h"
#include "jams/monitors/cuda_spectrum_general_kernel.cuh"
#include "jams/helpers/consts.h"
#include "jams/cuda/cuda_common.h"

namespace {
    std::vector<hipFloatComplex> generate_expQR_float(const std::vector<std::vector<Vec3>> &qvecs, const Vec3& R) {

      const auto num_qvectors = qvecs.size();
      const auto num_qpoints = qvecs[0].size();

      std::vector<hipFloatComplex> result(num_qvectors * num_qpoints);

      std::complex<float> ImagTwoPi_f = {0.0f, static_cast<float>(2.0*kTwoPi)};
      for (auto q = 0; q < num_qpoints; ++q) {
        for (auto n = 0; n < num_qvectors; ++n) {
          const std::complex<float> val = exp(ImagTwoPi_f * static_cast<float>(dot(qvecs[n][q], R)));
          result[num_qvectors * q + n] = {val.real(), val.imag()};
        }
      }
      return result;
    }
}

CudaSpectrumGeneralMonitor::CudaSpectrumGeneralMonitor(const libconfig::Setting &settings) : SpectrumGeneralMonitor(
        settings) {

}

CudaSpectrumGeneralMonitor::~CudaSpectrumGeneralMonitor() {
  using namespace std;
  using namespace std::chrono;
  using namespace std::placeholders;
  using namespace globals;

  cout << "calculating correlation function" << std::endl;
  auto start_time = time_point_cast<milliseconds>(system_clock::now());
  cout << "start   " << get_date_string(start_time) << "\n\n";
  cout.flush();

  std::cout << duration_string(start_time, system_clock::now()) << " calculating fft time => frequency" << std::endl;

  this->apply_time_fourier_transform();

  std::cout << duration_string(start_time, system_clock::now()) << " done" << std::endl;

  jblib::Array<hipFloatComplex, 1> hst_spin_data(spin_data_.elements());
  for (auto i = 0; i < spin_data_.elements(); ++i) {
    hst_spin_data[i].x = static_cast<float>(spin_data_[i].real());
    hst_spin_data[i].y = static_cast<float>(spin_data_[i].imag());
  }
  jblib::CudaArray<hipFloatComplex, 1> dev_spin_data(hst_spin_data);



  std::vector<std::vector<Vec3>> qvecs(num_qvectors_);
  for (auto n = 0; n < num_qvectors_; ++n) {
    auto qvec_rand = qmax_ * uniform_random_sphere(jams::random_generator());
    std::cout << "qvec " << n << ": " << qvec_rand << std::endl;
    std::vector<Vec3> qpoints(num_qpoints_);
    for (auto i = 0; i < num_qpoints_; ++i){
      qpoints[i] = qvec_rand * (i / double(num_qpoints_-1));
    }
    qvecs[n] = qpoints;
  }


  vector<Vec3> r(num_spins);
  for (auto i = 0; i < num_spins; ++i) {
    r[i] = lattice->atom_position(i);
  }

  // support for lattice vacancies (we will skip these in the spectrum loop)
  vector<bool> is_vacancy(num_spins, false);
  for (auto i = 0; i < num_spins; ++i) {
    if (s(i, 0) == 0.0 && s(i, 1) == 0.0 && s(i, 2) == 0.0) {
      is_vacancy[i] = true;
    }
  }



  jblib::Array<hipFloatComplex, 2> SQw(num_qpoints_, padded_size_/2+1);
  for (auto i = 0; i < SQw.elements(); ++i) {
    SQw[i].x = 0.0;
    SQw[i].y = 0.0;
  }

  jblib::CudaArray<hipFloatComplex, 1> dev_SQw(SQw);

  const auto num_w_points = padded_size_/2+1;

  hipFloatComplex *dev_qfactors = nullptr;
  CHECK_CUDA_STATUS(hipMalloc((void**)&dev_qfactors, (num_qpoints_ * num_qvectors_)*sizeof(hipFloatComplex)));

  const dim3 block_size = {64, 8, 1};
  auto grid_size = cuda_grid_size(block_size, {num_w_points, num_qpoints_, 1});

  // generate spectrum looping over all i,j
  for (unsigned i = 0; i < globals::num_spins; ++i) {
    if (is_vacancy[i]) continue;
    std::cout << duration_string(start_time, system_clock::now()) << " " << i << std::endl;
    for (unsigned j = 0; j < globals::num_spins; ++j) {
      if (is_vacancy[j]) continue;

//      for (unsigned n = 0; n < qvecs.size(); ++n) {
//       precalculate the exponential factors for the spatial fourier transform
        const auto qfactors = generate_expQR_float(qvecs, lattice->displacement(j, i));


      CHECK_CUDA_STATUS(hipMemcpy(dev_qfactors, qfactors.data(),
                                        num_qpoints_ * num_qvectors_ * sizeof(hipFloatComplex), hipMemcpyHostToDevice));


        CudaSpectrumGeneralKernel <<< grid_size, block_size >> >
                                                  (i, j, num_w_points, num_qpoints_, num_qvectors_, padded_size_, dev_qfactors, dev_spin_data.data(), dev_SQw.data());
        DEBUG_CHECK_CUDA_ASYNC_STATUS;
//      }
    }

    if (i%10 == 0) {
      dev_SQw.copy_to_host_array(SQw);
      std::ofstream cfile(seedname + "_corr.tsv");
      cfile << "q\tfrequency\tRe_SQw\tIm_SQw\n";
      for (unsigned q = 0; q < num_qpoints_; ++q) {
        for (unsigned w = 0; w < padded_size_/2+1; ++w) {
          cfile << qmax_ * (q / double(num_qpoints_-1)) << "\t";
          cfile << 0.5*w * freq_delta_ << "\t";
          cfile << SQw(q, w).x / static_cast<double>(padded_size_*(i + 1)*num_qvectors_) << "\t";
          cfile << SQw(q, w).y / static_cast<double>(padded_size_*(i + 1)*num_qvectors_) << "\n";
        }
      }
      cfile.flush();
      cfile.close();
    }

  }

  auto end_time = time_point_cast<milliseconds>(system_clock::now());
  cout << "finish  " << get_date_string(end_time) << "\n\n";
  cout << "runtime " << duration_string(start_time, end_time) << "\n";
  cout.flush();

}

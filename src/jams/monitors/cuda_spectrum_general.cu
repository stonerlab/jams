#include "hip/hip_runtime.h"
//
// Created by Joseph Barker on 2018-11-22.
//

#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>

#include "jams/core/globals.h"
#include "jams/core/lattice.h"
#include "jams/cuda/cuda_common.h"
#include "jams/helpers/duration.h"
#include "jams/helpers/random.h"

#include "jams/monitors/spectrum_general.h"
#include "jams/monitors/cuda_spectrum_general.h"
#include "jams/monitors/cuda_spectrum_general_kernel.cuh"
#include "jams/helpers/consts.h"
#include "jams/cuda/cuda_common.h"

namespace {
    std::vector<hipFloatComplex> generate_expQR_float(const std::vector<std::vector<Vec3>> &qvecs, const Vec3& R) {

      const auto num_qvectors = qvecs.size();
      const auto num_qpoints = qvecs[0].size();

      std::vector<hipFloatComplex> result(num_qvectors * num_qpoints);

      std::complex<float> ImagTwoPi_f = {0.0f, static_cast<float>(2.0*kTwoPi)};
      for (auto q = 0; q < num_qpoints; ++q) {
        for (auto n = 0; n < num_qvectors; ++n) {
          const std::complex<float> val = exp(ImagTwoPi_f * static_cast<float>(dot(qvecs[n][q], R)));
          result[num_qvectors * q + n] = {val.real(), val.imag()};
        }
      }
      return result;
    }
}

CudaSpectrumGeneralMonitor::CudaSpectrumGeneralMonitor(const libconfig::Setting &settings) : SpectrumGeneralMonitor(
        settings) {

}

CudaSpectrumGeneralMonitor::~CudaSpectrumGeneralMonitor() {
  using namespace std;
  using namespace std::chrono;
  using namespace std::placeholders;
  using namespace globals;

  cout << "calculating correlation function" << std::endl;
  auto start_time = time_point_cast<milliseconds>(system_clock::now());
  cout << "start   " << get_date_string(start_time) << "\n\n";
  cout.flush();

  std::cout << duration_string(start_time, system_clock::now()) << " calculating fft time => frequency" << std::endl;

  this->apply_time_fourier_transform();

  std::cout << duration_string(start_time, system_clock::now()) << " done" << std::endl;

  jams::MultiArray<hipFloatComplex, 1> spin_data_float_(spin_data_.elements());

  auto count = 0;
  for (auto i = 0; i < spin_data_.size(0); ++i) {
    for (auto j = 0; j < spin_data_.size(1); ++j) {
      spin_data_float_(count).x = static_cast<float>(spin_data_(i,j).real());
      spin_data_float_(count).y = static_cast<float>(spin_data_(i,j).imag());
      count++;
    }
  }

  std::vector<std::vector<Vec3>> qvecs(num_qvectors_);
  for (auto n = 0; n < num_qvectors_; ++n) {
    auto qvec_rand = qmax_ * uniform_random_sphere(jams::random_generator());
    std::cout << "qvec " << n << ": " << qvec_rand << std::endl;
    std::vector<Vec3> qpoints(num_qpoints_);
    for (auto i = 0; i < num_qpoints_; ++i){
      qpoints[i] = qvec_rand * (i / double(num_qpoints_-1));
    }
    qvecs[n] = qpoints;
  }


  vector<Vec3> r(num_spins);
  for (auto i = 0; i < num_spins; ++i) {
    r[i] = lattice->atom_position(i);
  }

  // support for lattice vacancies (we will skip these in the spectrum loop)
  vector<bool> is_vacancy(num_spins, false);
  for (auto i = 0; i < num_spins; ++i) {
    if (s(i, 0) == 0.0 && s(i, 1) == 0.0 && s(i, 2) == 0.0) {
      is_vacancy[i] = true;
    }
  }



  jams::MultiArray<hipFloatComplex, 2> SQw(num_qpoints_, padded_size_/2+1);
  SQw.zero();

  const auto num_w_points = padded_size_/2+1;

  hipFloatComplex *dev_qfactors = nullptr;
  CHECK_CUDA_STATUS(hipMalloc((void**)&dev_qfactors, (num_qpoints_ * num_qvectors_)*sizeof(hipFloatComplex)));

  const dim3 block_size = {64, 8, 1};
  auto grid_size = cuda_grid_size(block_size, {num_w_points, num_qpoints_, 1});

  // generate spectrum looping over all i,j
  for (unsigned i = 0; i < globals::num_spins; ++i) {
    if (is_vacancy[i]) continue;
    std::cout << duration_string(start_time, system_clock::now()) << " " << i << std::endl;
    for (unsigned j = 0; j < globals::num_spins; ++j) {
      if (is_vacancy[j]) continue;

//      for (unsigned n = 0; n < qvecs.size(); ++n) {
//       precalculate the exponential factors for the spatial fourier transform
        const auto qfactors = generate_expQR_float(qvecs, lattice->displacement(j, i));


      CHECK_CUDA_STATUS(hipMemcpy(dev_qfactors, qfactors.data(),
                                        num_qpoints_ * num_qvectors_ * sizeof(hipFloatComplex), hipMemcpyHostToDevice));


        CudaSpectrumGeneralKernel <<< grid_size, block_size >> >
                                                  (i, j, num_w_points, num_qpoints_, num_qvectors_, padded_size_, dev_qfactors, spin_data_float_.device_data(), SQw.device_data());
        DEBUG_CHECK_CUDA_ASYNC_STATUS;
//      }
    }

    if (i%10 == 0) {
      std::ofstream cfile(seedname + "_corr.tsv");
      cfile << "q\tfrequency\tRe_SQw\tIm_SQw\n";
      for (unsigned q = 0; q < num_qpoints_; ++q) {
        for (unsigned w = 0; w < padded_size_/2+1; ++w) {
          cfile << qmax_ * (q / double(num_qpoints_-1)) << "\t";
          cfile << 0.5*w * freq_delta_ << "\t";
          cfile << SQw(q, w).x / static_cast<double>(padded_size_*(i + 1)*num_qvectors_) << "\t";
          cfile << SQw(q, w).y / static_cast<double>(padded_size_*(i + 1)*num_qvectors_) << "\n";
        }
      }
      cfile.flush();
      cfile.close();
    }

  }

  auto end_time = time_point_cast<milliseconds>(system_clock::now());
  cout << "finish  " << get_date_string(end_time) << "\n\n";
  cout << "runtime " << duration_string(start_time, end_time) << "\n";
  cout.flush();

}

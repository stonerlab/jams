#include "hip/hip_runtime.h"
#include <jams/cuda/cuda_spin_ops.h>
#include <jams/cuda/cuda_device_vector_ops.h>

__global__ void cuda_rotate_spins_kernel(double* spins, const int* indices, const unsigned size,
                                         double Rxx, double Rxy, double Rxz,
                                         double Ryx, double Ryy, double Ryz,
                                         double Rzx, double Rzy, double Rzz) {
  unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx < size) {
    double s[3] = {spins[3*indices[idx] + 0], spins[3*indices[idx] + 1], spins[3*indices[idx] + 2]};

    spins[3*indices[idx] + 0] = Rxx * s[0] + Rxy * s[1] + Rxz * s[2];
    spins[3*indices[idx] + 1] = Ryx * s[0] + Ryy * s[1] + Ryz * s[2];
    spins[3*indices[idx] + 2] = Rzx * s[0] + Rzy * s[1] + Rzz * s[2];
  }

}

void jams::rotate_spins_cuda(jams::MultiArray<double, 2> &spins,
                             const Mat3 &rotation_matrix,
                             const jams::MultiArray<int, 1> &indices) {

  dim3 block_size;
  block_size.x = 128;

  dim3 grid_size;
  grid_size.x = (indices.size() + block_size.x - 1) / block_size.x;

  cuda_rotate_spins_kernel<<<grid_size, block_size>>>(
      spins.device_data(), indices.device_data(), indices.size(),
      rotation_matrix[0][0], rotation_matrix[0][1], rotation_matrix[0][2],
      rotation_matrix[1][0], rotation_matrix[1][1], rotation_matrix[1][2],
      rotation_matrix[2][0], rotation_matrix[2][1], rotation_matrix[2][2]);
}


__global__ void cuda_scale_spins_kernel(double* spins, const int* indices, const unsigned size,
                                         const double scale_factor) {
  unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx < size) {
    spins[3*indices[idx] + 0] *= scale_factor;
    spins[3*indices[idx] + 1] *= scale_factor;
    spins[3*indices[idx] + 2] *= scale_factor;
  }

}

void jams::scale_spins_cuda(jams::MultiArray<double, 2> &spins,
                             const double &scale_factor,
                             const jams::MultiArray<int, 1> &indices) {

  dim3 block_size;
  block_size.x = 128;

  dim3 grid_size;
  grid_size.x = (indices.size() + block_size.x - 1) / block_size.x;

  cuda_scale_spins_kernel<<<grid_size, block_size>>>(
      spins.device_data(), indices.device_data(), indices.size(), scale_factor);
}


__global__ void cuda_add_to_spins_kernel(double* spins, const int* indices, const unsigned size,
                                        const double additional_length) {
  unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx < size) {
    double s[3] = {spins[3*indices[idx] + 0], spins[3*indices[idx] + 1], spins[3*indices[idx] + 2]};

    double s_norm = norm(s);

    double scale_factor = (s_norm == 0.0) ? additional_length : (s_norm + additional_length) / s_norm;

    spins[3*indices[idx] + 0] *= scale_factor;
    spins[3*indices[idx] + 1] *= scale_factor;
    spins[3*indices[idx] + 2] *= scale_factor;
  }

}

void jams::add_to_spins_cuda(jams::MultiArray<double, 2> &spins,
                            const double &additional_length,
                            const jams::MultiArray<int, 1> &indices) {

  dim3 block_size;
  block_size.x = 128;

  dim3 grid_size;
  grid_size.x = (indices.size() + block_size.x - 1) / block_size.x;

  cuda_add_to_spins_kernel<<<grid_size, block_size>>>(
      spins.device_data(), indices.device_data(), indices.size(), additional_length);
}

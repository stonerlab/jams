// Copyright 2014 Joseph Barker. All rights reserved.

#include <hipblas.h>

#include "cuda_solver.h"
#include "cuda_solver_kernels.h"

#include "jams/helpers/consts.h"
#include "cuda_defs.h"
#include "cuda_sparsematrix.h"
#include "jams/helpers/exception.h"
#include "jams/core/globals.h"
#include "jams/core/hamiltonian.h"
#include "jams/core/solver.h"
#include "jams/core/thermostat.h"
#include "jams/helpers/utils.h"
#include "jams/solvers/cuda_llg_heun.h"
#include "jams/solvers/cpu_llg_heun.h"
#include "jams/solvers/cpu_monte_carlo_metropolis.h"

#include "jams/cuda/wrappers/stream.h"

using namespace std;

void CudaSolver::sync_device_data() {
  dev_s_.copy_to_host_array(globals::s);
  dev_h_.copy_to_host_array(globals::h);
  dev_ds_dt_.copy_to_host_array(globals::ds_dt);
}

void CudaSolver::initialize(const libconfig::Setting& settings) {
  using namespace globals;

  Solver::initialize(settings);

  cout << "\ninitializing CUDA base solver\n";
  cout << "  initialising CUDA streams\n";

  is_cuda_solver_ = true;

//-----------------------------------------------------------------------------
// Transfer the the other arrays to the device
//-----------------------------------------------------------------------------

  cout << "  transfering array data to device\n";
  jblib::Array<double, 2> zero(num_spins, 3, 0.0);

  // spin arrays
  dev_s_        = jblib::CudaArray<double, 1>(s);
  dev_s_old_    = jblib::CudaArray<double, 1>(s);
  dev_ds_dt_    = jblib::CudaArray<double, 1>(zero);

  // field array
  dev_h_        = jblib::CudaArray<double, 1>(zero);

  // materials array
  jblib::Array<double, 2> mat(num_spins, 3);

  dev_gyro_      = jblib::CudaArray<double, 1>(gyro);
  dev_alpha_     = jblib::CudaArray<double, 1>(alpha);


  cout << "\n";
}

void CudaSolver::run() {
}

void CudaSolver::compute_fields() {
  using namespace globals;

  for (std::vector<Hamiltonian*>::iterator it = hamiltonians_.begin() ; it != hamiltonians_.end(); ++it) {
    (*it)->calculate_fields();
  }

  // zero the field array
  if (hipMemsetAsync(dev_h_.data(), 0.0, num_spins3*sizeof(double), dev_stream_.get()) != hipSuccess) {
    throw cuda_api_exception("", __FILE__, __LINE__, __PRETTY_FUNCTION__);
  }

  const double alpha = 1.0;
  for (std::vector<Hamiltonian*>::iterator it = hamiltonians_.begin() ; it != hamiltonians_.end(); ++it) {
    hipblasDaxpy(globals::num_spins3, alpha, (*it)->dev_ptr_field(), 1, dev_h_.data(), 1);
  }
}

CudaSolver::~CudaSolver() {
}

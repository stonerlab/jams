#include "hip/hip_runtime.h"
#include "cuda_defs.h"

// y_ij <-- alpha_i * beta * x_ij

__global__ void cuda_array_elementwise_scale_kernel_general_(
    const unsigned int n,            // n elements in i index
    const unsigned int m,            // m elements in j index
    const double * alpha,   // scale factors array of length n
    const double   beta,    // uniform scale factor
    double * x,             // input array
    const unsigned int incx,         // input increment
    double * y,             // output array
    const unsigned int incy)         // output increment
{
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;

    if (idx < n) {
        if (idy < m) {
            y[idx * m + idy + incy] = alpha[idx] * beta * x[idx * m + idy + incx];
        }
    }
}

__global__ void cuda_array_elementwise_scale_kernel_noinc_(
    const unsigned int n,            // n elements in i index
    const unsigned int m,            // m elements in j index
    const double * alpha,   // scale factors array of length n
    const double   beta,    // uniform scale factor
    double * x,             // input array
    double * y)             // output array
{
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;

    if (idx < n) {
        if (idy < m) {
            y[idx * m + idy] = alpha[idx] * beta * x[idx * m + idy];
        }
    }
}

__global__ void cuda_array_elementwise_scale_kernel_noinc_self_(
    const unsigned int n,            // n elements in i index
    const unsigned int m,            // m elements in j index
    const double * alpha,   // scale factors array of length n
    const double   beta,    // uniform scale factor
    double * x)             // input/output array
{
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;

    if (idx < n) {
        if (idy < m) {
            x[idx * m + idy] = alpha[idx] * beta * x[idx * m + idy];
        }
    }
}

void cuda_array_elementwise_scale(
    const unsigned int n,            // n elements in i index
    const unsigned int m,            // m elements in j index
    const double * alpha,   // scale factors array of length n
    const double   beta,    // uniform scale factor
    double * x,             // input array
    const unsigned int incx,         // input increment
    double * y,             // output array
    const unsigned int incy,         // output increment
    hipStream_t stream = 0    // cuda stream
)
{
    dim3 block_size;
    block_size.x = 32;

    // if (m < 4) {
    //     block_size.y = m;
    // } else {
        block_size.y = 4;
    // }

    dim3 grid_size;
    grid_size.x = (n + block_size.x - 1) / block_size.x;
    grid_size.y = (m + block_size.y - 1) / block_size.y;

    if (incx == 1 && incy == 1) {
        if (x == y) {
            cuda_array_elementwise_scale_kernel_noinc_self_<<<grid_size, block_size, 0, stream>>>(n, m, alpha, beta, x);
            cuda_kernel_error_check();
            return;
        } else {
            cuda_array_elementwise_scale_kernel_noinc_<<<grid_size, block_size, 0, stream>>>(n, m, alpha, beta, x, y);
            cuda_kernel_error_check();
            return;
        }
    }

    cuda_array_elementwise_scale_kernel_general_<<<grid_size, block_size, 0, stream>>>(n, m, alpha, beta, x, incx, y, incy);
    cuda_kernel_error_check();
    return;
}

__global__ void cuda_array_double_to_float_kernel(
    const unsigned int n,            // n elements in i index
    const double * in,
    float * out
)
{
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < n) {
        out[idx] = float(in[idx]);
    }
}

__global__ void cuda_array_float_to_double_kernel(
    const unsigned int n,            // n elements in i index
    const float * in,
    double * out
) 
{
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < n) {
        out[idx] = double(in[idx]);
    }
}

void cuda_array_double_to_float(
    const unsigned int n,            // n elements in i index
    const double * in,
    float * out,  
    hipStream_t stream     // cuda stream
)
{
    dim3 block_size;
    block_size.x = 128;

    dim3 grid_size;
    grid_size.x = (n + block_size.x - 1) / block_size.x;

    cuda_array_double_to_float_kernel<<<grid_size, block_size, 0, stream>>>(n, in, out);
}

void cuda_array_float_to_double(
    const unsigned int n,            // n elements in i index
    const float * in,
    double * out,  
    hipStream_t stream     // cuda stream
) 
{
    dim3 block_size;
    block_size.x = 128;

    dim3 grid_size;
    grid_size.x = (n + block_size.x - 1) / block_size.x;

    cuda_array_float_to_double_kernel<<<grid_size, block_size, 0, stream>>>(n, in, out);
}

#include "hip/hip_runtime.h"
#include "core/globals.h"
#include "core/utils.h"
#include "core/maths.h"
#include "core/consts.h"
#include "core/cuda_defs.h"

#include "hamiltonian/uniaxial.h"
#include "hamiltonian/uniaxial_kernel.h"

UniaxialHamiltonian::UniaxialHamiltonian(const libconfig::Setting &settings)
: Hamiltonian(settings),
  mca_order_(),
  mca_value_()
{
    ::output.write("initialising Uniaxial Hamiltonian\n");
    // output in default format for now
    outformat_ = TEXT;

    // resize member arrays
    energy_.resize(globals::num_spins);
    field_.resize(globals::num_spins, 3);
    field_.zero();

    bool has_d2z = false;
    bool has_d4z = false;
    bool has_d6z = false;


    // don't allow mixed specification of anisotropy
    if ( (settings.exists("K1") || settings.exists("K2") || settings.exists("K3")) &&
         (settings.exists("d2z") || settings.exists("d4z") || settings.exists("d6z")) ) {
      jams_error("UniaxialHamiltonian: anisotropy should only be specified in terms of K1, K2, K3 or d2z, d4z, d6z in the config file");
    }

    // deal with magnetic anisotropy constants
    jblib::Array<double, 1> K1(globals::num_spins, 0.0);
    jblib::Array<double, 1> K2(globals::num_spins, 0.0);
    jblib::Array<double, 1> K3(globals::num_spins, 0.0);

    if(settings.exists("K1")) {
        if (settings["K1"].getLength() != lattice.num_materials()) {
            jams_error("UniaxialHamiltonian: K1 must be specified for every material");
        }
        for (int i = 0; i < globals::num_spins; ++i) {
            K1(i) = double(settings["K1"][lattice.atom_material(i)])/kBohrMagneton;
        }
        has_d2z = true;
    }


    if(settings.exists("K2")) {
        if (settings["K2"].getLength() != lattice.num_materials()) {
            jams_error("UniaxialHamiltonian: K2 must be specified for every material");
        }
        for (int i = 0; i < globals::num_spins; ++i) {
            K2(i) = double(settings["K2"][lattice.atom_material(i)])/kBohrMagneton;
        }
        has_d2z = true;
        has_d4z = true;
    }

    if(settings.exists("K3")) {
        if (settings["K3"].getLength() != lattice.num_materials()) {
            jams_error("UniaxialHamiltonian: K3 must be specified for every material");
        }
        for (int i = 0; i < globals::num_spins; ++i) {
            K3(i) = double(settings["K3"][lattice.atom_material(i)])/kBohrMagneton;
        }
        has_d2z = true;
        has_d4z = true;
        has_d6z = true;
    }

    if (has_d2z) {
        mca_order_.push_back(2);
        jblib::Array<double, 1> mca(globals::num_spins, 0.0);
        for (int i = 0; i < globals::num_spins; ++i) {
            mca(i) = -(2.0/3.0)*(K1(i) + (8.0/7.0)*K2(i) + (8.0/7.0)*K3(i));
        }
        mca_value_.push_back(mca);
    }


    if (has_d4z) {
        mca_order_.push_back(4);
        jblib::Array<double, 1> mca(globals::num_spins, 0.0);
        for (int i = 0; i < globals::num_spins; ++i) {
            mca(i) = ((8.0/35.0)*K2(i) + (144.0/385.0)*K3(i));
        }
        mca_value_.push_back(mca);
    }
    if (has_d6z) {
        mca_order_.push_back(6);
        jblib::Array<double, 1> mca(globals::num_spins, 0.0);
        for (int i = 0; i < globals::num_spins; ++i) {
            mca(i) = -((16.0/231.0)*K3(i));
        }
        mca_value_.push_back(mca);
    }


    // deal with magnetocrystalline anisotropy coefficients
    if(settings.exists("d2z")) {
        if (settings["d2z"].getLength() != lattice.num_materials()) {
            jams_error("UniaxialHamiltonian: d2z must be specified for every material");
        }
        mca_order_.push_back(2);

        jblib::Array<double, 1> mca(globals::num_spins, 0.0);
        for (int i = 0; i < globals::num_spins; ++i) {
            mca(i) = double(settings["d2z"][lattice.atom_material(i)])/kBohrMagneton;
        }
        mca_value_.push_back(mca);
    }



    if(settings.exists("d4z")) {
        if (settings["d4z"].getLength() != lattice.num_materials()) {
            jams_error("UniaxialHamiltonian: d4z must be specified for every material");
        }
        mca_order_.push_back(4);
        jblib::Array<double, 1> mca(globals::num_spins, 0.0);
        for (int i = 0; i < globals::num_spins; ++i) {
            mca(i) = double(settings["d4z"][lattice.atom_material(i)])/kBohrMagneton;
        }
        mca_value_.push_back(mca);
    }

    if(settings.exists("d6z")) {
        if (settings["d6z"].getLength() != lattice.num_materials()) {
            jams_error("UniaxialHamiltonian: d6z must be specified for every material");
        }
        mca_order_.push_back(6);
        jblib::Array<double, 1> mca(globals::num_spins, 0.0);
        for (int i = 0; i < globals::num_spins; ++i) {
            mca(i) = double(settings["d6z"][lattice.atom_material(i)])/kBohrMagneton;
        }
        mca_value_.push_back(mca);
    }


    // transfer arrays to cuda device if needed
#ifdef CUDA
    if (solver->is_cuda_solver()) {
        dev_energy_ = jblib::CudaArray<double, 1>(energy_);
        dev_field_ = jblib::CudaArray<double, 1>(field_);

        jblib::Array<int, 1> tmp_mca_order(mca_order_.size());
        for (int i = 0; i < mca_order_.size(); ++i) {
            tmp_mca_order[i] = mca_order_[i];
        }

        dev_mca_order_ = jblib::CudaArray<int, 1>(tmp_mca_order);

        jblib::Array<double, 1> tmp_mca_value(mca_order_.size() * globals::num_spins);

        for (int i = 0; i < globals::num_spins; ++i) {
            for (int j = 0; j < mca_order_.size(); ++j) {
                tmp_mca_value[ mca_order_.size() * i + j] = mca_value_[j](i);
            }
        }
        dev_mca_value_ = tmp_mca_value;
    }
#endif

}

// --------------------------------------------------------------------------

double UniaxialHamiltonian::calculate_total_energy() {
    double e_total = 0.0;
    for (int i = 0; i < globals::num_spins; ++i) {
        e_total += calculate_one_spin_energy(i);
    }
     return e_total;
}

// --------------------------------------------------------------------------

double UniaxialHamiltonian::calculate_one_spin_energy(const int i) {
    using namespace globals;
    double energy = 0.0;

    for (int n = 0; n < mca_order_.size(); ++n) {
        energy += mca_value_[n](i) * legendre_poly(s(i, 2), mca_order_[n]);
    }

    return energy;
}

// --------------------------------------------------------------------------

double UniaxialHamiltonian::calculate_one_spin_energy_difference(const int i, const jblib::Vec3<double> &spin_initial, const jblib::Vec3<double> &spin_final) {
    using std::pow;

    double e_initial = 0.0;
    double e_final = 0.0;

    for (int n = 0; n < mca_order_.size(); ++n) {
        e_initial += mca_value_[n](i) * legendre_poly(spin_initial.z, mca_order_[n]);
    }

    for (int n = 0; n < mca_order_.size(); ++n) {
        e_final += mca_value_[n](i) * legendre_poly(spin_final.z, mca_order_[n]);
    }

    return e_final - e_initial;
}

// --------------------------------------------------------------------------

void UniaxialHamiltonian::calculate_energies() {
    for (int i = 0; i < globals::num_spins; ++i) {
        energy_[i] = calculate_one_spin_energy(i);
    }
}

// --------------------------------------------------------------------------

void UniaxialHamiltonian::calculate_one_spin_field(const int i, double local_field[3]) {
    using namespace globals;
    using std::pow;
    const double sz = s(i, 2);
    local_field[0] = 0.0;
    local_field[1] = 0.0;
    local_field[2] = 0.0;

    for (int n = 0; n < mca_order_.size(); ++n) {
        local_field[2] += -mca_value_[n](i) * legendre_dpoly(sz, mca_order_[n]);
    }

}



// --------------------------------------------------------------------------

void UniaxialHamiltonian::calculate_fields() {

    // dev_s needs to be found from the solver

    if (solver->is_cuda_solver()) {
#ifdef CUDA
        cuda_uniaxial_field_kernel<<<(globals::num_spins+BLOCKSIZE-1)/BLOCKSIZE, BLOCKSIZE >>>
            (globals::num_spins, mca_order_.size(), dev_mca_order_.data(), dev_mca_value_.data(), solver->dev_ptr_spin(), dev_field_.data());
#endif  // CUDA
    } else {
        field_.zero();
        for (int n = 0; n < mca_order_.size(); ++n) {
            for (int i = 0; i < globals::num_spins; ++i) {
                field_(i, 2) += -mca_value_[n](i) * legendre_dpoly(globals::s(i, 2), mca_order_[n]);
            }
        }
    }
}
// --------------------------------------------------------------------------

void UniaxialHamiltonian::output_energies(OutputFormat format) {
    switch(format) {
        case TEXT:
            output_energies_text();
        case HDF5:
            jams_error("Uniaxial energy output: HDF5 not yet implemented");
        default:
            jams_error("Uniaxial energy output: unknown format");
    }
}

// --------------------------------------------------------------------------

void UniaxialHamiltonian::output_fields(OutputFormat format) {
    switch(format) {
        case TEXT:
            output_fields_text();
        case HDF5:
            jams_error("Uniaxial energy output: HDF5 not yet implemented");
        default:
            jams_error("Uniaxial energy output: unknown format");
    }
}

// --------------------------------------------------------------------------

void UniaxialHamiltonian::output_energies_text() {

}

// --------------------------------------------------------------------------

void UniaxialHamiltonian::output_fields_text() {

}

double UniaxialHamiltonian::calculate_bond_energy_difference(const int i, const int j, const Vec3 &sj_initial, const Vec3 &sj_final) {
  if (i != j) {
    return 0.0;
    } else {
  return calculate_one_spin_energy_difference(i, sj_initial, sj_final);
    }
}

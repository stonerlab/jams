#include "hip/hip_runtime.h"
#include "core/globals.h"
#include "core/consts.h"
#include "core/utils.h"

#include "hamiltonian/dipole_bruteforce.h"
#include "hamiltonian/dipole_bruteforce_kernel.h"


DipoleHamiltonianBruteforce::DipoleHamiltonianBruteforce(const libconfig::Setting &settings)
: HamiltonianStrategy(settings) {
    jblib::Vec3<double> super_cell_dim(0.0, 0.0, 0.0);

    for (int n = 0; n < 3; ++n) {
        super_cell_dim[n] = 0.5*double(lattice.size(n));
    }

    r_cutoff_ = *std::max_element(super_cell_dim.begin(), super_cell_dim.end());

    settings.lookupValue("r_cutoff", r_cutoff_);
    output.write("  r_cutoff: %e\n", r_cutoff_);

    dipole_prefactor_ = kVacuumPermeadbility*kBohrMagneton /(4*kPi*::lattice.parameter() * ::lattice.parameter() * ::lattice.parameter());


#ifdef CUDA
    if (solver->is_cuda_solver()) {
    bool super_cell_pbc[3];
    float super_unit_cell[3][3];
    float super_unit_cell_inv[3][3];

    for (int i = 0; i < 3; ++i) {
        super_cell_pbc[i] = ::lattice.is_periodic(i);
    }

    for (int i = 0; i < 3; ++i) {
        for (int j = 0; j < 3; ++j) {
            super_unit_cell[i][j] = ::lattice.unit_cell_vector(i)[j] * ::lattice.size(j);
        }
    }
    matrix_invert(super_unit_cell, super_unit_cell_inv);

    float r_cutoff_float = r_cutoff_;

    float f_dipole_prefactor = dipole_prefactor_;

    hipMemcpyToSymbol(HIP_SYMBOL(dev_dipole_prefactor),    &f_dipole_prefactor,       sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(dev_r_cutoff),           &r_cutoff_float,       sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(dev_super_cell_pbc),      super_cell_pbc,      3 * sizeof(bool));
    hipMemcpyToSymbol(HIP_SYMBOL(dev_super_unit_cell),     super_unit_cell,     9 * sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(dev_super_unit_cell_inv), super_unit_cell_inv, 9 * sizeof(float));

    jblib::Array<float, 1> f_mus(globals::num_spins);
    for (int i = 0; i < globals::num_spins; ++i) {
      f_mus[i] = globals::mus[i];
    }

    dev_mus_ = jblib::CudaArray<float, 1>(f_mus);

    jblib::Array<float, 2> r(globals::num_spins, 3);

    for (int i = 0; i < globals::num_spins; ++i) {
        for (int j = 0; j < 3; ++j) {
            r(i, j) = lattice.atom_position(i)[j];
        }
    }

    dev_r_ = jblib::CudaArray<float, 1>(r);

    hipStreamCreate(&dev_stream_);

    dev_blocksize_ = 128;
    }
#endif  // CUDA

}

// --------------------------------------------------------------------------

double DipoleHamiltonianBruteforce::calculate_total_energy() {
   double e_total = 0.0;
   for (int i = 0; i < globals::num_spins; ++i) {
       e_total += calculate_one_spin_energy(i);
   }
    return e_total;
}

// --------------------------------------------------------------------------


double DipoleHamiltonianBruteforce::calculate_one_spin_energy(const int i, const jblib::Vec3<double> &s_i) {
    double h[3];
    calculate_one_spin_field(i, h);
    return -(s_i[0]*h[0] + s_i[1]*h[1] + s_i[2]*h[2]);
}

// --------------------------------------------------------------------------

double DipoleHamiltonianBruteforce::calculate_one_spin_energy(const int i) {
    jblib::Vec3<double> s_i(globals::s(i, 0), globals::s(i, 1), globals::s(i, 2));
    return 0.5*calculate_one_spin_energy(i, s_i);
}

// --------------------------------------------------------------------------

double DipoleHamiltonianBruteforce::calculate_one_spin_energy_difference(const int i, const jblib::Vec3<double> &spin_initial, const jblib::Vec3<double> &spin_final) {
    double h[3];
    calculate_one_spin_field(i, h);
    double e_initial = -(spin_initial[0]*h[0] + spin_initial[1]*h[1] + spin_initial[2]*h[2]);
    double e_final = -(spin_final[0]*h[0] + spin_final[1]*h[1] + spin_final[2]*h[2]);
    return 0.5*(e_final - e_initial);
}
// --------------------------------------------------------------------------

void DipoleHamiltonianBruteforce::calculate_energies(jblib::Array<double, 1>& energies) {
    assert(energies.size() == globals::num_spins);
    for (int i = 0; i < globals::num_spins; ++i) {
        energies[i] = calculate_one_spin_energy(i);
    }
}

// --------------------------------------------------------------------------

void DipoleHamiltonianBruteforce::calculate_one_spin_field(const int i, double h[3]) {
    int n,j;
    double r_abs, s_j_dot_rhat, w0;
    jblib::Vec3<double> r_ij, s_j, field;

    const double prefactor = globals::mus(i) * dipole_prefactor_;

    h[0] = 0.0; h[1] = 0.0; h[2] = 0.0;

    for (j = 0; j < globals::num_spins; ++j) {
        if (j == i) continue;

        r_ij = lattice.displacement(i, j);

        r_abs = r_ij.norm_sq();

        if (r_abs > r_cutoff_ * r_cutoff_) continue;

        r_abs = 1.0 / sqrt(r_abs);

        w0 = prefactor * globals::mus(j) * (r_abs * r_abs * r_abs);

        s_j = {globals::s(j, 0), globals::s(j, 1), globals::s(j, 2)};
        s_j_dot_rhat = dot(s_j, r_ij) * r_abs;

        #pragma unroll
        for (n = 0; n < 3; ++n) {
            h[n] += (3.0 * r_ij[n] * s_j_dot_rhat  * r_abs - s_j[n]) * w0;
        }
    }
}

// --------------------------------------------------------------------------

void DipoleHamiltonianBruteforce::calculate_fields(jblib::Array<double, 2>& fields) {
    for (int i = 0; i < globals::num_spins; ++i) {
        double h[3];

        calculate_one_spin_field(i, h);

        for (int n = 0; n < 3; ++n) {
            fields(i, n) = h[n];
        }
    }
}

void DipoleHamiltonianBruteforce::calculate_fields(jblib::CudaArray<double, 1>& fields) {
    dipole_bruteforce_kernel<<<(globals::num_spins+dev_blocksize_-1)/dev_blocksize_, dev_blocksize_, 0, dev_stream_ >>>
        (solver->dev_ptr_spin(), dev_r_.data(), dev_mus_.data(), globals::num_spins, fields.data());
}

// --------------------------------------------------------------------------

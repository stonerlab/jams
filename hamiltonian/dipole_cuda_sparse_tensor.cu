#include "hip/hip_runtime.h"
#include <cmath>

#include <cblas.h>
#include "core/cuda_defs.h"
#include "core/cuda_array_kernels.h"

#include "core/globals.h"
#include "core/consts.h"
#include "core/utils.h"

#include "hamiltonian/dipole_cuda_sparse_tensor.h"

DipoleHamiltonianCUDASparseTensor::DipoleHamiltonianCUDASparseTensor(const libconfig::Setting &settings)
: HamiltonianStrategy(settings),
    use_double_precision(false)   // default to float precision
 {
    using std::pow;
    double r_abs;
    jblib::Vec3<double> r_ij, r_hat, s_j;

    jblib::Vec3<int> L_max(0, 0, 0);
    jblib::Vec3<double> super_cell_dim(0.0, 0.0, 0.0);

    for (int n = 0; n < 3; ++n) {
        super_cell_dim[n] = 0.5*double(lattice.size(n));
    }

    r_cutoff_ = *std::max_element(super_cell_dim.begin(), super_cell_dim.end());

    if (settings.exists("r_cutoff")) {
        r_cutoff_ = settings["r_cutoff"];
    }


    // printf("  super cell max extent (cartesian):\n    %f %f %f\n", super_cell_dim[0], super_cell_dim[1], super_cell_dim[2]);

    for (int n = 0; n < 3; ++n) {
        if (lattice.is_periodic(n)) {
            L_max[n] = ceil(r_cutoff_/super_cell_dim[n]);
        }
    }

    printf("  image vector max extent (fractional):\n    %d %d %d\n", L_max[0], L_max[1], L_max[2]);

    dev_float_spins_.resize(globals::num_spins3);
    dev_float_fields_.resize(globals::num_spins3);

    interaction_matrix_.resize(globals::num_spins3, globals::num_spins3);

    interaction_matrix_.setMatrixType(SPARSE_MATRIX_TYPE_SYMMETRIC);
    interaction_matrix_.setMatrixMode(SPARSE_FILL_MODE_LOWER);
    
    const double prefactor = kVacuumPermeadbility*kBohrMagneton/(4*kPi*pow(::lattice.parameter(),3));

    jblib::Matrix<double, 3, 3> Id( 1, 0, 0, 0, 1, 0, 0, 0, 1 );


    for (int i = 0; i < globals::num_spins; ++i) {
        for (int j = 0; j < i; ++j) {

            if (j == i) continue;

            auto r_ij = lattice.displacement(i, j);

            const auto r_abs_sq = r_ij.norm_sq();

            if (r_abs_sq > (r_cutoff_*r_cutoff_)) continue;

        const auto r_abs = sqrt(r_abs_sq);

        const auto w0 = prefactor * globals::mus(j) / (r_abs_sq * r_abs_sq * r_abs);

        const jblib::Vec3<double> s_j = {globals::s(j, 0), globals::s(j, 1), globals::s(j, 2)};
        
        const auto s_j_dot_rhat = 3.0 * dot(s_j, r_ij);
	
        r_hat = r_ij / r_abs;

        for (int m = 0; m < 3; ++m) {
            for (int n = 0; n < 3; ++n) {
                if (3 * i + m >= 3 * j + n) {
                    double value = (3*r_hat[m]*r_hat[n] - Id[m][n])*prefactor*globals::mus(i)*globals::mus(j)/(r_abs * r_abs * r_abs);
                                    interaction_matrix_.insertValue(3 * i + m, 3 * j + n, float(value));
              	}
            }
        }
      }

    }

    ::output.write("    converting interaction matrix format from MAP to CSR\n");
    interaction_matrix_.convertMAP2CSR();
    ::output.write("    exchange matrix memory (CSR): %f MB\n", interaction_matrix_.calculateMemory());

    // set up things on the device
    if (solver->is_cuda_solver()) { 
        hipStreamCreate(&dev_stream_);


        hipsparseStatus_t cusparse_return_status;


        ::output.write("    initialising CUSPARSE\n");
        cusparse_return_status = hipsparseCreate(&cusparse_handle_);
        if (cusparse_return_status != HIPSPARSE_STATUS_SUCCESS) {
          jams_error("CUSPARSE Library initialization failed");
        }
        hipsparseSetStream(cusparse_handle_, dev_stream_);


        cusparse_return_status = hipsparseCreateMatDescr(&cusparse_descra_);
        if (cusparse_return_status != HIPSPARSE_STATUS_SUCCESS) {
          jams_error("CUSPARSE Matrix descriptor initialization failed");
        }

        hipsparseSetMatType(cusparse_descra_, HIPSPARSE_MATRIX_TYPE_SYMMETRIC);
        hipsparseSetMatIndexBase(cusparse_descra_, HIPSPARSE_INDEX_BASE_ZERO);

        // row
        ::output.write("    allocating csr row on device\n");
        cuda_api_error_check(
          hipMalloc((void**)&dev_csr_interaction_matrix_.row, (interaction_matrix_.rows()+1)*sizeof(int)));
        
        ::output.write("    memcpy csr row to device\n");
        cuda_api_error_check(hipMemcpy(dev_csr_interaction_matrix_.row, interaction_matrix_.rowPtr(),
              (interaction_matrix_.rows()+1)*sizeof(int), hipMemcpyHostToDevice));

        // col
        ::output.write("    allocating csr col on device\n");
        cuda_api_error_check(
          hipMalloc((void**)&dev_csr_interaction_matrix_.col, (interaction_matrix_.nonZero())*sizeof(int)));

        ::output.write("    memcpy csr col to device\n");
        cuda_api_error_check(hipMemcpy(dev_csr_interaction_matrix_.col, interaction_matrix_.colPtr(),
              (interaction_matrix_.nonZero())*sizeof(int), hipMemcpyHostToDevice));

        // val
        ::output.write("    allocating csr val on device\n");
        cuda_api_error_check(
          hipMalloc((void**)&dev_csr_interaction_matrix_.val, (interaction_matrix_.nonZero())*sizeof(float)));

        ::output.write("    memcpy csr val to device\n");
        cuda_api_error_check(hipMemcpy(dev_csr_interaction_matrix_.val, interaction_matrix_.valPtr(),
              (interaction_matrix_.nonZero())*sizeof(float), hipMemcpyHostToDevice));

    }
}

// --------------------------------------------------------------------------

double DipoleHamiltonianCUDASparseTensor::calculate_total_energy() {
   double e_total = 0.0;
   for (int i = 0; i < globals::num_spins; ++i) {
       e_total += calculate_one_spin_energy(i);
   }
    return 0.5*e_total;
}

// --------------------------------------------------------------------------


double DipoleHamiltonianCUDASparseTensor::calculate_one_spin_energy(const int i, const jblib::Vec3<double> &s_i) {
    double h[3];
    calculate_one_spin_field(i, h);
    return -(s_i[0]*h[0] + s_i[1]*h[1] + s_i[2]*h[2]);
}

// --------------------------------------------------------------------------

double DipoleHamiltonianCUDASparseTensor::calculate_one_spin_energy(const int i) {
    jblib::Vec3<double> s_i(globals::s(i, 0), globals::s(i, 1), globals::s(i, 2));
    return calculate_one_spin_energy(i, s_i);
}

// --------------------------------------------------------------------------

double DipoleHamiltonianCUDASparseTensor::calculate_one_spin_energy_difference(const int i, const jblib::Vec3<double> &spin_initial, const jblib::Vec3<double> &spin_final) {
    double h[3];
    calculate_one_spin_field(i, h);
    double e_initial = -(spin_initial[0]*h[0] + spin_initial[1]*h[1] + spin_initial[2]*h[2]);
    double e_final = -(spin_final[0]*h[0] + spin_final[1]*h[1] + spin_final[2]*h[2]);
    return 0.5*(e_final - e_initial);
}
// --------------------------------------------------------------------------

void DipoleHamiltonianCUDASparseTensor::calculate_energies(jblib::Array<double, 1>& energies) {
    assert(energies.size() == globals::num_spins);
    for (int i = 0; i < globals::num_spins; ++i) {
        energies[i] = calculate_one_spin_energy(i);
    }
}

// --------------------------------------------------------------------------

void DipoleHamiltonianCUDASparseTensor::calculate_one_spin_field(const int i, double h[3]) {
    jams_error("DipoleHamiltonianCUDASparseTensor::calculate_one_spin_field CPU unimplemented");
}


// --------------------------------------------------------------------------

void DipoleHamiltonianCUDASparseTensor::calculate_fields(jblib::Array<double, 2>& fields) {
    jams_error("DipoleHamiltonianCUDASparseTensor::calculate_fields CPU unimplemented");
}

void DipoleHamiltonianCUDASparseTensor::calculate_fields(jblib::CudaArray<double, 1>& fields) {

    // cast spin array to floats
    cuda_array_double_to_float(globals::num_spins3, solver->dev_ptr_spin(), dev_float_spins_.data(), dev_stream_);

    const float one = 1.0;
    const float zero = 0.0;
    hipsparseStatus_t stat =
    hipsparseScsrmv(cusparse_handle_,
      HIPSPARSE_OPERATION_NON_TRANSPOSE,
      globals::num_spins3,
      globals::num_spins3,
      interaction_matrix_.nonZero(),
      &one,
      cusparse_descra_,
      dev_csr_interaction_matrix_.val,
      dev_csr_interaction_matrix_.row,
      dev_csr_interaction_matrix_.col,
      dev_float_spins_.data(),
      &zero,
      dev_float_fields_.data());
    assert(stat == HIPSPARSE_STATUS_SUCCESS);

    cuda_array_float_to_double(globals::num_spins3, dev_float_fields_.data(), fields.data(), dev_stream_);
}

// --------------------------------------------------------------------------

#include "hip/hip_runtime.h"
#include <set>
#include <tuple>

#include "core/exception.h"
#include "core/globals.h"
#include "core/consts.h"
#include "core/cuda_defs.h"
#include "core/cuda_sparsematrix.h"
#include "core/interactions.h"
#include "core/utils.h"

#include "jblib/math/summations.h"

#include "hamiltonian/exchange.h"

//---------------------------------------------------------------------

void ExchangeHamiltonian::insert_interaction(const int i, const int j, const jblib::Matrix<double, 3, 3> &value) {
  for (int m = 0; m < 3; ++m) {
    for (int n = 0; n < 3; ++n) {
      if (std::abs(value[m][n]) > energy_cutoff_) {
        if(interaction_matrix_.getMatrixType() == SPARSE_MATRIX_TYPE_SYMMETRIC) {
          if(interaction_matrix_.getMatrixMode() == SPARSE_FILL_MODE_LOWER) {
            if (i >= j) {
              interaction_matrix_.insertValue(3*i+m, 3*j+n, value[m][n]);
            }
          } else {
            if (i <= j) {
              interaction_matrix_.insertValue(3*i+m, 3*j+n, value[m][n]);
            }
          }
        } else {
          interaction_matrix_.insertValue(3*i+m, 3*j+n, value[m][n]);
        }
      }
    }
  }
}

//---------------------------------------------------------------------

ExchangeHamiltonian::ExchangeHamiltonian(const libconfig::Setting &settings)
: Hamiltonian(settings) {

  //---------------------------------------------------------------------
  // read settings
  //---------------------------------------------------------------------

    is_debug_enabled_ = false;
    settings.lookupValue("debug", is_debug_enabled_);

    std::string interaction_filename = settings.lookup("exc_file").c_str();
    std::ifstream interaction_file(interaction_filename.c_str());
    if (interaction_file.fail()) {
      jams_error("failed to open interaction file %s", interaction_filename.c_str());
    }
    ::output.write("\ninteraction filename (%s)\n", interaction_filename.c_str());

    bool use_symops = true;
    settings.lookupValue("symops", use_symops);

    bool print_unfolded = false;
    settings.lookupValue("print_unfolded", print_unfolded);

    energy_cutoff_ = 1E-26;  // Joules
    settings.lookupValue("energy_cutoff", energy_cutoff_);
    ::output.write("\ninteraction energy cutoff\n  %e\n", energy_cutoff_);

    distance_tolerance_ = 1e-3; // fractional coordinate units
    settings.lookupValue("distance_tolerance", distance_tolerance_);
    ::output.write("\ndistance_tolerance\n  %e\n", distance_tolerance_);
    
    safety_check_distance_tolerance(distance_tolerance_);

    if (is_debug_enabled_) {
      std::ofstream pos_file("debug_pos.dat");
      for (int n = 0; n < lattice.num_materials(); ++n) {
        for (int i = 0; i < globals::num_spins; ++i) {
          if (lattice.atom_material(i) == n) {
            pos_file << i << "\t" <<  lattice.atom_position(i).x << "\t" <<  lattice.atom_position(i).y << "\t" << lattice.atom_position(i).z << "\n";
          }
        }
        pos_file << "\n\n";
      }
      pos_file.close();
    }

    // output in default format for now
    outformat_ = TEXT;


    //---------------------------------------------------------------------
    // generate interaction list
    //---------------------------------------------------------------------
    generate_neighbour_list_from_file(interaction_file, energy_cutoff_, use_symops, print_unfolded || is_debug_enabled_, neighbour_list_);

    if (is_debug_enabled_) {
      std::ofstream debug_file("DEBUG_exchange_nbr_list.tsv");
      write_neighbour_list(debug_file, neighbour_list_);
      debug_file.close();
    }

    //---------------------------------------------------------------------
    // create sparse matrix
    //---------------------------------------------------------------------
   
    interaction_matrix_.resize(globals::num_spins3, globals::num_spins3);

    if (solver->is_cuda_solver()) {
#ifdef CUDA
      interaction_matrix_.setMatrixType(SPARSE_MATRIX_TYPE_GENERAL);
      // interaction_matrix_.setMatrixMode(SPARSE_FILL_MODE_LOWER);
#endif  //CUDA
    } else {
      interaction_matrix_.setMatrixType(SPARSE_MATRIX_TYPE_GENERAL);
    }

    ::output.write("\ncomputed interactions\n");

    for (int i = 0; i < neighbour_list_.size(); ++i) {
      for (auto const &j: neighbour_list_[i]) {
        insert_interaction(i, j.first, j.second);
      }
    }

    // resize member arrays
    energy_.resize(globals::num_spins);
    field_.resize(globals::num_spins, 3);

    ::output.write("  converting interaction matrix format from MAP to CSR\n");
    interaction_matrix_.convertMAP2CSR();
    ::output.write("  exchange matrix memory (CSR): %f MB\n", interaction_matrix_.calculateMemory());

    // transfer arrays to cuda device if needed
    if (solver->is_cuda_solver()) {
#ifdef CUDA

        hipStreamCreate(&dev_stream_);

        dev_energy_ = jblib::CudaArray<double, 1>(energy_);
        dev_field_  = jblib::CudaArray<double, 1>(field_);

        if (interaction_matrix_.getMatrixFormat() == SPARSE_MATRIX_FORMAT_CSR) {
          ::output.write("  * Initialising CUSPARSE...\n");
          hipsparseStatus_t status;
          status = hipsparseCreate(&cusparse_handle_);
          if (status != HIPSPARSE_STATUS_SUCCESS) {
            jams_error("CUSPARSE Library initialization failed");
          }
          hipsparseSetStream(cusparse_handle_, dev_stream_);


          // create matrix descriptor
          status = hipsparseCreateMatDescr(&cusparse_descra_);
          if (status != HIPSPARSE_STATUS_SUCCESS) {
            jams_error("CUSPARSE Matrix descriptor initialization failed");
          }
          hipsparseSetMatType(cusparse_descra_,HIPSPARSE_MATRIX_TYPE_GENERAL);
          hipsparseSetMatIndexBase(cusparse_descra_,HIPSPARSE_INDEX_BASE_ZERO);

          ::output.write("  allocating memory on device\n");
          cuda_api_error_check(
            hipMalloc((void**)&dev_csr_interaction_matrix_.row, (interaction_matrix_.rows()+1)*sizeof(int)));
          cuda_api_error_check(
            hipMalloc((void**)&dev_csr_interaction_matrix_.col, (interaction_matrix_.nonZero())*sizeof(int)));
          cuda_api_error_check(
            hipMalloc((void**)&dev_csr_interaction_matrix_.val, (interaction_matrix_.nonZero())*sizeof(double)));

          cuda_api_error_check(hipMemcpy(dev_csr_interaction_matrix_.row, interaction_matrix_.rowPtr(),
                (interaction_matrix_.rows()+1)*sizeof(int), hipMemcpyHostToDevice));

          cuda_api_error_check(hipMemcpy(dev_csr_interaction_matrix_.col, interaction_matrix_.colPtr(),
                (interaction_matrix_.nonZero())*sizeof(int), hipMemcpyHostToDevice));

          cuda_api_error_check(hipMemcpy(dev_csr_interaction_matrix_.val, interaction_matrix_.valPtr(),
                (interaction_matrix_.nonZero())*sizeof(double), hipMemcpyHostToDevice));

        } else if (interaction_matrix_.getMatrixFormat() == SPARSE_MATRIX_FORMAT_DIA) {
          // ::output.write("  converting interaction matrix format from map to dia");
          // interaction_matrix_.convertMAP2DIA();
          ::output.write("  estimated memory usage (DIA): %f MB\n", interaction_matrix_.calculateMemory());
          dev_dia_interaction_matrix_.blocks = std::min<int>(DIA_BLOCK_SIZE, (globals::num_spins3+DIA_BLOCK_SIZE-1)/DIA_BLOCK_SIZE);
          ::output.write("  allocating memory on device\n");

          // allocate rows
          cuda_api_error_check(
            hipMalloc((void**)&dev_dia_interaction_matrix_.row, (interaction_matrix_.diags())*sizeof(int)));
          // allocate values
          cuda_api_error_check(
            hipMallocPitch((void**)&dev_dia_interaction_matrix_.val, &dev_dia_interaction_matrix_.pitch,
              (interaction_matrix_.rows())*sizeof(double), interaction_matrix_.diags()));
          // copy rows
          cuda_api_error_check(
            hipMemcpy(dev_dia_interaction_matrix_.row, interaction_matrix_.dia_offPtr(),
              (size_t)((interaction_matrix_.diags())*(sizeof(int))), hipMemcpyHostToDevice));
          // convert val array into double which may be float or double
          std::vector<double> float_values(interaction_matrix_.rows()*interaction_matrix_.diags(), 0.0);

          for (int i = 0; i < interaction_matrix_.rows()*interaction_matrix_.diags(); ++i) {
            float_values[i] = static_cast<double>(interaction_matrix_.val(i));
          }

          // copy values
          cuda_api_error_check(
            hipMemcpy2D(dev_dia_interaction_matrix_.val, dev_dia_interaction_matrix_.pitch, &float_values[0],
              interaction_matrix_.rows()*sizeof(double), interaction_matrix_.rows()*sizeof(double),
              interaction_matrix_.diags(), hipMemcpyHostToDevice));

          dev_dia_interaction_matrix_.pitch = dev_dia_interaction_matrix_.pitch/sizeof(double);
        }
#endif
  }

}

// --------------------------------------------------------------------------

double ExchangeHamiltonian::calculate_total_energy() {
    jblib::KahanSum total_energy;

    for (int i = 0; i < globals::num_spins; ++i) {
        total_energy.add(calculate_one_spin_energy(i));
    }
    return total_energy.value();
}

// --------------------------------------------------------------------------

double ExchangeHamiltonian::calculate_one_spin_energy(const int i) {
    using namespace globals;
    assert(interaction_matrix_.getMatrixType() == SPARSE_MATRIX_TYPE_GENERAL);

    double jij_sj[3] = {0.0, 0.0, 0.0};
    const double *val = interaction_matrix_.valPtr();
    const int    *indx = interaction_matrix_.colPtr();
    const int    *ptrb = interaction_matrix_.ptrB();
    const int    *ptre = interaction_matrix_.ptrE();
    const double *x   = s.data();

    for (int m = 0; m < 3; ++m) {
      int begin = ptrb[3*i+m]; int end = ptre[3*i+m];
      for (int j = begin; j < end; ++j) {
        jij_sj[m] = jij_sj[m] + x[ indx[j] ]*val[j];
      }
    }
    return -(s(i,0)*jij_sj[0] + s(i,1)*jij_sj[1] + s(i,2)*jij_sj[2]);
}

// --------------------------------------------------------------------------

double ExchangeHamiltonian::calculate_one_spin_energy_difference(const int i, const jblib::Vec3<double> &spin_initial, const jblib::Vec3<double> &spin_final) {
    assert(interaction_matrix_.getMatrixType() == SPARSE_MATRIX_TYPE_GENERAL);

    double local_field[3], e_initial, e_final;

    calculate_one_spin_field(i, local_field);

    e_initial = -(spin_initial[0]*local_field[0] + spin_initial[1]*local_field[1] + spin_initial[2]*local_field[2]);
    e_final = -(spin_final[0]*local_field[0] + spin_final[1]*local_field[1] + spin_final[2]*local_field[2]);

    return e_final - e_initial;
}

// --------------------------------------------------------------------------

void ExchangeHamiltonian::calculate_energies() {
    for (int i = 0; i < globals::num_spins; ++i) {
        energy_[i] = calculate_one_spin_energy(i);
    }
}

// --------------------------------------------------------------------------

void ExchangeHamiltonian::calculate_one_spin_field(const int i, double local_field[3]) {
    using namespace globals;
    assert(interaction_matrix_.getMatrixType() == SPARSE_MATRIX_TYPE_GENERAL);

    local_field[0] = 0.0, local_field[1] = 0.0; local_field[2] = 0.0;

    const double *val = interaction_matrix_.valPtr();
    const int    *indx = interaction_matrix_.colPtr();
    const int    *ptrb = interaction_matrix_.ptrB();
    const int    *ptre = interaction_matrix_.ptrE();
    const double *x   = s.data();
    int j, m, begin, end;

    for (m = 0; m < 3; ++m) {
      begin = ptrb[3*i+m]; end = ptre[3*i+m];
      for (j = begin; j < end; ++j) {
        // k = indx[j];
        local_field[m] = local_field[m] + x[ indx[j] ]*val[j];
      }
    }
}

// --------------------------------------------------------------------------

void ExchangeHamiltonian::calculate_fields() {
    // dev_s needs to be found from the solver

    if (solver->is_cuda_solver()) {
#ifdef CUDA
      if (interaction_matrix_.getMatrixFormat() == SPARSE_MATRIX_FORMAT_CSR) {
        const double one = 1.0;
        const double zero = 0.0;
        hipsparseStatus_t stat =
        hipsparseDcsrmv(cusparse_handle_,
          HIPSPARSE_OPERATION_NON_TRANSPOSE,
          globals::num_spins3,
          globals::num_spins3,
          interaction_matrix_.nonZero(),
          &one,
          cusparse_descra_,
          dev_csr_interaction_matrix_.val,
          dev_csr_interaction_matrix_.row,
          dev_csr_interaction_matrix_.col,
          solver->dev_ptr_spin(),
          &zero,
          dev_field_.data());
        assert(stat == HIPSPARSE_STATUS_SUCCESS);
      } else if (interaction_matrix_.getMatrixFormat() == SPARSE_MATRIX_FORMAT_DIA) {
        spmv_dia_kernel<<< dev_dia_interaction_matrix_.blocks, DIA_BLOCK_SIZE >>>
            (globals::num_spins3, globals::num_spins3, interaction_matrix_.diags(), dev_dia_interaction_matrix_.pitch, 1.0, 0.0,
            dev_dia_interaction_matrix_.row, dev_dia_interaction_matrix_.val, solver->dev_ptr_spin(), dev_field_.data());
      }
#endif  // CUDA
    } else {
      if (interaction_matrix_.getMatrixType() == SPARSE_MATRIX_TYPE_GENERAL) {
        // general matrix (i.e. Monte Carlo Solvers)
        char transa[1] = {'N'};
        char matdescra[6] = {'G', 'L', 'N', 'C', 'N', 'N'};
#ifdef MKL
        double one = 1.0;
        double zero = 0.0;
        mkl_dcsrmv(transa, &globals::num_spins3, &globals::num_spins3, &one, matdescra, interaction_matrix_.valPtr(),
          interaction_matrix_.colPtr(), interaction_matrix_.ptrB(), interaction_matrix_.ptrE(), globals::s.data(), &zero, field_.data());
#else
        jams_dcsrmv(transa, globals::num_spins3, globals::num_spins3, 1.0, matdescra, interaction_matrix_.valPtr(),
          interaction_matrix_.colPtr(), interaction_matrix_.ptrB(), interaction_matrix_.ptrE(), globals::s.data(), 0.0, field_.data());
#endif
      } else {
        // symmetric matrix (i.e. Heun Solvers)
        char transa[1] = {'N'};
        char matdescra[6] = {'S', 'L', 'N', 'C', 'N', 'N'};
#ifdef MKL
        double one = 1.0;
        double zero = 0.0;
        mkl_dcsrmv(transa, &globals::num_spins3, &globals::num_spins3, &one, matdescra, interaction_matrix_.valPtr(),
          interaction_matrix_.colPtr(), interaction_matrix_.ptrB(), interaction_matrix_.ptrE(), globals::s.data(), &zero, field_.data());
#else
        jams_dcsrmv(transa, globals::num_spins3, globals::num_spins3, 1.0, matdescra, interaction_matrix_.valPtr(),
          interaction_matrix_.colPtr(), interaction_matrix_.ptrB(), interaction_matrix_.ptrE(), globals::s.data(), 0.0, field_.data());
#endif
      }
    }
}
// --------------------------------------------------------------------------

void ExchangeHamiltonian::output_energies(OutputFormat format) {
    switch(format) {
        case TEXT:
            output_energies_text();
        case HDF5:
            jams_error("Exchange energy output: HDF5 not yet implemented");
        default:
            jams_error("Exchange energy output: unknown format");
    }
}

// --------------------------------------------------------------------------

void ExchangeHamiltonian::output_fields(OutputFormat format) {
    switch(format) {
        case TEXT:
            output_fields_text();
        case HDF5:
            jams_error("Exchange energy output: HDF5 not yet implemented");
        default:
            jams_error("Exchange energy output: unknown format");
    }
}

// --------------------------------------------------------------------------

void ExchangeHamiltonian::output_energies_text() {
    using namespace globals;

#ifdef CUDA
    if (globals::is_cuda_solver_used) {
        dev_energy_.copy_to_host_array(energy_);
    }
#endif  // CUDA

    int outcount = 0;

    const std::string filename(seedname+"_eng_uniaxial_"+zero_pad_number(outcount)+".dat");

    std::ofstream outfile(filename.c_str());

    outfile << "# type | rx (nm) | ry (nm) | rz (nm) | d2z | d4z | d6z" << std::endl;

    for (int i = 0; i < globals::num_spins; ++i) {
        // spin type
        outfile << lattice.atom_material(i);

        // real position
        for (int j = 0; j < 3; ++j) {
            outfile <<  lattice.parameter()*lattice.atom_position(i)[j];
        }

        // energy

    }
    outfile.close();
}

// --------------------------------------------------------------------------

void ExchangeHamiltonian::output_fields_text() {

#ifdef CUDA
    if (globals::is_cuda_solver_used) {
        dev_field_.copy_to_host_array(field_);
    }
#endif  // CUDA

    int outcount = 0;

    const std::string filename(seedname+"_field_uniaxial_"+zero_pad_number(outcount)+".dat");

    // using direct file access for performance
    std::ofstream outfile(filename.c_str());
    outfile.setf(std::ios::right);

    outfile << "#";
    outfile << std::setw(16) << "type";
    outfile << std::setw(16) << "rx (nm)";
    outfile << std::setw(16) << "ry (nm)";
    outfile << std::setw(16) << "rz (nm)";
    outfile << std::setw(16) << "hx (nm)";
    outfile << std::setw(16) << "hy (nm)";
    outfile << std::setw(16) << "hz (nm)";
    outfile << "\n";

    for (int i = 0; i < globals::num_spins; ++i) {
        // spin type
        outfile << std::setw(16) << lattice.atom_material(i);

        // real position
        for (int j = 0; j < 3; ++j) {
            outfile << std::setw(16) << std::fixed << lattice.parameter()*lattice.atom_position(i)[j];
        }

        // fields
        for (int j = 0; j < 3; ++j) {
            outfile << std::setw(16) << std::scientific << field_(i,j);
        }
        outfile << "\n";
    }
    outfile.close();
}

sparse_matrix_format_t ExchangeHamiltonian::sparse_matrix_format() {
  return interaction_matrix_format_;
}

//---------------------------------------------------------------------

void ExchangeHamiltonian::set_sparse_matrix_format(std::string &format_name) {
  if (capitalize(format_name) == "CSR") {
    interaction_matrix_format_ = SPARSE_MATRIX_FORMAT_CSR;
  } else if (capitalize(format_name) == "DIA") {
    if (solver->is_cuda_solver() != true) {
      jams_error("ExchangeHamiltonian::set_sparse_matrix_format: DIA format is only supported for CUDA");
    }
    interaction_matrix_format_ = SPARSE_MATRIX_FORMAT_DIA;
  } else {
    jams_error("ExchangeHamiltonian::set_sparse_matrix_format: Unknown format requested %s", format_name.c_str());
  }
}

//---------------------------------------------------------------------

double ExchangeHamiltonian::calculate_bond_energy_difference(const int i, const int j, const Vec3 &sj_initial, const Vec3 &sj_final) {
  using namespace globals;

  return 0.0;
  // if (i == j) {
  //   return 0.0;
  // } else {

  //   // Mat3 J;

  //   // J = neighbour_list_[i][j];

  //   // try {
  //   //   J = neighbour_list_[i].at(j);
  //   // }
  //   // catch(std::out_of_range) {
  //   //   return 0.0;
  //   // }

  //   Vec3 Js = neighbour_list_.interactions(i)[j] * (sj_final - sj_initial);
  //   return -(s(i, 0) * Js[0] + s(i, 1) * Js[1] + s(i, 2) * Js[2]);
  // }
}

#include "hip/hip_runtime.h"
#include <set>

#include "core/globals.h"
#include "core/utils.h"
#include "core/consts.h"
#include "core/cuda_defs.h"
#include "core/cuda_sparsematrix.h"



#include "hamiltonian/exchange_neartree.h"

void ExchangeNeartreeHamiltonian::insert_interaction(const int i, const int j, const jblib::Matrix<double, 3, 3> &value) {
  for (int m = 0; m < 3; ++m) {
    for (int n = 0; n < 3; ++n) {
      if (std::abs(value[m][n]) > energy_cutoff_) {
        if(interaction_matrix_.getMatrixType() == SPARSE_MATRIX_TYPE_SYMMETRIC) {
          if(interaction_matrix_.getMatrixMode() == SPARSE_FILL_MODE_LOWER) {
            if (i >= j) {
              interaction_matrix_.insertValue(3*i+m, 3*j+n, value[m][n]);
            }
          } else {
            if (i <= j) {
              interaction_matrix_.insertValue(3*i+m, 3*j+n, value[m][n]);
            }
          }
        } else {
          interaction_matrix_.insertValue(3*i+m, 3*j+n, value[m][n]);
        }
      }
    }
  }}

ExchangeNeartreeHamiltonian::ExchangeNeartreeHamiltonian(const libconfig::Setting &settings)
: Hamiltonian(settings) {

    is_debug_enabled_ = false;
    std::ofstream debug_file;

    if (settings.exists("debug")) {
      is_debug_enabled_ = settings["debug"];
    }

    // if (settings.exists("sparse_format")) {
    //   set_sparse_matrix_format(std::string(settings["sparse_format"]));
    // }

    if (is_debug_enabled_) {
      debug_file.open("debug_exchange.dat");

      std::ofstream pos_file("debug_pos.dat");
      for (int n = 0; n < lattice.num_materials(); ++n) {
        for (int i = 0; i < globals::num_spins; ++i) {
          if (lattice.atom_material(i) == n) {
            pos_file << i << "\t" <<  lattice.atom_position(i).x << "\t" <<  lattice.atom_position(i).y << "\t" << lattice.atom_position(i).z << "\n";
          }
        }
        pos_file << "\n\n";
      }
      pos_file.close();
    }

    // output in default format for now
    outformat_ = TEXT;

    energy_cutoff_ = 1E-26;  // Joules
    if (settings.exists("energy_cutoff")) {
        energy_cutoff_ = settings["energy_cutoff"];
    }
    ::output.write("\ninteraction energy cutoff\n  %e\n", energy_cutoff_);

    distance_tolerance_ = 1e-3; // fractional coordinate units
    if (settings.exists("distance_tolerance")) {
        distance_tolerance_ = settings["distance_tolerance"];
    }

    ::output.write("\ndistance_tolerance\n  %e\n", distance_tolerance_);

    // --- SAFETY ---
    // check that no atoms in the unit cell are closer together than the distance_tolerance_
    for (int i = 0; i < lattice.num_unit_cell_positions(); ++i) {
      for (int j = i+1; j < lattice.num_unit_cell_positions(); ++j) {
        if( abs(lattice.unit_cell_position(i) - lattice.unit_cell_position(j)) < distance_tolerance_ ) {
          jams_error("Atoms %d and %d in the unit_cell are closer together (%f) than the distance_tolerance (%f).\n"
                     "Check position file or relax distance_tolerance for exchange module",
                      i, j, abs(lattice.unit_cell_position(i) - lattice.unit_cell_position(j)), distance_tolerance_);
        }
      }
    }
    // --------------

    //---------------------------------------------------------------------
    // read interactions from config
    //---------------------------------------------------------------------

    if (!settings.exists("interactions")) {
      jams_error("No interactions defined in ExchangeNeartree hamiltonian");
    }

    int type_id_A, type_id_B;
    std::string type_name_A, type_name_B;
    double jij_radius, jij_value;

    interaction_list_.resize(lattice.num_materials());

    for (int i = 0; i < settings["interactions"].getLength(); ++i) {

      type_name_A = settings["interactions"][i][0].c_str();
      type_name_B = settings["interactions"][i][1].c_str();

      jij_radius = settings["interactions"][i][2];
      jij_value = double(settings["interactions"][i][3]) / kBohrMagneton;

      // std::cout << type_name_A << "\t" << type_name_B << "\t" << jij_radius << "\t" << jij_value << std::endl;

      type_id_A = lattice.material_id(type_name_A);
      type_id_B = lattice.material_id(type_name_B);

      // std::cout << type_id_A << "\t" << type_id_B << "\t" << jij_radius << "\t" << jij_value << std::endl;

      InteractionNT jij = {type_id_A, type_id_B, jij_radius, jij_value};

      interaction_list_[type_id_A].push_back(jij);
    }

    //---------------------------------------------------------------------
    // create interaction matrix
    //---------------------------------------------------------------------

    interaction_matrix_.resize(globals::num_spins3, globals::num_spins3);
    interaction_matrix_.setMatrixType(SPARSE_MATRIX_TYPE_GENERAL);

    ::output.write("\ncomputed interactions\n");

    int counter = 0;
    for (int i = 0; i < globals::num_spins; ++i) {
      std::vector<bool> is_already_interacting(globals::num_spins, false);

      int type = lattice.atom_material(i);

      for (int j = 0; j < interaction_list_[type].size(); ++j) {
        std::vector<Atom> nbr_lower;
        std::vector<Atom> nbr_upper;

        lattice.atom_neighbours(i, interaction_list_[type][j].radius - distance_tolerance_, nbr_lower);
        lattice.atom_neighbours(i, interaction_list_[type][j].radius + distance_tolerance_, nbr_upper);


        std::vector<Atom> nbr(std::max(nbr_lower.size(), nbr_upper.size()));

        auto compare_func = [](Atom a, Atom b) { return a.id > b.id; };

        std::sort(nbr_lower.begin(), nbr_lower.end(), compare_func);
        std::sort(nbr_upper.begin(), nbr_upper.end(), compare_func);

        auto it = std::set_difference(nbr_upper.begin(), nbr_upper.end(), nbr_lower.begin(), nbr_lower.end(), nbr.begin(), compare_func);

        nbr.resize(it - nbr.begin());

        for (const Atom n : nbr) {
          if (n.id == i) {
            continue;
          }

          if (n.material == interaction_list_[type][j].material[1]) {

            // don't allow self interaction
            if (is_already_interacting[n.id]) {
              jams_error("Multiple interactions between spins %d and %d.\n", i, n.id);
            }
            is_already_interacting[n.id] = true;

            double jij = interaction_list_[type][j].value;

            // std::cout << i << "\t" << n.id << "\t" << jij << std::endl;

            insert_interaction(i, n.id, {jij, 0.0, 0.0, 0.0, jij, 0.0, 0.0, 0.0, jij});
            counter++;

            if (is_debug_enabled_) {
              debug_file << i << "\t" << n.id << "\t";
              debug_file << lattice.atom_position(i).x << "\t";
              debug_file << lattice.atom_position(i).y << "\t";
              debug_file << lattice.atom_position(i).z << "\t";
              debug_file << lattice.atom_position(n.id).x << "\t";
              debug_file << lattice.atom_position(n.id).y << "\t";
              debug_file << lattice.atom_position(n.id).z << "\n";
            }
          }
        }
      }
      if (is_debug_enabled_) {
        debug_file << "\n\n";
      }
    }

    if (is_debug_enabled_) {
      debug_file.close();
    }

    ::output.write("  total interactions: %d\n", counter);

    // resize member arrays
    energy_.resize(globals::num_spins);
    field_.resize(globals::num_spins, 3);

    ::output.write("  converting interaction matrix format from MAP to CSR\n");
    interaction_matrix_.convertMAP2CSR();
    ::output.write("  exchange matrix memory (CSR): %f MB\n", interaction_matrix_.calculateMemory());

    //---------------------------------------------------------------------
    // initialize CUDA arrays
    //---------------------------------------------------------------------

    if (solver->is_cuda_solver()) {
#ifdef CUDA

        hipStreamCreate(&dev_stream_);

        dev_energy_ = jblib::CudaArray<double, 1>(energy_);
        dev_field_ = jblib::CudaArray<double, 1>(field_);

        if (interaction_matrix_.getMatrixFormat() == SPARSE_MATRIX_FORMAT_CSR) {
          ::output.write("  * Initialising CUSPARSE...\n");
          hipsparseStatus_t status;
          status = hipsparseCreate(&cusparse_handle_);
          if (status != HIPSPARSE_STATUS_SUCCESS) {
            jams_error("CUSPARSE Library initialization failed");
          }
          hipsparseSetStream(cusparse_handle_, dev_stream_);


          // create matrix descriptor
          status = hipsparseCreateMatDescr(&cusparse_descra_);
          if (status != HIPSPARSE_STATUS_SUCCESS) {
            jams_error("CUSPARSE Matrix descriptor initialization failed");
          }
          hipsparseSetMatType(cusparse_descra_,HIPSPARSE_MATRIX_TYPE_GENERAL);
          hipsparseSetMatIndexBase(cusparse_descra_,HIPSPARSE_INDEX_BASE_ZERO);

          ::output.write("  allocating memory on device\n");
          cuda_api_error_check(
            hipMalloc((void**)&dev_csr_interaction_matrix_.row, (interaction_matrix_.rows()+1)*sizeof(int)));
          cuda_api_error_check(
            hipMalloc((void**)&dev_csr_interaction_matrix_.col, (interaction_matrix_.nonZero())*sizeof(int)));
          cuda_api_error_check(
            hipMalloc((void**)&dev_csr_interaction_matrix_.val, (interaction_matrix_.nonZero())*sizeof(double)));

          cuda_api_error_check(hipMemcpy(dev_csr_interaction_matrix_.row, interaction_matrix_.rowPtr(),
                (interaction_matrix_.rows()+1)*sizeof(int), hipMemcpyHostToDevice));

          cuda_api_error_check(hipMemcpy(dev_csr_interaction_matrix_.col, interaction_matrix_.colPtr(),
                (interaction_matrix_.nonZero())*sizeof(int), hipMemcpyHostToDevice));

          cuda_api_error_check(hipMemcpy(dev_csr_interaction_matrix_.val, interaction_matrix_.valPtr(),
                (interaction_matrix_.nonZero())*sizeof(double), hipMemcpyHostToDevice));

        } else if (interaction_matrix_.getMatrixFormat() == SPARSE_MATRIX_FORMAT_DIA) {
          // ::output.write("  converting interaction matrix format from map to dia");
          // interaction_matrix_.convertMAP2DIA();
          ::output.write("  estimated memory usage (DIA): %f MB\n", interaction_matrix_.calculateMemory());
          dev_dia_interaction_matrix_.blocks = std::min<int>(DIA_BLOCK_SIZE, (globals::num_spins3+DIA_BLOCK_SIZE-1)/DIA_BLOCK_SIZE);
          ::output.write("  allocating memory on device\n");

          // allocate rows
          cuda_api_error_check(
            hipMalloc((void**)&dev_dia_interaction_matrix_.row, (interaction_matrix_.diags())*sizeof(int)));
          // allocate values
          cuda_api_error_check(
            hipMallocPitch((void**)&dev_dia_interaction_matrix_.val, &dev_dia_interaction_matrix_.pitch,
              (interaction_matrix_.rows())*sizeof(double), interaction_matrix_.diags()));
          // copy rows
          cuda_api_error_check(
            hipMemcpy(dev_dia_interaction_matrix_.row, interaction_matrix_.dia_offPtr(),
              (size_t)((interaction_matrix_.diags())*(sizeof(int))), hipMemcpyHostToDevice));
          // convert val array into double which may be float or double
          std::vector<double> float_values(interaction_matrix_.rows()*interaction_matrix_.diags(), 0.0);

          for (int i = 0; i < interaction_matrix_.rows()*interaction_matrix_.diags(); ++i) {
            float_values[i] = static_cast<double>(interaction_matrix_.val(i));
          }

          // copy values
          cuda_api_error_check(
            hipMemcpy2D(dev_dia_interaction_matrix_.val, dev_dia_interaction_matrix_.pitch, &float_values[0],
              interaction_matrix_.rows()*sizeof(double), interaction_matrix_.rows()*sizeof(double),
              interaction_matrix_.diags(), hipMemcpyHostToDevice));

          dev_dia_interaction_matrix_.pitch = dev_dia_interaction_matrix_.pitch/sizeof(double);
        }
#endif
  }

}

// --------------------------------------------------------------------------

double ExchangeNeartreeHamiltonian::calculate_total_energy() {
    double e_total = 0.0;
    for (int i = 0; i < globals::num_spins; ++i) {
        e_total += calculate_one_spin_energy(i);
    }
    return e_total;
}

// --------------------------------------------------------------------------

double ExchangeNeartreeHamiltonian::calculate_one_spin_energy(const int i) {
    using namespace globals;
    assert(interaction_matrix_.getMatrixType() == SPARSE_MATRIX_TYPE_GENERAL);

    double jij_sj[3] = {0.0, 0.0, 0.0};
    const double *val = interaction_matrix_.valPtr();
    const int    *indx = interaction_matrix_.colPtr();
    const int    *ptrb = interaction_matrix_.ptrB();
    const int    *ptre = interaction_matrix_.ptrE();
    const double *x   = s.data();

    for (int m = 0; m < 3; ++m) {
      int begin = ptrb[3*i+m]; int end = ptre[3*i+m];
      for (int j = begin; j < end; ++j) {
        jij_sj[m] = jij_sj[m] + x[ indx[j] ]*val[j];
      }
    }
    return -(s(i,0)*jij_sj[0] + s(i,1)*jij_sj[1] + s(i,2)*jij_sj[2]);
}

// --------------------------------------------------------------------------

double ExchangeNeartreeHamiltonian::calculate_one_spin_energy_difference(const int i, const jblib::Vec3<double> &spin_initial, const jblib::Vec3<double> &spin_final) {
    assert(interaction_matrix_.getMatrixType() == SPARSE_MATRIX_TYPE_GENERAL);

    double local_field[3], e_initial, e_final;

    calculate_one_spin_field(i, local_field);

    e_initial = -(spin_initial[0]*local_field[0] + spin_initial[1]*local_field[1] + spin_initial[2]*local_field[2]);
    e_final = -(spin_final[0]*local_field[0] + spin_final[1]*local_field[1] + spin_final[2]*local_field[2]);

    return e_final - e_initial;
}

// --------------------------------------------------------------------------

void ExchangeNeartreeHamiltonian::calculate_energies() {
    for (int i = 0; i < globals::num_spins; ++i) {
        energy_[i] = calculate_one_spin_energy(i);
    }
}

// --------------------------------------------------------------------------

void ExchangeNeartreeHamiltonian::calculate_one_spin_field(const int i, double local_field[3]) {
    using namespace globals;
    assert(interaction_matrix_.getMatrixType() == SPARSE_MATRIX_TYPE_GENERAL);

    local_field[0] = 0.0, local_field[1] = 0.0; local_field[2] = 0.0;

    const double *val = interaction_matrix_.valPtr();
    const int    *indx = interaction_matrix_.colPtr();
    const int    *ptrb = interaction_matrix_.ptrB();
    const int    *ptre = interaction_matrix_.ptrE();
    const double *x   = s.data();
    int j, m, begin, end;

    for (m = 0; m < 3; ++m) {
      begin = ptrb[3*i+m]; end = ptre[3*i+m];
      for (j = begin; j < end; ++j) {
        // k = indx[j];
        local_field[m] = local_field[m] + x[ indx[j] ]*val[j];
      }
    }
}

// --------------------------------------------------------------------------

void ExchangeNeartreeHamiltonian::calculate_fields() {
    // dev_s needs to be found from the solver

    if (solver->is_cuda_solver()) {
#ifdef CUDA
      if (interaction_matrix_.getMatrixFormat() == SPARSE_MATRIX_FORMAT_CSR) {
        const double one = 1.0;
        const double zero = 0.0;
        hipsparseStatus_t stat =
        hipsparseDcsrmv(cusparse_handle_,
          HIPSPARSE_OPERATION_NON_TRANSPOSE,
          globals::num_spins3,
          globals::num_spins3,
          interaction_matrix_.nonZero(),
          &one,
          cusparse_descra_,
          dev_csr_interaction_matrix_.val,
          dev_csr_interaction_matrix_.row,
          dev_csr_interaction_matrix_.col,
          solver->dev_ptr_spin(),
          &zero,
          dev_field_.data());
        assert(stat == HIPSPARSE_STATUS_SUCCESS);
      } else if (interaction_matrix_.getMatrixFormat() == SPARSE_MATRIX_FORMAT_DIA) {
        spmv_dia_kernel<<< dev_dia_interaction_matrix_.blocks, DIA_BLOCK_SIZE >>>
            (globals::num_spins3, globals::num_spins3, interaction_matrix_.diags(), dev_dia_interaction_matrix_.pitch, 1.0, 0.0,
            dev_dia_interaction_matrix_.row, dev_dia_interaction_matrix_.val, solver->dev_ptr_spin(), dev_field_.data());
      }
#endif  // CUDA
    } else {
      if (interaction_matrix_.getMatrixType() == SPARSE_MATRIX_TYPE_GENERAL) {
        // general matrix (i.e. Monte Carlo Solvers)
        char transa[1] = {'N'};
        char matdescra[6] = {'G', 'L', 'N', 'C', 'N', 'N'};
#ifdef MKL
        double one = 1.0;
        double zero = 0.0;
        mkl_dcsrmv(transa, &globals::num_spins3, &globals::num_spins3, &one, matdescra, interaction_matrix_.valPtr(),
          interaction_matrix_.colPtr(), interaction_matrix_.ptrB(), interaction_matrix_.ptrE(), globals::s.data(), &zero, field_.data());
#else
        jams_dcsrmv(transa, globals::num_spins3, globals::num_spins3, 1.0, matdescra, interaction_matrix_.valPtr(),
          interaction_matrix_.colPtr(), interaction_matrix_.ptrB(), interaction_matrix_.ptrE(), globals::s.data(), 0.0, field_.data());
#endif
      } else {
        // symmetric matrix (i.e. Heun Solvers)
        char transa[1] = {'N'};
        char matdescra[6] = {'S', 'L', 'N', 'C', 'N', 'N'};
#ifdef MKL
        double one = 1.0;
        double zero = 0.0;
        mkl_dcsrmv(transa, &globals::num_spins3, &globals::num_spins3, &one, matdescra, interaction_matrix_.valPtr(),
          interaction_matrix_.colPtr(), interaction_matrix_.ptrB(), interaction_matrix_.ptrE(), globals::s.data(), &zero, field_.data());
#else
        jams_dcsrmv(transa, globals::num_spins3, globals::num_spins3, 1.0, matdescra, interaction_matrix_.valPtr(),
          interaction_matrix_.colPtr(), interaction_matrix_.ptrB(), interaction_matrix_.ptrE(), globals::s.data(), 0.0, field_.data());
#endif
      }
    }
}
// --------------------------------------------------------------------------

void ExchangeNeartreeHamiltonian::output_energies(OutputFormat format) {
    switch(format) {
        case TEXT:
            output_energies_text();
        case HDF5:
            jams_error("Exchange energy output: HDF5 not yet implemented");
        default:
            jams_error("Exchange energy output: unknown format");
    }
}

// --------------------------------------------------------------------------

void ExchangeNeartreeHamiltonian::output_fields(OutputFormat format) {
    switch(format) {
        case TEXT:
            output_fields_text();
        case HDF5:
            jams_error("Exchange energy output: HDF5 not yet implemented");
        default:
            jams_error("Exchange energy output: unknown format");
    }
}

// --------------------------------------------------------------------------

void ExchangeNeartreeHamiltonian::output_energies_text() {
    using namespace globals;

#ifdef CUDA
    if (globals::is_cuda_solver_used) {
        dev_energy_.copy_to_host_array(energy_);
    }
#endif  // CUDA

    int outcount = 0;

    const std::string filename(seedname+"_eng_uniaxial_"+zero_pad_number(outcount)+".dat");

    std::ofstream outfile(filename.c_str());

    outfile << "# type | rx (nm) | ry (nm) | rz (nm) | d2z | d4z | d6z" << std::endl;

    for (int i = 0; i < globals::num_spins; ++i) {
        // spin type
        outfile << lattice.atom_material(i);

        // real position
        for (int j = 0; j < 3; ++j) {
            outfile <<  lattice.parameter()*lattice.atom_position(i)[j];
        }

        // energy

    }
    outfile.close();
}

// --------------------------------------------------------------------------

void ExchangeNeartreeHamiltonian::output_fields_text() {

#ifdef CUDA
    if (globals::is_cuda_solver_used) {
        dev_field_.copy_to_host_array(field_);
    }
#endif  // CUDA

    int outcount = 0;

    const std::string filename(seedname+"_field_uniaxial_"+zero_pad_number(outcount)+".dat");

    // using direct file access for performance
    std::ofstream outfile(filename.c_str());
    outfile.setf(std::ios::right);

    outfile << "#";
    outfile << std::setw(16) << "type";
    outfile << std::setw(16) << "rx (nm)";
    outfile << std::setw(16) << "ry (nm)";
    outfile << std::setw(16) << "rz (nm)";
    outfile << std::setw(16) << "hx (nm)";
    outfile << std::setw(16) << "hy (nm)";
    outfile << std::setw(16) << "hz (nm)";
    outfile << "\n";

    for (int i = 0; i < globals::num_spins; ++i) {
        // spin type
        outfile << std::setw(16) << lattice.atom_material(i);

        // real position
        for (int j = 0; j < 3; ++j) {
            outfile << std::setw(16) << std::fixed << lattice.parameter()*lattice.atom_position(i)[j];
        }

        // fields
        for (int j = 0; j < 3; ++j) {
            outfile << std::setw(16) << std::scientific << field_(i,j);
        }
        outfile << "\n";
    }
    outfile.close();
}

sparse_matrix_format_t ExchangeNeartreeHamiltonian::sparse_matrix_format() {
  return interaction_matrix_format_;
}

void ExchangeNeartreeHamiltonian::set_sparse_matrix_format(std::string &format_name) {
  if (capitalize(format_name) == "CSR") {
    interaction_matrix_format_ = SPARSE_MATRIX_FORMAT_CSR;
  } else if (capitalize(format_name) == "DIA") {
    if (solver->is_cuda_solver() != true) {
      jams_error("ExchangeNeartreeHamiltonian::set_sparse_matrix_format: DIA format is only supported for CUDA");
    }
    interaction_matrix_format_ = SPARSE_MATRIX_FORMAT_DIA;
  } else {
    jams_error("ExchangeNeartreeHamiltonian::set_sparse_matrix_format: Unknown format requested %s", format_name.c_str());
  }
}

double ExchangeNeartreeHamiltonian::calculate_bond_energy_difference(const int i, const int j, const Vec3 &sj_initial, const Vec3 &sj_final) {
  using namespace globals;

  if (i == j) {
    return 0.0;
  } else {
    return 0.0;
  }
}

#include "hip/hip_runtime.h"
// Copyright 2014 Joseph Barker. All rights reserved.

#define __STDC_FORMAT_MACROS
#include <inttypes.h>

#include <cmath>
#include <string>
#include <iomanip>

#include "thermostats/cuda_langevin_bose.h"
#include "thermostats/cuda_langevin_bose_kernel.h"

#include "core/globals.h"
#include "core/lattice.h"
#include "core/consts.h"

#include "monitors/magnetisation.h"

CudaLangevinBoseThermostat::CudaLangevinBoseThermostat(const double &temperature, const double &sigma, const int num_spins)
: Thermostat(temperature, sigma, num_spins),
  debug_(false),
  dev_noise_(3*num_spins) {
  ::output.write("\n  initialising CUDA Langevin semi-quantum noise thermostat\n");

  debug_ = true;

  if (debug_) {
    ::output.write("    DEBUG ON\n");
    std::string name = seedname + "_noise.dat";
    outfile_.open(name.c_str());
  }

  w_max_ = 50*kTHz;

  const double dt = ::config.lookup("sim.t_step");
  tau_ = (dt * kBoltzmann) / kHBar;

  ::output.write("    omega_max = %6.6f (THz)\n", w_max_ / kTHz);
  ::output.write("    hbar*w/kB = %4.4e\n", (kHBar * w_max_) / (kBoltzmann));
  ::output.write("    delta tau = %4.4e * T\n", tau_);

  ::output.write("    initialising CUDA streams\n");

  if (hipStreamCreate(&dev_stream_) != hipSuccess){
    jams_error("Failed to create CUDA stream in CudaLangevinBoseThermostat");
  }

  ::output.write("    initialising CURAND\n");

  // initialize and seed the CURAND generator on the device
  if (hiprandCreateGenerator(&dev_rng_, HIPRAND_RNG_PSEUDO_DEFAULT) != HIPRAND_STATUS_SUCCESS) {
    jams_error("Failed to create CURAND generator in CudaLangevinBoseThermostat");
  }

  const uint64_t dev_rng_seed = rng.uniform()*18446744073709551615ULL;
  ::output.write("    seeding CURAND (%" PRIu64 ")\n", dev_rng_seed);

  if (hiprandSetPseudoRandomGeneratorSeed(dev_rng_, dev_rng_seed) != HIPRAND_STATUS_SUCCESS) {
    jams_error("Failed to set CURAND seed in CudaLangevinBoseThermostat");
  }

  if (hiprandGenerateSeeds(dev_rng_) != HIPRAND_STATUS_SUCCESS) {
    jams_error("Failed to generate CURAND seeds in CudaLangevinBoseThermostat");
  }

  ::output.write("    allocating GPU memory\n");
  dev_eta_.resize(6*globals::num_spins3);
  dev_zeta_.resize(8*globals::num_spins3);

  // initialize zeta and eta with random variables
  hiprandSetStream(dev_rng_, dev_stream_);
  if (hiprandGenerateNormalDouble(dev_rng_, dev_eta_.data(), dev_eta_.size(), 0.0, 1.0)
       != HIPRAND_STATUS_SUCCESS) {
    jams_error("hiprandGenerateNormalDouble failure in CudaLangevinBoseThermostat::constructor");
  }

  if (hiprandGenerateNormalDouble(dev_rng_, dev_zeta_.data(), dev_zeta_.size(), 0.0, 0.0)
       != HIPRAND_STATUS_SUCCESS) {
    jams_error("hiprandGenerateNormalDouble failure in CudaLangevinBoseThermostat::constructor");
  }
}

void CudaLangevinBoseThermostat::update() {
  int block_size = 64;
  int grid_size = (globals::num_spins3 + block_size - 1) / block_size;

  // if (hiprandGenerateNormalDouble(dev_rng_, dev_eta_.data(), dev_eta_.size(), 0.0, sqrt(this->temperature())) != HIPRAND_STATUS_SUCCESS) {
  //   jams_error("hiprandGenerateNormalDouble failure in CudaLangevinBoseThermostat::update");
  // }

  const double w_m = (kHBar * w_max_) / (kBoltzmann * this->temperature());
  // const double reduced_temperature = sqrt(this->temperature()) ;
  const double reduced_temperature = this->temperature() * sqrt( (2.0 * kBoltzmann * globals::alpha(0) * globals::mus(0)) / (kBohrMagneton) );


  hiprandGenerateNormalDouble(dev_rng_, dev_eta_.data(), dev_eta_.size(), 0.0, 1.0);
  bose_stochastic_process_cuda_kernel<<<grid_size, block_size, 0, dev_stream_ >>> (dev_noise_.data(), dev_zeta_.data(), dev_eta_.data(), tau_ * this->temperature(), reduced_temperature, w_m, globals::num_spins3);

  if (debug_) {
    jblib::Array<double, 1> dbg_noise(dev_noise_.size(), 0.0);
    dev_noise_.copy_to_host_array(dbg_noise);
    outfile_ << dbg_noise(0) << std::endl;
  }
}

CudaLangevinBoseThermostat::~CudaLangevinBoseThermostat() {
  hiprandDestroyGenerator(dev_rng_);
  hipStreamDestroy(dev_stream_);
  if (debug_) {
    outfile_.close();
  }
}

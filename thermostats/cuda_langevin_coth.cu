#include "hip/hip_runtime.h"
// Copyright 2014 Joseph Barker. All rights reserved.

#define __STDC_FORMAT_MACROS
#include <inttypes.h>

#include <cmath>
#include <string>
#include <iomanip>

#include "thermostats/cuda_langevin_coth.h"
#include "thermostats/cuda_langevin_coth_kernel.h"

#include "core/globals.h"
#include "core/lattice.h"

#include "monitors/magnetisation.h"

CudaLangevinCothThermostat::CudaLangevinCothThermostat(const double &temperature, const double &sigma, const int num_spins)
: Thermostat(temperature, sigma, num_spins),
  debug_(false),
  dev_noise_(3*num_spins) {
  ::output.write("\n  initialising CUDA Langevin semi-quantum noise thermostat\n");

  ::output.write("    initialising CUDA streams\n");

  dev_streams_ = new hipStream_t [2];
  for (int i = 0; i < 2; ++i) {
    if (hipStreamCreate(&dev_streams_[i]) != hipSuccess){
      jams_error("Failed to create CUDA streams in CudaLangevinCothThermostat");
    }
  }

  ::output.write("    initialising CURAND\n");

  // initialize and seed the CURAND generator on the device
  if (hiprandCreateGenerator(&dev_rng_, HIPRAND_RNG_PSEUDO_DEFAULT) != HIPRAND_STATUS_SUCCESS) {
    jams_error("Failed to create CURAND generator in CudaLangevinCothThermostat");
  }

  const uint64_t dev_rng_seed = rng.uniform()*18446744073709551615ULL;
  ::output.write("    seeding CURAND (%" PRIu64 ")", dev_rng_seed);

  if (hiprandSetPseudoRandomGeneratorSeed(dev_rng_, dev_rng_seed) != HIPRAND_STATUS_SUCCESS) {
    jams_error("Failed to set CURAND seed in CudaLangevinCothThermostat");
  }

  if (hiprandGenerateSeeds(dev_rng_) != HIPRAND_STATUS_SUCCESS) {
    jams_error("Failed to generate CURAND seeds in CudaLangevinCothThermostat");
  }

  ::output.write("    allocating GPU memory\n");
  dev_eta_.resize(6*globals::num_spins3);
  dev_zeta_.resize(8*globals::num_spins3);

  // initialize zeta and eta with random variables
  if (hiprandGenerateNormalDouble(dev_rng_, dev_eta_.data(), dev_eta_.size(), 0.0, sqrt(this->temperature()))
       != HIPRAND_STATUS_SUCCESS) {
    jams_error("hiprandGenerateNormalDouble failure in CudaLangevinCothThermostat::constructor");
  }

  if (hiprandGenerateNormalDouble(dev_rng_, dev_zeta_.data(), dev_zeta_.size(), 0.0, 0.0)
       != HIPRAND_STATUS_SUCCESS) {
    jams_error("hiprandGenerateNormalDouble failure in CudaLangevinCothThermostat::constructor");
  }

}

void CudaLangevinCothThermostat::update() {

  // if (hiprandGenerateNormalDouble(dev_rng_, dev_eta_linear_.data(), dev_eta_linear_.size(), 0.0, sqrt(this->temperature()))
  //      != HIPRAND_STATUS_SUCCESS) {
  //   jams_error("hiprandGenerateNormalDouble failure in CudaLangevinCothThermostat::update");
  // }

  // if (hiprandGenerateNormalDouble(dev_rng_, dev_eta_bose_.data(), dev_eta_bose_.size(), 0.0, sqrt(this->temperature()))
  //      != HIPRAND_STATUS_SUCCESS) {
  //   jams_error("hiprandGenerateNormalDouble failure in CudaLangevinCothThermostat::update");
  // }


  hiprandSetStream(dev_rng_, dev_streams_[0]);
  hiprandGenerateNormalDouble(dev_rng_, dev_eta_.data(), dev_eta_.size(), 0.0, sqrt(this->temperature()));

  // dim3 block_size, grid_size;

  // block_size.x = 4;
  // block_size.y = 64;
  // grid_size.x = (4 + block_size.x - 1) / block_size.x;
  // grid_size.y = (globals::num_spins3 + block_size.y - 1) / block_size.y;


  int block_size = 64;
  int grid_size = (globals::num_spins3 + block_size - 1) / block_size;

  coth_stochastic_process_cuda_kernel<<<grid_size, block_size, 0, dev_streams_[0] >>> (dev_noise_.data(), dev_zeta_.data(), dev_eta_.data(), 0.0005);

  // linear_stochastic_process_cuda_kernel<<<grid_size, block_size, 0, dev_streams_[0]>>>(dev_zeta_linear_.data(), dev_eta_linear_.data(), 0.0005);

  // block_size.x = 2;
  // block_size.y = 32;
  // grid_size.x = (2 + block_size.x - 1) / block_size.x;
  // grid_size.y = (globals::num_spins3 + block_size.y - 1) / block_size.y;
  // bose_stochastic_process_cuda_kernel<<<grid_size, block_size, 0, dev_streams_[1]>>>(dev_zeta_bose_.data(), dev_eta_bose_.data(), 0.0005);

  //hipDeviceSynchronize();

  // hipStreamSynchronize(dev_streams_[0]);
  // hipStreamSynchronize(dev_streams_[1]);

  // block_size.x = 32;
  // block_size.y = 1;
  // grid_size.x = globals::num_spins3 / block_size.x;
  // grid_size.y = 1;
  // combine_stochastic_process_cuda_kernel<<<grid_size, block_size, 0, dev_streams_[0]>>>(dev_noise_.data(), dev_zeta_linear_.data(), dev_zeta_bose_.data(), dev_eta_linear_.data());
}

CudaLangevinCothThermostat::~CudaLangevinCothThermostat() {
  hiprandDestroyGenerator(dev_rng_);
  for (int i = 0; i < 2; ++i) {
    hipStreamDestroy(dev_streams_[i]);
  }
}

#include "hip/hip_runtime.h"
// Copyright 2014 Joseph Barker. All rights reserved.

#define __STDC_FORMAT_MACROS
#include <inttypes.h>

#include <cmath>
#include <string>
#include <iomanip>

#include "thermostats/cuda_langevin_coth.h"
#include "thermostats/cuda_langevin_coth_kernel.h"

#include "core/globals.h"
#include "core/lattice.h"

#include "monitors/magnetisation.h"

CudaLangevinCothThermostat::CudaLangevinCothThermostat(const double &temperature, const double &sigma, const int num_spins)
: Thermostat(temperature, sigma, num_spins),
  debug_(false),
  dev_noise_(3*num_spins) {
  ::output.write("\n  initialising CUDA Langevin semi-quantum noise thermostat\n");

  ::output.write("    initialising CUDA streams\n");

  if (hipStreamCreate(&dev_stream_) != hipSuccess){
    jams_error("Failed to create CUDA stream in CudaLangevinCothThermostat");
  }

  ::output.write("    initialising CURAND\n");

  // initialize and seed the CURAND generator on the device
  if (hiprandCreateGenerator(&dev_rng_, HIPRAND_RNG_PSEUDO_DEFAULT) != HIPRAND_STATUS_SUCCESS) {
    jams_error("Failed to create CURAND generator in CudaLangevinCothThermostat");
  }

  const uint64_t dev_rng_seed = rng.uniform()*18446744073709551615ULL;
  ::output.write("    seeding CURAND (%" PRIu64 ")", dev_rng_seed);

  if (hiprandSetPseudoRandomGeneratorSeed(dev_rng_, dev_rng_seed) != HIPRAND_STATUS_SUCCESS) {
    jams_error("Failed to set CURAND seed in CudaLangevinCothThermostat");
  }

  if (hiprandGenerateSeeds(dev_rng_) != HIPRAND_STATUS_SUCCESS) {
    jams_error("Failed to generate CURAND seeds in CudaLangevinCothThermostat");
  }

  ::output.write("    allocating GPU memory\n");
  dev_eta_.resize(6*globals::num_spins3);
  dev_zeta_.resize(8*globals::num_spins3);

  // initialize zeta and eta with random variables
  hiprandSetStream(dev_rng_, dev_stream_);
  if (hiprandGenerateNormalDouble(dev_rng_, dev_eta_.data(), dev_eta_.size(), 0.0, sqrt(this->temperature()))
       != HIPRAND_STATUS_SUCCESS) {
    jams_error("hiprandGenerateNormalDouble failure in CudaLangevinCothThermostat::constructor");
  }

  if (hiprandGenerateNormalDouble(dev_rng_, dev_zeta_.data(), dev_zeta_.size(), 0.0, 0.0)
       != HIPRAND_STATUS_SUCCESS) {
    jams_error("hiprandGenerateNormalDouble failure in CudaLangevinCothThermostat::constructor");
  }

}

void CudaLangevinCothThermostat::update() {
  int block_size = 64;
  int grid_size = (globals::num_spins3 + block_size - 1) / block_size;

  // if (hiprandGenerateNormalDouble(dev_rng_, dev_eta_.data(), dev_eta_.size(), 0.0, sqrt(this->temperature())) != HIPRAND_STATUS_SUCCESS) {
  //   jams_error("hiprandGenerateNormalDouble failure in CudaLangevinCothThermostat::update");
  // }
  hiprandGenerateNormalDouble(dev_rng_, dev_eta_.data(), dev_eta_.size(), 0.0, sqrt(this->temperature()));
  coth_stochastic_process_cuda_kernel<<<grid_size, block_size, 0, dev_stream_ >>> (dev_noise_.data(), dev_zeta_.data(), dev_eta_.data(), 0.0005, globals::num_spins3);
}

CudaLangevinCothThermostat::~CudaLangevinCothThermostat() {
  hiprandDestroyGenerator(dev_rng_);
  hipStreamDestroy(dev_stream_);
}

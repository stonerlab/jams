#include "hip/hip_runtime.h"
// Copyright 2014 Joseph Barker. All rights reserved.

#define __STDC_FORMAT_MACROS
#include <inttypes.h>

#include <cmath>
#include <string>
#include <iomanip>

#include "thermostats/cuda_langevin_coth.h"
#include "thermostats/cuda_langevin_coth_kernel.h"

#include "core/globals.h"
#include "core/lattice.h"
#include "core/consts.h"

#include "monitors/magnetisation.h"

CudaLangevinCothThermostat::CudaLangevinCothThermostat(const double &temperature, const double &sigma, const int num_spins)
: Thermostat(temperature, sigma, num_spins),
  debug_(false),
  dev_noise_(3*num_spins) {
  ::output.write("\n  initialising CUDA Langevin semi-quantum noise thermostat\n");

  if (debug_) {
    ::output.write("    DEBUG ON\n");
    std::string name = seedname + "_noise.dat";
    outfile.open(name.c_str());
  }

  w_max = 100*1E12;

  ::output.write("    omega_max = %6.6f (THz)\n", w_max/1E12);
  ::output.write("    hbar*w/kB = %4.4e\n", (hbar_si * w_max) / (boltzmann_si));
  ::output.write("    delta tau = %4.4e * T\n", (1e-16 * boltzmann_si) / hbar_si);

  ::output.write("    initialising CUDA streams\n");

  if (hipStreamCreate(&dev_stream_) != hipSuccess){
    jams_error("Failed to create CUDA stream in CudaLangevinCothThermostat");
  }

  ::output.write("    initialising CURAND\n");

  // initialize and seed the CURAND generator on the device
  if (hiprandCreateGenerator(&dev_rng_, HIPRAND_RNG_PSEUDO_DEFAULT) != HIPRAND_STATUS_SUCCESS) {
    jams_error("Failed to create CURAND generator in CudaLangevinCothThermostat");
  }

  const uint64_t dev_rng_seed = rng.uniform()*18446744073709551615ULL;
  ::output.write("    seeding CURAND (%" PRIu64 ")\n", dev_rng_seed);

  if (hiprandSetPseudoRandomGeneratorSeed(dev_rng_, dev_rng_seed) != HIPRAND_STATUS_SUCCESS) {
    jams_error("Failed to set CURAND seed in CudaLangevinCothThermostat");
  }

  if (hiprandGenerateSeeds(dev_rng_) != HIPRAND_STATUS_SUCCESS) {
    jams_error("Failed to generate CURAND seeds in CudaLangevinCothThermostat");
  }

  ::output.write("    allocating GPU memory\n");
  dev_eta_.resize(6*globals::num_spins3);
  dev_zeta_.resize(8*globals::num_spins3);

  // initialize zeta and eta with random variables
  hiprandSetStream(dev_rng_, dev_stream_);
  if (hiprandGenerateNormalDouble(dev_rng_, dev_eta_.data(), dev_eta_.size(), 0.0, sqrt(this->temperature()))
       != HIPRAND_STATUS_SUCCESS) {
    jams_error("hiprandGenerateNormalDouble failure in CudaLangevinCothThermostat::constructor");
  }

  if (hiprandGenerateNormalDouble(dev_rng_, dev_zeta_.data(), dev_zeta_.size(), 0.0, 0.0)
       != HIPRAND_STATUS_SUCCESS) {
    jams_error("hiprandGenerateNormalDouble failure in CudaLangevinCothThermostat::constructor");
  }

}

void CudaLangevinCothThermostat::update() {
  int block_size = 64;
  int grid_size = (globals::num_spins3 + block_size - 1) / block_size;

  // if (hiprandGenerateNormalDouble(dev_rng_, dev_eta_.data(), dev_eta_.size(), 0.0, sqrt(this->temperature())) != HIPRAND_STATUS_SUCCESS) {
  //   jams_error("hiprandGenerateNormalDouble failure in CudaLangevinCothThermostat::update");
  // }

  const double w_m = (hbar_si * w_max) / (boltzmann_si * this->temperature());
  const double d_tau = (1e-16 * boltzmann_si * this->temperature()) / hbar_si;
  const double reduced_temperature = sqrt(this->temperature());
  hiprandGenerateNormalDouble(dev_rng_, dev_eta_.data(), dev_eta_.size(), 0.0, 1.0);
  coth_stochastic_process_cuda_kernel<<<grid_size, block_size, 0, dev_stream_ >>> (dev_noise_.data(), dev_zeta_.data(), dev_eta_.data(), d_tau, reduced_temperature, w_m, globals::num_spins3);

  if (debug_) {
    jblib::Array<double, 1> dbg_noise(dev_noise_.size(), 0.0);
    dev_noise_.copy_to_host_array(dbg_noise);
    outfile << dbg_noise(0) << std::endl;
  }
}

CudaLangevinCothThermostat::~CudaLangevinCothThermostat() {
  hiprandDestroyGenerator(dev_rng_);
  hipStreamDestroy(dev_stream_);
  if (debug_) {
    outfile.close();
  }
}

#include "hip/hip_runtime.h"
// Copyright 2014 Joseph Barker. All rights reserved.

#include "core/globals.h"
#include "core/solver.h"
#include "core/cuda_solver.h"
#include "core/cuda_solver_kernels.h"


#include "core/consts.h"

#include "core/utils.h"
#include "core/cuda_defs.h"

#include "solvers/cuda_heunllg.h"
#include "solvers/heunllg.h"
#include "solvers/metropolismc.h"
#include "core/cuda_sparsematrix.h"


void CudaSolver::initialize(int argc, char **argv, double idt) {
  using namespace globals;

  Solver::initialize(argc, argv, idt);

  ::output.write("\ninitializing CUDA base solver\n");

  ::output.write("  initialising CUDA streams\n");

  dev_streams_ = new hipStream_t[2];

  for (int i = 0; i < 2; ++i) {
    if (hipStreamCreate(&dev_streams_[i]) != hipSuccess){
      jams_error("Failed to create CUDA stream in CudaLangevinCothThermostat");
    }
  }

  ::output.write("  converting J1ij_t format from map to dia");
  J1ij_t.convertMAP2DIA();

  ::output.write("  estimated memory usage (dia): %f MB\n", J1ij_t.calculateMemory());
  dev_J1ij_t_.blocks = std::min<int>(DIA_BLOCK_SIZE, (num_spins3+DIA_BLOCK_SIZE-1)/DIA_BLOCK_SIZE);

  ::output.write("  allocating memory on device\n");

//-----------------------------------------------------------------------------
// fourier transforms
//-----------------------------------------------------------------------------

  for (int i = 0; i < 3; ++i) {
    num_kpoints_[i] = globals::wij.size(i);
  }

  jblib::Vec3<int> num_hermitian_kpoints = num_kpoints_;
  num_hermitian_kpoints.z = (num_kpoints_.z/2) + 1;

  globals::wq.resize(num_kpoints_.x, num_kpoints_.y, (num_kpoints_.z/2)+1, 3, 3);

  ::output.write("  kspace dimensions: %d %d %d\n", num_kpoints_.x, num_kpoints_.y, num_kpoints_.z);

  ::output.write("  FFT planning\n");

  // perform the wij -> wq transformation on the host
  fftw_plan interaction_fft_transform  = fftw_plan_many_dft_r2c(3, &num_kpoints_[0], 9, wij.data(),  NULL, 9, 1, wq.data(), NULL, 9, 1, FFTW_ESTIMATE|FFTW_PRESERVE_INPUT);
  ::output.write("  FFT transform interaction matrix\n");
  fftw_execute(interaction_fft_transform);

  ::output.write("  FFT transfering arrays to device\n");

  // convert fftw_complex data into hipfftDoubleComplex format and copy to the device
  jblib::Array<hipfftDoubleComplex, 5> convert_wq(num_hermitian_kpoints.x, num_hermitian_kpoints.y, num_hermitian_kpoints.z, 3, 3);

  for (int i = 0; i < globals::wq.elements(); ++i) {
    convert_wq[i].x = globals::wq[i][0];
    convert_wq[i].y = globals::wq[i][1];
  }
  dev_wq_ = jblib::CudaArray<hipfftDoubleComplex, 1>(convert_wq);

  jblib::Array<double, 4> s3d(num_kpoints_.x, num_kpoints_.y, num_kpoints_.z, 3, 0.0);
  dev_s3d_ = jblib::CudaArray<double, 1>(s3d);
  dev_h3d_ = jblib::CudaArray<double, 1>(s3d);

  dev_sq_.resize(num_hermitian_kpoints.x*num_hermitian_kpoints.y*num_hermitian_kpoints.z*3);
  dev_hq_.resize(num_hermitian_kpoints.x*num_hermitian_kpoints.y*num_hermitian_kpoints.z*3);

  r_to_k_mapping_ = jblib::CudaArray<int, 1>(lattice.kspace_inv_map_);


  if (hipfftPlanMany(&spin_fft_forward_transform, 3, &num_kpoints_[0], &num_kpoints_[0], 3, 1, &num_hermitian_kpoints[0], 3, 1, HIPFFT_D2Z, 3) != HIPFFT_SUCCESS) {
    jams_error("CUFFT failure planning spin_fft_forward_transform");
  }
  if (cufftSetCompatibilityMode(spin_fft_forward_transform, CUFFT_COMPATIBILITY_NATIVE) != HIPFFT_SUCCESS) {
    jams_error("CUFFT failure changing to compatability mode native for spin_fft_forward_transform");
  }
  if (hipfftPlanMany(&field_fft_backward_transform, 3, &num_kpoints_[0], &num_hermitian_kpoints[0], 3, 1, &num_kpoints_[0], 3, 1, HIPFFT_Z2D, 3) != HIPFFT_SUCCESS) {
    jams_error("CUFFT failure planning field_fft_backward_transform");
  }
  if (cufftSetCompatibilityMode(field_fft_backward_transform, CUFFT_COMPATIBILITY_NATIVE) != HIPFFT_SUCCESS) {
    jams_error("CUFFT failure changing to compatability mode native for field_fft_backward_transform");
  }


//-----------------------------------------------------------------------------
// transfer sparse matrix to device - optionally converting double precision to
// single
//-----------------------------------------------------------------------------


  // allocate rows
  CUDA_CALL(hipMalloc((void**)&dev_J1ij_t_.row, (J1ij_t.diags())*sizeof(int)));
  // allocate values
  CUDA_CALL(hipMallocPitch((void**)&dev_J1ij_t_.val, &dev_J1ij_t_.pitch, (J1ij_t.rows())*sizeof(CudaFastFloat), J1ij_t.diags()));
  // copy rows
  CUDA_CALL(hipMemcpy(dev_J1ij_t_.row, J1ij_t.dia_offPtr(), (size_t)((J1ij_t.diags())*(sizeof(int))), hipMemcpyHostToDevice));
  // convert val array into CudaFastFloat which may be float or double
  std::vector<CudaFastFloat> float_values(J1ij_t.rows()*J1ij_t.diags(), 0.0);
  for (int i = 0; i < J1ij_t.rows()*J1ij_t.diags(); ++i) {
    float_values[i] = static_cast<CudaFastFloat>(J1ij_t.val(i));
  }
  // copy values
  CUDA_CALL(hipMemcpy2D(dev_J1ij_t_.val, dev_J1ij_t_.pitch, &float_values[0], J1ij_t.rows()*sizeof(CudaFastFloat), J1ij_t.rows()*sizeof(CudaFastFloat), J1ij_t.diags(), hipMemcpyHostToDevice));
  dev_J1ij_t_.pitch = dev_J1ij_t_.pitch/sizeof(CudaFastFloat);

//-----------------------------------------------------------------------------
// Transfer the the other arrays to the device
//-----------------------------------------------------------------------------

  ::output.write("  transfering array data to device\n");

  // spin arrays
  dev_s_        = jblib::CudaArray<double, 1>(s);
  dev_s_new_    = jblib::CudaArray<double, 1>(s);

  // field array
  jblib::Array<CudaFastFloat, 2> zero(num_spins, 3, 0.0);
  dev_h_        = jblib::CudaArray<CudaFastFloat, 1>(zero);

  // materials array
  jblib::Array<CudaFastFloat, 2> mat(num_spins, 4);
  jblib::Array<double, 1> sigma;

  sigma.resize(num_spins);
  for(int i = 0; i!=num_spins; ++i) {
    sigma(i) = sqrt( (2.0*boltzmann_si*alpha(i)) / (time_step_*mus(i)*mu_bohr_si) );
  }

  for(int i = 0; i!=num_spins; ++i){
    mat(i, 0) = static_cast<CudaFastFloat>(mus(i));
    mat(i, 1) = static_cast<CudaFastFloat>(gyro(i));
    mat(i, 2) = static_cast<CudaFastFloat>(alpha(i));
    mat(i, 3) = static_cast<CudaFastFloat>(sigma(i));
  }
  dev_mat_      = jblib::CudaArray<CudaFastFloat, 1>(mat);

  // anisotropy arrays
  jblib::Array<CudaFastFloat, 1> dz(num_spins);
  for (int i = 0; i < num_spins; ++i) {
    dz[i] = static_cast<CudaFastFloat>(globals::d2z[i]);
  }
  dev_d2z_ = jblib::CudaArray<CudaFastFloat, 1>(dz);

  for (int i = 0; i < num_spins; ++i) {
    dz[i] = static_cast<CudaFastFloat>(globals::d4z[i]);
  }
  dev_d4z_ = jblib::CudaArray<CudaFastFloat, 1>(dz);

  for (int i = 0; i < num_spins; ++i) {
    dz[i] = static_cast<CudaFastFloat>(globals::d6z[i]);
  }
  dev_d6z_ = jblib::CudaArray<CudaFastFloat, 1>(dz);

  ::output.write("\n");
}

void CudaSolver::run() {
}

void CudaSolver::compute_fields() {
  using namespace globals;

  // zero the field array
  hipMemsetAsync(dev_h_.data(), 0.0, num_spins3*sizeof(CudaFastFloat), ::cuda_streams[0]);

  if (optimize::use_fft) {
    cuda_realspace_to_kspace_mapping<<<(num_spins+BLOCKSIZE-1)/BLOCKSIZE, BLOCKSIZE>>>(dev_s_.data(), r_to_k_mapping_.data(), num_spins, num_kpoints_.x, num_kpoints_.y, num_kpoints_.z, dev_s3d_.data());

    if (hipfftExecD2Z(spin_fft_forward_transform, dev_s3d_.data(), dev_sq_.data()) != HIPFFT_SUCCESS) {
      jams_error("CUFFT failure executing spin_fft_forward_transform");
    }

    const int convolution_size = num_kpoints_.x*num_kpoints_.y*((num_kpoints_.z/2)+1);
    const int real_size = num_kpoints_.x*num_kpoints_.y*num_kpoints_.z;

    cuda_fft_convolution<<<(convolution_size+BLOCKSIZE-1)/BLOCKSIZE, BLOCKSIZE >>>(convolution_size, real_size, dev_wq_.data(), dev_sq_.data(), dev_hq_.data());
    if (hipfftExecZ2D(field_fft_backward_transform, dev_hq_.data(), dev_h3d_.data()) != HIPFFT_SUCCESS) {
      jams_error("CUFFT failure executing field_fft_backward_transform");
    }

    cuda_kspace_to_realspace_mapping<<<(num_spins+BLOCKSIZE-1)/BLOCKSIZE, BLOCKSIZE>>>(dev_h3d_.data(), r_to_k_mapping_.data(), num_spins, num_kpoints_.x, num_kpoints_.y, num_kpoints_.z, dev_h_.data());
  }

  hipStreamSynchronize(::cuda_streams[0]); // block until hipMemsetAsync is finished

  cuda_anisotropy_kernel<<<(num_spins+BLOCKSIZE-1)/BLOCKSIZE, BLOCKSIZE, 0, dev_streams_[1]>>>
  (num_spins, dev_d2z_.data(), dev_d4z_.data(), dev_d6z_.data(), dev_s_.data(), dev_h_.data());

  // bilinear interactions
  if(J1ij_t.nonZero() > 0){
    spmv_dia_kernel<<< dev_J1ij_t_.blocks, DIA_BLOCK_SIZE, 0, dev_streams_[0] >>>
    (num_spins3, num_spins3, J1ij_t.diags(), dev_J1ij_t_.pitch, 1.0, 1.0,
     dev_J1ij_t_.row, dev_J1ij_t_.val, dev_s_.data(), dev_h_.data());
  }

  // anisotropy interactions
}

CudaSolver::~CudaSolver() {
  CUDA_CALL(hipFree(dev_J1ij_t_.row));
  CUDA_CALL(hipFree(dev_J1ij_t_.col));
  CUDA_CALL(hipFree(dev_J1ij_t_.val));

  for (int i = 0; i < 2; ++i) {
    hipStreamDestroy(dev_streams_[i]);
  }
}

#include "hip/hip_runtime.h"
// Copyright 2014 Joseph Barker. All rights reserved.

#include <hipblas.h>

#include "core/globals.h"
#include "core/solver.h"
#include "core/cuda_solver.h"
#include "core/cuda_solver_kernels.h"


#include "core/consts.h"

#include "core/utils.h"
#include "core/cuda_defs.h"

#include "solvers/cuda_heunllg.h"
#include "solvers/heunllg.h"
#include "solvers/metropolismc.h"
#include "core/cuda_sparsematrix.h"


void CudaSolver::initialize(int argc, char **argv, double idt) {
  using namespace globals;

  Solver::initialize(argc, argv, idt);

  ::output.write("\ninitializing CUDA base solver\n");

  ::output.write("  initialising CUDA streams\n");

  is_cuda_solver_ = true;

  // if (hipblasCreate(&cublas_handle) != HIPBLAS_STATUS_SUCCESS) {
  //   jams_error("CudaSolver: CUBLAS initialization failed");
  // }

  dev_streams_ = new hipStream_t[2];

  for (int i = 0; i < 2; ++i) {
    if (hipStreamCreate(&dev_streams_[i]) != hipSuccess){
      jams_error("Failed to create CUDA stream in CudaSolver");
    }
  }


//-----------------------------------------------------------------------------
// fourier transforms
//-----------------------------------------------------------------------------

  // for (int i = 0; i < 3; ++i) {
    // num_kpoints_[i] = globals::wij.size(i);
  // }

  // jblib::Vec3<int> num_hermitian_kpoints = num_kpoints_;
  // num_hermitian_kpoints.z = (num_kpoints_.z/2) + 1;

  // globals::wq.resize(num_kpoints_.x, num_kpoints_.y, (num_kpoints_.z/2)+1, 3, 3);

  // ::output.write("  kspace dimensions: %d %d %d\n", num_kpoints_.x, num_kpoints_.y, num_kpoints_.z);

  // ::output.write("  FFT planning\n");

  // perform the wij -> wq transformation on the host
  // fftw_plan interaction_fft_transform  = fftw_plan_many_dft_r2c(3, &num_kpoints_[0], 9, wij.data(),  NULL, 9, 1, wq.data(), NULL, 9, 1, FFTW_ESTIMATE|FFTW_PRESERVE_INPUT);
  // ::output.write("  FFT transform interaction matrix\n");
  // fftw_execute(interaction_fft_transform);

  // ::output.write("  FFT transfering arrays to device\n");

  // convert fftw_complex data into hipfftDoubleComplex format and copy to the device
  // jblib::Array<hipfftDoubleComplex, 5> convert_wq(num_hermitian_kpoints.x, num_hermitian_kpoints.y, num_hermitian_kpoints.z, 3, 3);

  // for (int i = 0; i < globals::wq.elements(); ++i) {
  //   convert_wq[i].x = globals::wq[i][0];
  //   convert_wq[i].y = globals::wq[i][1];
  // }
  // dev_wq_ = jblib::CudaArray<hipfftDoubleComplex, 1>(convert_wq);

  // jblib::Array<double, 4> s3d(num_kpoints_.x, num_kpoints_.y, num_kpoints_.z, 3, 0.0);
  // dev_s3d_ = jblib::CudaArray<double, 1>(s3d);
  // dev_h3d_ = jblib::CudaArray<double, 1>(s3d);

  // dev_sq_.resize(num_hermitian_kpoints.x*num_hermitian_kpoints.y*num_hermitian_kpoints.z*3);
  // dev_hq_.resize(num_hermitian_kpoints.x*num_hermitian_kpoints.y*num_hermitian_kpoints.z*3);

  // r_to_k_mapping_ = jblib::CudaArray<int, 1>(lattice.kspace_inv_map_);


  // if (hipfftPlanMany(&spin_fft_forward_transform, 3, &num_kpoints_[0], &num_kpoints_[0], 3, 1, &num_hermitian_kpoints[0], 3, 1, HIPFFT_D2Z, 3) != HIPFFT_SUCCESS) {
  //   jams_error("CUFFT failure planning spin_fft_forward_transform");
  // }
  // if (cufftSetCompatibilityMode(spin_fft_forward_transform, CUFFT_COMPATIBILITY_NATIVE) != HIPFFT_SUCCESS) {
  //   jams_error("CUFFT failure changing to compatability mode native for spin_fft_forward_transform");
  // }
  // if (hipfftPlanMany(&field_fft_backward_transform, 3, &num_kpoints_[0], &num_hermitian_kpoints[0], 3, 1, &num_kpoints_[0], 3, 1, HIPFFT_Z2D, 3) != HIPFFT_SUCCESS) {
  //   jams_error("CUFFT failure planning field_fft_backward_transform");
  // }
  // if (cufftSetCompatibilityMode(field_fft_backward_transform, CUFFT_COMPATIBILITY_NATIVE) != HIPFFT_SUCCESS) {
  //   jams_error("CUFFT failure changing to compatability mode native for field_fft_backward_transform");
  // }

//-----------------------------------------------------------------------------
// Transfer the the other arrays to the device
//-----------------------------------------------------------------------------

  ::output.write("  transfering array data to device\n");

  // spin arrays
  dev_s_        = jblib::CudaArray<double, 1>(s);
  dev_s_new_    = jblib::CudaArray<double, 1>(s);

  // field array
  jblib::Array<double, 2> zero(num_spins, 3, 0.0);
  dev_h_        = jblib::CudaArray<double, 1>(zero);

  // materials array
  jblib::Array<double, 2> mat(num_spins, 4);
  jblib::Array<double, 1> sigma(num_spins);

  // sigma.resize(num_spins);
  for(int i = 0; i < num_spins; ++i) {
    sigma(i) = sqrt( (2.0*boltzmann_si*alpha(i)) / (time_step_*mus(i)*mu_bohr_si) );
  }

  for(int i = 0; i < num_spins; ++i){
    mat(i, 0) = static_cast<double>(mus(i));
    mat(i, 1) = static_cast<double>(gyro(i));
    mat(i, 2) = static_cast<double>(alpha(i));
    mat(i, 3) = static_cast<double>(sigma(i));
  }
  dev_mat_      = jblib::CudaArray<double, 1>(mat);

  ::output.write("\n");
}

void CudaSolver::run() {
}

void CudaSolver::compute_fields() {
  using namespace globals;

  // zero the field array
  hipMemsetAsync(dev_h_.data(), 0.0, num_spins3*sizeof(double), ::cuda_streams[0]);

  // if (optimize::use_fft) {
  //   cuda_realspace_to_kspace_mapping<<<(num_spins+BLOCKSIZE-1)/BLOCKSIZE, BLOCKSIZE>>>(dev_s_.data(), r_to_k_mapping_.data(), num_spins, num_kpoints_.x, num_kpoints_.y, num_kpoints_.z, dev_s3d_.data());

  //   if (hipfftExecD2Z(spin_fft_forward_transform, dev_s3d_.data(), dev_sq_.data()) != HIPFFT_SUCCESS) {
  //     jams_error("CUFFT failure executing spin_fft_forward_transform");
  //   }

  //   const int convolution_size = num_kpoints_.x*num_kpoints_.y*((num_kpoints_.z/2)+1);
  //   const int real_size = num_kpoints_.x*num_kpoints_.y*num_kpoints_.z;

  //   cuda_fft_convolution<<<(convolution_size+BLOCKSIZE-1)/BLOCKSIZE, BLOCKSIZE >>>(convolution_size, real_size, dev_wq_.data(), dev_sq_.data(), dev_hq_.data());
  //   if (hipfftExecZ2D(field_fft_backward_transform, dev_hq_.data(), dev_h3d_.data()) != HIPFFT_SUCCESS) {
  //     jams_error("CUFFT failure executing field_fft_backward_transform");
  //   }

  //   cuda_kspace_to_realspace_mapping<<<(num_spins+BLOCKSIZE-1)/BLOCKSIZE, BLOCKSIZE>>>(dev_h3d_.data(), r_to_k_mapping_.data(), num_spins, num_kpoints_.x, num_kpoints_.y, num_kpoints_.z, dev_h_.data());
  // }

  hipStreamSynchronize(::cuda_streams[0]); // block until hipMemsetAsync is finished

  for (std::vector<Hamiltonian*>::iterator it = hamiltonians_.begin() ; it != hamiltonians_.end(); ++it) {
    (*it)->calculate_fields();
  }

  const double alpha = 1.0;
  for (std::vector<Hamiltonian*>::iterator it = hamiltonians_.begin() ; it != hamiltonians_.end(); ++it) {
    hipblasDaxpy(globals::num_spins3, alpha, (*it)->dev_ptr_field(), 1, dev_h_.data(), 1);
  }
}

CudaSolver::~CudaSolver() {
  for (int i = 0; i < 2; ++i) {
    hipStreamDestroy(dev_streams_[i]);
  }
}

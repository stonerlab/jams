#include "hip/hip_runtime.h"
// Copyright 2014 Joseph Barker. All rights reserved.

#include "physics/cuda_metadynamics.h"

#include <libconfig.h++>

#include "core/globals.h"
#include "core/exception.h"


CudaMetadynamicsPhysics::CudaMetadynamicsPhysics(const libconfig::Setting &settings)
  : Physics(settings),
    debug_(false),
    dev_stream_(),
    dev_field_(globals::num_spins3),
    cv_theta(0),
    collective_variable_deriv(globals::num_spins, 3),
    gaussian_centers(),
    gaussian_width(0.2),
    gaussian_height(0.1),
    gaussian_placement_interval(1000)
  {

  output.write("  * CUDA metadynamics physics module\n");

  config.lookupValue("debug", debug_);

  if (debug_) {
    ::output.write("    DEBUG ON\n");  
  }

  if (hipStreamCreate(&dev_stream_) != hipSuccess){
    jams_error("Failed to create CUDA stream in CudaMetadynamicsPhysics");
  }

  // zero the field array
  if (hipMemsetAsync(dev_field_.data(), 0.0, globals::num_spins3*sizeof(double), dev_stream_) != hipSuccess) {
    throw cuda_api_exception("", __FILE__, __LINE__, __PRETTY_FUNCTION__);
  }

}

CudaMetadynamicsPhysics::~CudaMetadynamicsPhysics() {
}

void CudaMetadynamicsPhysics::update(const int &iterations, const double &time, const double &dt) {
  using namespace globals;

  // calculate collective variables
  calculate_collective_variables();

  if (iterations % gaussian_placement_interval == 0) {
    gaussian_centers.push_back(cv_theta);

    if (cv_theta - 2.0*gaussian_width < 0.0) {
      gaussian_centers.push_back(-cv_theta);
    } 

    if (cv_theta + 2.0*gaussian_width > kPi) {
      gaussian_centers.push_back(kPi + (kPi - cv_theta) );
    }
    
    output_gaussians(std::cerr);
  }

  calculate_fields();

  dev_field_.copy_from_host_array(field_);
}

void CudaMetadynamicsPhysics::calculate_collective_variables() {

  // DO THIS ON THE GPU EVENTUALLY
  Vec3 mag = {0.0, 0.0, 0.0};

  for (auto i = 0; i < globals::num_spins; ++i) {
    for (auto j = 0; j < 3; ++j) {
      mag[j] += globals::s(i, j);
    }
  }

  cv_theta = azimuthal_angle(mag);

  const auto mm = abs(mag);
  const auto mz = mag.z;

  auto m0 = 1.0 / (mm * mm * sqrt(1.0 - (mz * mz) / (mm * mm + 1e-5) ));
  //auto m0 = 1.0;
  if (isinf(m0)) {
    m0 = 1e100;
  }

  for (int i = 0; i < globals::num_spins; ++i) {
    collective_variable_deriv(i, 0) = m0 * (mz / mm) * globals::s(i, 0);
    collective_variable_deriv(i, 1) = m0 * (mz / mm) * globals::s(i, 1);
    collective_variable_deriv(i, 2) = m0 * (mz / mm) * globals::s(i, 2) - mm;
  }
}

void CudaMetadynamicsPhysics::calculate_fields() {

  auto potential_deriv = 0.0;
  for (auto it = gaussian_centers.begin(); it != gaussian_centers.end(); ++it){

    auto x = (cv_theta - (*it)) ;
    auto gaussian = gaussian_height * exp(-0.5 * x * x / (gaussian_width * gaussian_width));

    potential_deriv = potential_deriv - 0.5* gaussian * x / (gaussian_width * gaussian_width);
  }

  for (auto i = 0; i < globals::num_spins; ++i) {
    for (auto j = 0; j < 3; ++j) {
      field_(i, j) = -potential_deriv * collective_variable_deriv(i, j);
    }
  }

//  for (auto i = 0; i < globals::num_spins; ++i) {
//      std::cerr << field_(i, 0)<< "\t" << field_(i, 1) << "\t" << field_(i, 2) << std::endl;
//  }
//  exit(0);

}

void CudaMetadynamicsPhysics::output_gaussians(std::ostream &out) {
  auto theta = 0.0;
  auto delta_theta = gaussian_width/10.0;

  do {
    auto potential = 0.0;

    for (auto it = gaussian_centers.begin(); it != gaussian_centers.end(); ++it){
      auto x = (theta - (*it)) ;
      auto gaussian = gaussian_height * exp(-0.5 * x * x / (gaussian_width * gaussian_width));
      potential += gaussian;
    }

    std::cerr << theta << "\t" << potential << std::endl;

    theta += delta_theta;
  } while (theta < kPi);

  std::cerr << "\n\n";

}
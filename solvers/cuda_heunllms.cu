#include "hip/hip_runtime.h"
// Copyright 2014 Joseph Barker. All rights reserved.

#include "solvers/cuda_heunllms.h"

#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hipsparse.h>

#include <algorithm>
#include <cmath>

#include "core/consts.h"
#include "core/cuda_sparse.h"
#include "core/cuda_sparse_types.h"
#include "core/globals.h"

#include "solvers/cuda_heunllms_kernel.h"

#include "jblib/containers/array.h"

void CUDAHeunLLMSSolver::sync_device_data()
{
  using namespace globals;
  CUDA_CALL(hipMemcpy(s.data(), s_dev, (size_t)(num_spins3*sizeof(double)), hipMemcpyDeviceToHost));
}

void CUDAHeunLLMSSolver::initialize(int argc, char **argv, double idt)
{
  using namespace globals;

  // initialize base class
  Solver::initialize(argc, argv, idt);

  output.write("  * CUDA Heun LLMS solver (GPU)\n");

  sigma.resize(num_spins);

  libconfig::Setting &matcfg = config.lookup("materials");

  for(int i = 0; i<num_spins; ++i){
    int type_num = lattice.getType(i);
	omega_corr(i) = matcfg[type_num]["t_corr"];
    omega_corr(i) = 1.0/(gamma_electron_si*omega_corr(i));

    sigma(i) = sqrt( (2.0*boltzmann_si*alpha(i)*omega_corr(i)*omega_corr(i)) / (dt*mus(i)*mu_bohr_si) );

  	gyro(i) = matcfg[type_num]["gyro"];
	gyro(i) = -gyro(i)/mus(i);
  }


  //-------------------------------------------------------------------
  //  initialize hiprand
  //-------------------------------------------------------------------

  output.write("  * Initialising CURAND...\n");
  // hiprand generator
  CURAND_CALL(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));


  // TODO: set random seed from config
  const unsigned long long gpuseed = rng.uniform()*18446744073709551615ULL;
  CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen, gpuseed));
  CURAND_CALL(hiprandGenerateSeeds(gen));
  /*CUDA_CALL(cudaThreadSetLimit(hipLimitStackSize, 1024));*/
  /*CUDA_CALL(hipDeviceSynchronize());*/


  //-------------------------------------------------------------------
  //  Allocate device memory
  //-------------------------------------------------------------------

  output.write("  * Converting MAP to DIA\n");
  J1ij_s.convertMAP2DIA();
  J1ij_t.convertMAP2DIA();
  J2ij_s.convertMAP2DIA();
  J2ij_t.convertMAP2DIA();
  output.write("  * J1ij scalar matrix memory (DIA): %f MB\n", J1ij_s.calculateMemory());
  output.write("  * J1ij tensor matrix memory (DIA): %f MB\n", J1ij_t.calculateMemory());
  output.write("  * J2ij scalar matrix memory (DIA): %f MB\n", J2ij_s.calculateMemory());
  output.write("  * J2ij tensor matrix memory (DIA): %f MB\n", J2ij_t.calculateMemory());

  output.write("  * Converting J4 MAP to CSR\n");
  /*J4ijkl_s.convertMAP2CSR();*/
  output.write("  * J4ijkl scalar matrix memory (CSR): %f MB\n", J4ijkl_s.calculateMemoryUsage());


  output.write("  * Allocating device memory...\n");
  // spin arrays
  CUDA_CALL(hipMalloc((void**)&s_dev, num_spins3*sizeof(double)));
  CUDA_CALL(hipMalloc((void**)&sf_dev, num_spins3*sizeof(float)));
  CUDA_CALL(hipMalloc((void**)&s_new_dev, num_spins3*sizeof(double)));

  // stochastic process arrays
  CUDA_CALL(hipMalloc((void**)&u_dev, num_spins3*sizeof(double)));
  CUDA_CALL(hipMalloc((void**)&u_new_dev, num_spins3*sizeof(double)));
  CUDA_CALL(hipMalloc((void**)&omega_corr_dev, num_spins*sizeof(float)));

  // field arrays
  CUDA_CALL(hipMalloc((void**)&h_dev, num_spins3*sizeof(float)));
  CUDA_CALL(hipMalloc((void**)&e_dev, num_spins3*sizeof(float)));

  if(num_spins3%2 == 0) {
    // wiener processes
    CUDA_CALL(hipMalloc((void**)&w_dev, num_spins3*sizeof(float)));
  } else {
    CUDA_CALL(hipMalloc((void**)&w_dev, (num_spins3+1)*sizeof(float)));
  }


  // bilinear scalar
  allocate_transfer_dia(J1ij_s, J1ij_s_dev);

  // bilinear tensor
  allocate_transfer_dia(J1ij_t, J1ij_t_dev);

  // biquadratic scalar
  allocate_transfer_dia(J2ij_s, J2ij_s_dev);

  // bilinear tensor
  allocate_transfer_dia(J2ij_t, J2ij_t_dev);

  allocate_transfer_csr_4d(J4ijkl_s, J4ijkl_s_dev);

  // material properties
  CUDA_CALL(hipMalloc((void**)&mat_dev, num_spins*4*sizeof(float)));

  //-------------------------------------------------------------------
  //  Copy data to device
  //-------------------------------------------------------------------

  output.write("  * Copying data to device memory...\n");
  // initial spins
  jblib::Array<float, 2> sf(num_spins, 3);
  for(int i = 0; i<num_spins; ++i) {
    for(int j = 0; j<3; ++j) {
      sf(i, j) = static_cast<float>(s(i, j));
    }
  }

  CUDA_CALL(hipMemcpy(s_dev, s.data(), (size_t)(num_spins3*sizeof(double)), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(sf_dev, sf.data(), (size_t)(num_spins3*sizeof(float)), hipMemcpyHostToDevice));

  jblib::Array<float, 2> mat(num_spins, 4);
  // material properties
  for(int i = 0; i<num_spins; ++i){
    mat(i, 0) = mus(i);
    mat(i, 1) = gyro(i);
    mat(i, 2) = alpha(i);
    mat(i, 3) = sigma(i);
  }

  CUDA_CALL(hipMemcpy(mat_dev, mat.data(), (size_t)(num_spins*4*sizeof(float)), hipMemcpyHostToDevice));

  eng.resize(num_spins, 3);


  //-------------------------------------------------------------------
  //  initialize arrays to zero
  //-------------------------------------------------------------------
  for(int i = 0; i<num_spins; ++i) {
    for(int j = 0; j<3; ++j) {
      sf(i, j) = 0.0;
    }
  }

  CUDA_CALL(hipMemcpy(w_dev, sf.data(), (size_t)(num_spins3*sizeof(float)), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(h_dev, sf.data(), (size_t)(num_spins3*sizeof(float)), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(e_dev, sf.data(), (size_t)(num_spins3*sizeof(float)), hipMemcpyHostToDevice));

  jblib::Array<float, 1> tmp(num_spins);
  for(int i = 0; i<num_spins; ++i) {
  	tmp(i) = omega_corr(i);
  }
  CUDA_CALL(hipMemcpy(omega_corr_dev, tmp.data(), (size_t)(num_spins*sizeof(float)), hipMemcpyHostToDevice));

  jblib::Array<double, 2> u(num_spins, 3);
  for(int i = 0; i<num_spins; ++i) {
    for(int j = 0; j<3; ++j) {
      u(i, j) = 0.0;
    }
  }
  CUDA_CALL(hipMemcpy(u_dev, u.data(), (size_t)(num_spins3*sizeof(double)), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(u_new_dev, u.data(), (size_t)(num_spins3*sizeof(double)), hipMemcpyHostToDevice));


  nblocks = (num_spins+BLOCKSIZE-1)/BLOCKSIZE;

  J1ij_s_dev.blocks = std::min<int>(DIA_BLOCK_SIZE, (num_spins+DIA_BLOCK_SIZE-1)/DIA_BLOCK_SIZE);
  J1ij_t_dev.blocks = std::min<int>(DIA_BLOCK_SIZE, (num_spins3+DIA_BLOCK_SIZE-1)/DIA_BLOCK_SIZE);

  J2ij_s_dev.blocks = std::min<int>(DIA_BLOCK_SIZE, (num_spins+DIA_BLOCK_SIZE-1)/DIA_BLOCK_SIZE);
  J2ij_t_dev.blocks = std::min<int>(DIA_BLOCK_SIZE, (num_spins3+DIA_BLOCK_SIZE-1)/DIA_BLOCK_SIZE);

  J4ijkl_s_dev.blocks = std::min<int>(CSR_4D_BLOCK_SIZE, (num_spins+CSR_4D_BLOCK_SIZE-1)/CSR_4D_BLOCK_SIZE);

  initialized = true;
}

void CUDAHeunLLMSSolver::run()
{
  using namespace globals;

  // generate wiener trajectories
  float stmp = sqrt(temperature);

  if(temperature > 0.0) {
    if(num_spins3%2 == 0) {
      CURAND_CALL(hiprandGenerateNormal(gen, w_dev, num_spins3, 0.0f, stmp));
    } else {
      CURAND_CALL(hiprandGenerateNormal(gen, w_dev, (num_spins3+1), 0.0f, stmp));
    }
  }

  jblib::Array<float, 2> tmp(num_spins, 3);


  // calculate interaction fields (and zero field array)

  //CUDA_CALL(hipBindTexture(0, tex_x_float, sf_dev));

  float beta=0;
  // bilinear scalar
  if(J1ij_s.nonZero() > 0){
    bilinear_scalar_interaction_dia_kernel<<< J1ij_s_dev.blocks, DIA_BLOCK_SIZE >>>(num_spins, num_spins,
      J1ij_s.diags(), J1ij_s_dev.pitch, 1.0, beta, J1ij_s_dev.row, J1ij_s_dev.val, sf_dev, h_dev);
    beta = 1.0;
  }

  // bilinear tensor
  if(J1ij_t.nonZero() > 0){
    spmv_dia_kernel<<< J1ij_t_dev.blocks, DIA_BLOCK_SIZE >>>(num_spins3, num_spins3,
      J1ij_t.diags(), J1ij_t_dev.pitch, beta, 1.0, J1ij_t_dev.row, J1ij_t_dev.val, sf_dev, h_dev);
    beta = 1.0;
  }

  // biquadratic scalar
  if(J2ij_s.nonZero() > 0){
    biquadratic_scalar_dia_kernel<<< J2ij_s_dev.blocks, DIA_BLOCK_SIZE >>>(num_spins, num_spins,
      J2ij_s.diags(), J2ij_s_dev.pitch, 2.0, beta, J2ij_s_dev.row, J2ij_s_dev.val, sf_dev, h_dev);
    beta = 1.0;
  }

  // biquadratic tensor
  if(J2ij_t.nonZero() > 0){
    spmv_dia_kernel<<< J2ij_t_dev.blocks, DIA_BLOCK_SIZE >>>(num_spins3, num_spins3,
      J2ij_t.diags(), J2ij_t_dev.pitch, 2.0, beta, J2ij_t_dev.row, J2ij_t_dev.val, sf_dev, h_dev);
    beta = 1.0;
  }

  if(J4ijkl_s.nonZeros() > 0){
    fourspin_scalar_interaction_csr_kernel<<< J4ijkl_s_dev.blocks, CSR_4D_BLOCK_SIZE>>>(num_spins, num_spins, 1.0, beta,
        J4ijkl_s_dev.pointers, J4ijkl_s_dev.coords, J4ijkl_s_dev.val, sf_dev, h_dev);
    beta = 1.0;
  }

  //CUDA_CALL(hipUnbindTexture(tex_x_float));

  // integrate
  cuda_heun_llms_kernelA<<<nblocks, BLOCKSIZE>>>
    (
      s_dev,
      sf_dev,
      s_new_dev,
      h_dev,
      w_dev,
	  u_dev,
	  u_new_dev,
	  omega_corr_dev,
      mat_dev,
      h_app[0],
      h_app[1],
      h_app[2],
      num_spins,
      dt
    );
	//   Array2D<float> hf(num_spins, 3);
	//   Array2D<float> sf(num_spins, 3);
	//   CUDA_CALL(hipMemcpy(hf.data(), h_dev, (size_t)(num_spins3*sizeof(float)), hipMemcpyDeviceToHost));
	// CUDA_CALL(hipMemcpy(sf.data(), sf_dev, (size_t)(num_spins3*sizeof(float)), hipMemcpyDeviceToHost));
	//
	//   for(int i = 0; i<num_spins; ++i){
	//       std::cout<<i<<"\t"<<sf(i, 0)<<"\t"<<sf(i, 1)<<"\t"<<sf(i, 2)<<"\t"<<hf(i, 0)<<"\t"<<hf(i, 1)<<"\t"<<hf(i, 2)<<std::endl;
	//   }


  // calculate interaction fields (and zero field array)

  beta=0.0;
  // bilinear scalar
  if(J1ij_s.nonZero() > 0){
    bilinear_scalar_interaction_dia_kernel<<< J1ij_s_dev.blocks, DIA_BLOCK_SIZE >>>(num_spins, num_spins,
      J1ij_s.diags(), J1ij_s_dev.pitch, 1.0, beta, J1ij_s_dev.row, J1ij_s_dev.val, sf_dev, h_dev);
    beta = 1.0;
  }

  // bilinear tensor
  if(J1ij_t.nonZero() > 0){
    spmv_dia_kernel<<< J1ij_t_dev.blocks, DIA_BLOCK_SIZE >>>(num_spins3, num_spins3,
      J1ij_t.diags(), J1ij_t_dev.pitch, 1.0, beta, J1ij_t_dev.row, J1ij_t_dev.val, sf_dev, h_dev);
    beta = 1.0;
  }

  // biquadratic scalar
  if(J2ij_s.nonZero() > 0){
    biquadratic_scalar_dia_kernel<<< J2ij_s_dev.blocks, DIA_BLOCK_SIZE >>>(num_spins, num_spins,
      J2ij_s.diags(), J2ij_s_dev.pitch, 2.0, beta, J2ij_s_dev.row, J2ij_s_dev.val, sf_dev, h_dev);
    beta = 1.0;
  }

  // biquadratic tensor
  if(J2ij_t.nonZero() > 0){
    spmv_dia_kernel<<< J2ij_t_dev.blocks, DIA_BLOCK_SIZE >>>(num_spins3, num_spins3,
      J2ij_t.diags(), J2ij_t_dev.pitch, 2.0, beta, J2ij_t_dev.row, J2ij_t_dev.val, sf_dev, h_dev);
    beta = 1.0;
  }

  if(J4ijkl_s.nonZeros() > 0){
    fourspin_scalar_interaction_csr_kernel<<< J4ijkl_s_dev.blocks, CSR_4D_BLOCK_SIZE>>>(num_spins, num_spins, 1.0, beta,
        J4ijkl_s_dev.pointers, J4ijkl_s_dev.coords, J4ijkl_s_dev.val, sf_dev, h_dev);
    beta = 1.0;
  }

  /*Array2D<float> hf(num_spins, 3);*/
  /*Array2D<float> sf(num_spins, 3);*/
  /*CUDA_CALL(hipMemcpy(hf.data(), h_dev, (size_t)(num_spins3*sizeof(float)), hipMemcpyDeviceToHost));*/
  /*CUDA_CALL(hipMemcpy(sf.data(), sf_dev, (size_t)(num_spins3*sizeof(float)), hipMemcpyDeviceToHost));*/

  /*for(int i = 0; i<num_spins; ++i){*/
      /*std::cout<<i<<sf(i, 0)<<"\t"<<sf(i, 1)<<"\t"<<sf(i, 2)<<"\t"<<hf(i, 0)<<"\t"<<hf(i, 1)<<"\t"<<hf(i, 2)<<std::endl;*/
  /*}*/

  //CUDA_CALL(hipUnbindTexture(tex_x_float));

  cuda_heun_llms_kernelB<<<nblocks, BLOCKSIZE>>>
    (
      s_dev,
      sf_dev,
      s_new_dev,
      h_dev,
      w_dev,
	  u_dev,
	  u_new_dev,
	  omega_corr_dev,
      mat_dev,
      h_app[0],
      h_app[1],
      h_app[2],
      num_spins,
      dt
    );
  iteration++;
}

void CUDAHeunLLMSSolver::compute_total_energy(double &e1_s, double &e1_t, double &e2_s, double &e2_t, double &e4_s){
  using namespace globals;

}

CUDAHeunLLMSSolver::~CUDAHeunLLMSSolver()
{
  hiprandDestroyGenerator(gen);

  hipsparseStatus_t status;

  status = hipsparseDestroyMatDescr(descra);
  if (status != HIPSPARSE_STATUS_SUCCESS) {
    jams_error("CUSPARSE matrix destruction failed");
  }
  CUDA_CALL(hipDeviceSynchronize());

  status = hipsparseDestroy(handle);
  if (status != HIPSPARSE_STATUS_SUCCESS) {
    jams_error("CUSPARSE Library destruction failed");
  }
  CUDA_CALL(hipDeviceSynchronize());

  //-------------------------------------------------------------------
  //  Free device memory
  //-------------------------------------------------------------------

  free_dia(J1ij_s_dev);
  free_dia(J1ij_t_dev);
  free_dia(J2ij_s_dev);
  free_dia(J2ij_t_dev);
  free_csr_4d(J4ijkl_s_dev);

  // spin arrays
  CUDA_CALL(hipFree(s_dev));
  CUDA_CALL(hipFree(sf_dev));
  CUDA_CALL(hipFree(s_new_dev));

  CUDA_CALL(hipFree(u_dev));
  CUDA_CALL(hipFree(u_new_dev));
  CUDA_CALL(hipFree(omega_corr_dev));

  // field arrays
  CUDA_CALL(hipFree(h_dev));
  CUDA_CALL(hipFree(e_dev));

  // wiener processes
  CUDA_CALL(hipFree(w_dev));


  // material arrays
  CUDA_CALL(hipFree(mat_dev));
}


#include "hip/hip_runtime.h"
// Copyright 2014 Joseph Barker. All rights reserved.

#include "solvers/cuda_semillg.h"

#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hipsparse.h>

#include <algorithm>
#include <cmath>

#include "core/consts.h"
#include "core/cuda_sparse.h"
#include "core/cuda_sparse_types.h"
#include "core/globals.h"

#include "solvers/cuda_semillg_kernel.h"

#include "jblib/containers/array.h"

void CUDASemiLLGSolver::sync_device_data()
{
  using namespace globals;
  CUDA_CALL(hipDeviceSynchronize());
  CUDA_CALL(hipMemcpy(s.data(), s_dev, (size_t)(num_spins3*sizeof(double)), hipMemcpyDeviceToHost));
  CUDA_CALL(hipDeviceSynchronize());
}

void CUDASemiLLGSolver::initialize(int argc, char **argv, double idt)
{
  using namespace globals;

  // initialize base class
  Solver::initialize(argc, argv, idt);

  sigma.resize(num_spins);

  for(int i = 0; i<num_spins; ++i) {
    sigma(i) = sqrt( (2.0*boltzmann_si*alpha(i)) / (dt*mus(i)*mu_bohr_si) );
  }


  output.write("  * CUDA Semi-Implicit LLG solver (GPU)\n");

  //-------------------------------------------------------------------
  //  initialize hiprand
  //-------------------------------------------------------------------

  output.write("  * Initialising CURAND...\n");
  // hiprand generator
  CURAND_CALL(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));


  // TODO: set random seed from config
  const unsigned long long gpuseed = rng.uniform()*18446744073709551615ULL;
  CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen, gpuseed));
  CURAND_CALL(hiprandGenerateSeeds(gen));
  CUDA_CALL(cudaThreadSetLimit(hipLimitStackSize, 1024));
  CUDA_CALL(hipDeviceSynchronize());


  //-------------------------------------------------------------------
  //  Allocate device memory
  //-------------------------------------------------------------------

  output.write("  * Converting MAP to DIA\n");
  J1ij_s.convertMAP2DIA();
  J1ij_t.convertMAP2DIA();
  J2ij_s.convertMAP2DIA();
  J2ij_t.convertMAP2DIA();
  output.write("  * J1ij scalar matrix memory (DIA): %f MB\n", J1ij_s.calculateMemory());
  output.write("  * J1ij tensor matrix memory (DIA): %f MB\n", J1ij_t.calculateMemory());
  output.write("  * J2ij scalar matrix memory (DIA): %f MB\n", J2ij_s.calculateMemory());
  output.write("  * J2ij tensor matrix memory (DIA): %f MB\n", J2ij_t.calculateMemory());

  /*output.write("  * Converting J4 MAP to CSR\n");*/
  /*J4ijkl_s.convertMAP2CSR();*/
  output.write("  * J2ij scalar matrix memory (DIA): %f MB\n", J4ijkl_s.calculateMemoryUsage());


  output.write("  * Allocating device memory...\n");
  // spin arrays
  CUDA_CALL(hipMalloc((void**)&s_dev, num_spins3*sizeof(double)));
  CUDA_CALL(hipMalloc((void**)&sf_dev, num_spins3*sizeof(float)));
  CUDA_CALL(hipMalloc((void**)&s_new_dev, num_spins3*sizeof(double)));

  // field arrays
  CUDA_CALL(hipMalloc((void**)&h_dev, num_spins3*sizeof(float)));
  CUDA_CALL(hipMalloc((void**)&e_dev, num_spins3*sizeof(float)));

  if(num_spins3%2 == 0) {
    // wiener processes
    CUDA_CALL(hipMalloc((void**)&w_dev, num_spins3*sizeof(float)));
  } else {
    CUDA_CALL(hipMalloc((void**)&w_dev, (num_spins3+1)*sizeof(float)));
  }


  // bilinear scalar
  allocate_transfer_dia(J1ij_s, J1ij_s_dev);

  // bilinear tensor
  allocate_transfer_dia(J1ij_t, J1ij_t_dev);

  // biquadratic scalar
  allocate_transfer_dia(J2ij_s, J2ij_s_dev);

  // bilinear tensor
  allocate_transfer_dia(J2ij_t, J2ij_t_dev);

  allocate_transfer_csr_4d(J4ijkl_s, J4ijkl_s_dev);

  // material properties
  CUDA_CALL(hipMalloc((void**)&mat_dev, num_spins*4*sizeof(float)));

  //-------------------------------------------------------------------
  //  Copy data to device
  //-------------------------------------------------------------------

  output.write("  * Copying data to device memory...\n");
  // initial spins
  jblib::Array<float, 2> sf(num_spins, 3);
  for(int i = 0; i<num_spins; ++i) {
    for(int j = 0; j<3; ++j) {
      sf(i, j) = static_cast<float>(s(i, j));
    }
  }
  CUDA_CALL(hipMemcpy(s_dev, s.data(), (size_t)(num_spins3*sizeof(double)), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(sf_dev, sf.data(), (size_t)(num_spins3*sizeof(float)), hipMemcpyHostToDevice));

  jblib::Array<float, 2> mat(num_spins, 4);
  // material properties
  for(int i = 0; i<num_spins; ++i){
    mat(i, 0) = mus(i);
    mat(i, 1) = gyro(i);
    mat(i, 2) = alpha(i);
    mat(i, 3) = sigma(i);
  }
  CUDA_CALL(hipMemcpy(mat_dev, mat.data(), (size_t)(num_spins*4*sizeof(float)), hipMemcpyHostToDevice));

  eng.resize(num_spins, 3);


  //-------------------------------------------------------------------
  //  initialize arrays to zero
  //-------------------------------------------------------------------
  for(int i = 0; i<num_spins; ++i) {
    for(int j = 0; j<3; ++j) {
      sf(i, j) = 0.0;
    }
  }

  CUDA_CALL(hipMemcpy(w_dev, sf.data(), (size_t)(num_spins3*sizeof(float)), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(h_dev, sf.data(), (size_t)(num_spins3*sizeof(float)), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(e_dev, sf.data(), (size_t)(num_spins3*sizeof(float)), hipMemcpyHostToDevice));

  nblocks = (num_spins+BLOCKSIZE-1)/BLOCKSIZE;

  J1ij_s_dev.blocks = std::min<int>(DIA_BLOCK_SIZE, (num_spins+DIA_BLOCK_SIZE-1)/DIA_BLOCK_SIZE);
  J1ij_t_dev.blocks = std::min<int>(DIA_BLOCK_SIZE, (num_spins3+DIA_BLOCK_SIZE-1)/DIA_BLOCK_SIZE);

  J2ij_s_dev.blocks = std::min<int>(DIA_BLOCK_SIZE, (num_spins+DIA_BLOCK_SIZE-1)/DIA_BLOCK_SIZE);
  J2ij_t_dev.blocks = std::min<int>(DIA_BLOCK_SIZE, (num_spins3+DIA_BLOCK_SIZE-1)/DIA_BLOCK_SIZE);

  J4ijkl_s_dev.blocks = std::min<int>(CSR_4D_BLOCK_SIZE, (num_spins+CSR_4D_BLOCK_SIZE-1)/CSR_4D_BLOCK_SIZE);

  initialized = true;

}


void CUDASemiLLGSolver::run()
{
  using namespace globals;

  // copy s_dev to s_new_dev
  // NOTE: this is part of the SEMILLG scheme
  CUDA_CALL(hipDeviceSynchronize());
  CUDA_CALL(hipMemcpy(s_new_dev, s_dev, (size_t)(num_spins3*sizeof(double)), hipMemcpyDeviceToDevice));

  // generate wiener trajectories
  float stmp = sqrt(temperature());

  if(temperature() > 0.0) {
    if(num_spins3%2 == 0) {
      CURAND_CALL(hiprandGenerateNormal(gen, w_dev, num_spins3, 0.0f, stmp));
    } else {
      CURAND_CALL(hiprandGenerateNormal(gen, w_dev, (num_spins3+1), 0.0f, stmp));
    }
  }
  CUDA_CALL(hipDeviceSynchronize());

    // calculate interaction fields (and zero field array)

  float beta=0.0;
  // bilinear scalar
  if(J1ij_s.nonZero() > 0){
    bilinear_scalar_interaction_dia_kernel<<< J1ij_s_dev.blocks, DIA_BLOCK_SIZE >>>(num_spins, num_spins,
      J1ij_s.diags(), J1ij_s_dev.pitch, 1.0, beta, J1ij_s_dev.row, J1ij_s_dev.val, sf_dev, h_dev);
    beta = 1.0;
  }

  // bilinear tensor
  if(J1ij_t.nonZero() > 0){
    spmv_dia_kernel<<< J1ij_t_dev.blocks, DIA_BLOCK_SIZE >>>(num_spins3, num_spins3,
      J1ij_t.diags(), J1ij_t_dev.pitch, 1.0, beta, J1ij_t_dev.row, J1ij_t_dev.val, sf_dev, h_dev);
    beta = 1.0;
  }

  // biquadratic scalar
  if(J2ij_s.nonZero() > 0){
    biquadratic_scalar_dia_kernel<<< J2ij_s_dev.blocks, DIA_BLOCK_SIZE >>>(num_spins, num_spins,
      J2ij_s.diags(), J2ij_s_dev.pitch, 2.0, beta, J2ij_s_dev.row, J2ij_s_dev.val, sf_dev, h_dev);
    beta = 1.0;
  }

  // biquadratic tensor
  if(J2ij_t.nonZero() > 0){
    spmv_dia_kernel<<< J2ij_t_dev.blocks, DIA_BLOCK_SIZE >>>(num_spins3, num_spins3,
      J2ij_t.diags(), J2ij_t_dev.pitch, 2.0, beta, J2ij_t_dev.row, J2ij_t_dev.val, sf_dev, h_dev);
    beta = 1.0;
  }

  if(J4ijkl_s.nonZeros() > 0){
    fourspin_scalar_interaction_csr_kernel<<< J4ijkl_s_dev.blocks, CSR_4D_BLOCK_SIZE>>>(num_spins, num_spins, 1.0, beta,
        J4ijkl_s_dev.pointers, J4ijkl_s_dev.coords, J4ijkl_s_dev.val, sf_dev, h_dev);
    beta = 1.0;
  }

  //CUDA_CALL(hipUnbindTexture(tex_x_float));

  // integrate
  cuda_semi_llg_kernelA<<<nblocks, BLOCKSIZE>>>
    (
      s_dev,
      sf_dev,
      h_dev,
      w_dev,
      mat_dev,
      h_app[0],
      h_app[1],
      h_app[2],
      num_spins,
      dt
    );
  CUDA_CALL(hipDeviceSynchronize());

   // calculate interaction fields (and zero field array)

  //CUDA_CALL(hipBindTexture(0, tex_x_float, sf_dev));

  beta=0.0;
  // bilinear scalar
  if(J1ij_s.nonZero() > 0){
    bilinear_scalar_interaction_dia_kernel<<< J1ij_s_dev.blocks, DIA_BLOCK_SIZE >>>(num_spins, num_spins,
      J1ij_s.diags(), J1ij_s_dev.pitch, 1.0, beta, J1ij_s_dev.row, J1ij_s_dev.val, sf_dev, h_dev);
    beta = 1.0;
  }

  // bilinear tensor
  if(J1ij_t.nonZero() > 0){
    spmv_dia_kernel<<< J1ij_t_dev.blocks, DIA_BLOCK_SIZE >>>(num_spins3, num_spins3,
      J1ij_t.diags(), J1ij_t_dev.pitch, beta, 1.0, J1ij_t_dev.row, J1ij_t_dev.val, sf_dev, h_dev);
    beta = 1.0;
  }

  // biquadratic scalar
  if(J2ij_s.nonZero() > 0){
    biquadratic_scalar_dia_kernel<<< J2ij_s_dev.blocks, DIA_BLOCK_SIZE >>>(num_spins, num_spins,
      J2ij_s.diags(), J2ij_s_dev.pitch, 2.0, beta, J2ij_s_dev.row, J2ij_s_dev.val, sf_dev, h_dev);
    beta = 1.0;
  }

  // biquadratic tensor
  if(J2ij_t.nonZero() > 0){
    spmv_dia_kernel<<< J2ij_t_dev.blocks, DIA_BLOCK_SIZE >>>(num_spins3, num_spins3,
      J2ij_t.diags(), J2ij_t_dev.pitch, 2.0, beta, J2ij_t_dev.row, J2ij_t_dev.val, sf_dev, h_dev);
    beta = 1.0;
  }

  if(J4ijkl_s.nonZeros() > 0){
    fourspin_scalar_interaction_csr_kernel<<< J4ijkl_s_dev.blocks, CSR_4D_BLOCK_SIZE>>>(num_spins, num_spins, 1.0, beta,
        J4ijkl_s_dev.pointers, J4ijkl_s_dev.coords, J4ijkl_s_dev.val, sf_dev, h_dev);
    beta = 1.0;
  }

  //CUDA_CALL(hipUnbindTexture(tex_x_float));
  cuda_semi_llg_kernelB<<<nblocks, BLOCKSIZE>>>
    (
      s_dev,
      sf_dev,
      s_new_dev,
      h_dev,
      w_dev,
      mat_dev,
      h_app[0],
      h_app[1],
      h_app[2],
      num_spins,
      dt
    );
  CUDA_CALL(hipDeviceSynchronize());

  iteration++;
}

CUDASemiLLGSolver::~CUDASemiLLGSolver()
{
      hiprandDestroyGenerator(gen);

  hipsparseStatus_t status;

  status = hipsparseDestroyMatDescr(descra);
  if (status != HIPSPARSE_STATUS_SUCCESS) {
    jams_error("CUSPARSE matrix destruction failed");
  }
  CUDA_CALL(hipDeviceSynchronize());

  status = hipsparseDestroy(handle);
  if (status != HIPSPARSE_STATUS_SUCCESS) {
    jams_error("CUSPARSE Library destruction failed");
  }
  CUDA_CALL(hipDeviceSynchronize());

  //-------------------------------------------------------------------
  //  Free device memory
  //-------------------------------------------------------------------

  free_dia(J1ij_s_dev);
  free_dia(J1ij_t_dev);
  free_dia(J2ij_s_dev);
  free_dia(J2ij_t_dev);
  free_csr_4d(J4ijkl_s_dev);

  // spin arrays
  CUDA_CALL(hipFree(s_dev));
  CUDA_CALL(hipFree(sf_dev));
  CUDA_CALL(hipFree(s_new_dev));

  // field arrays
  CUDA_CALL(hipFree(h_dev));
  CUDA_CALL(hipFree(e_dev));

  // wiener processes
  CUDA_CALL(hipFree(w_dev));


  // material arrays
  CUDA_CALL(hipFree(mat_dev));

}



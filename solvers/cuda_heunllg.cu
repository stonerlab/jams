// Copyright 2014 Joseph Barker. All rights reserved.

#include "solvers/cuda_heunllg.h"

#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hipsparse.h>

#include <algorithm>
#include <cmath>

#include "core/consts.h"
#include "core/exception.h"
#include "core/cuda_sparsematrix.h"
#include "core/globals.h"
#include "core/thermostat.h"

#include "solvers/cuda_heunllg_kernel.h"

#include "jblib/containers/array.h"

void CUDAHeunLLGSolver::initialize(int argc, char **argv, double idt)
{
  using namespace globals;

  CudaSolver::initialize(argc, argv, idt);

  ::output.write("\ninitializing CUDA Heun LLG solver\n");

  if (::config.exists("sim.thermostat")) {
    thermostat_ = Thermostat::create(::config.lookup("sim.thermostat"));
  } else {
    ::output.write("  DEFAULT thermostat\n");
    thermostat_ = Thermostat::create("CUDA_LANGEVIN_WHITE");
  }

  nblocks = (num_spins+BLOCKSIZE-1)/BLOCKSIZE;

  ::output.write("\n");

  if(hipStreamCreate(&dev_stream_) != hipSuccess) {
    throw cuda_api_exception("", __FILE__, __LINE__, __PRETTY_FUNCTION__);
  }

  if(hipMemcpyToSymbol(HIP_SYMBOL(dev_dt), &time_step_, sizeof(double)) != hipSuccess) {
    throw cuda_api_exception("", __FILE__, __LINE__, __PRETTY_FUNCTION__);
  }

  if(hipMemcpyToSymbol(HIP_SYMBOL(dev_num_spins), &globals::num_spins, sizeof(unsigned int)) != hipSuccess) {
    throw cuda_api_exception("", __FILE__, __LINE__, __PRETTY_FUNCTION__);
  }
}

void CUDAHeunLLGSolver::run()
{
  using namespace globals;

  dim3 block_size;
  block_size.x = 85;
  block_size.y = 3;

  dim3 grid_size;
  grid_size.x = (globals::num_spins + block_size.x - 1) / block_size.x;
  grid_size.y = (3 + block_size.y - 1) / block_size.y;

    hipMemcpyAsync(dev_s_old_.data(),           // void *               dst
               dev_s_.data(),               // const void *         src
               num_spins3*sizeof(double),   // size_t               count
               hipMemcpyDeviceToDevice,    // enum hipMemcpyKind  kind
               dev_stream_);                   // device stream

#ifdef DEBUG
  if (hipPeekAtLastError() != hipSuccess) {
    throw cuda_api_exception("", __FILE__, __LINE__, __PRETTY_FUNCTION__);
  }
#endif



    thermostat_->set_temperature(physics_module_->temperature());
    thermostat_->update();

    compute_fields();

    cuda_heun_llg_kernelA<<<grid_size, block_size>>>
        (dev_s_.data(), dev_ds_dt_.data(), dev_s_old_.data(),
          dev_h_.data(), thermostat_->noise(),
          dev_gyro_.data(), dev_alpha_.data());

#ifdef DEBUG
    if (hipPeekAtLastError() != hipSuccess) {
      throw cuda_api_exception("", __FILE__, __LINE__, __PRETTY_FUNCTION__);
    }
#endif

    compute_fields();

    cuda_heun_llg_kernelB<<<grid_size, block_size>>>
      (dev_s_.data(), dev_ds_dt_.data(), dev_s_old_.data(),
        dev_h_.data(), thermostat_->noise(),
        dev_gyro_.data(), dev_alpha_.data());

#ifdef DEBUG
    if (hipPeekAtLastError() != hipSuccess) {
      throw cuda_api_exception("", __FILE__, __LINE__, __PRETTY_FUNCTION__);
    }
#endif


    iteration_++;
}


CUDAHeunLLGSolver::~CUDAHeunLLGSolver()
{
}

